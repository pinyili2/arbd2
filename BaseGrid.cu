#include "hip/hip_runtime.h"

//////////////////////////////////////////////////////////////////////
// Grid base class that does just the basics.
// Author: Jeff Comer <jcomer2@illinois.edu>

#include "BaseGrid.h"
#include <hip/hip_runtime.h>


#define STRLEN 512

// Initialize the variables that get used a lot.
// Also, allocate the main value array.
void BaseGrid::init() {
	basisInv = basis.inverse();
	nynz = ny*nz;
	size = nx*ny*nz;
	val = new float[size];
}
BaseGrid::BaseGrid() {
	BaseGrid tmp(Matrix3(),Vector3(),1,1,1);
	val = new float[1];
	*this = tmp;									// TODO: verify that this is OK
	
	// basis = Matrix3();
	// origin = Vector3();
	// nx = 1;
	// ny = 1;
	// nz = 1;
	
	// init();
	// zero();
}

// The most obvious of constructors.
BaseGrid::BaseGrid(Matrix3 basis0, Vector3 origin0, int nx0, int ny0, int nz0) {
	basis = basis0;
	origin = origin0;
	nx = abs(nx0);
	ny = abs(ny0);
	nz = abs(nz0);
	
	init();
	zero();
}

// Make an orthogonal grid given the box dimensions and resolution.
BaseGrid::BaseGrid(Vector3 box, float dx) {
	dx = fabsf(dx);
	box.x = fabsf(box.x);
	box.y = fabsf(box.y);
	box.z = fabsf(box.z);

	// Tile the grid into the system box.
	// The grid spacing is always a bit smaller than dx.
	nx = int(ceilf(box.x/dx));
	ny = int(ceilf(box.y/dx));
	nz = int(ceilf(box.z/dx));
	if (nx <= 0) nx = 1;
	if (ny <= 0) ny = 1;
	if (nz <= 0) nz = 1;
	basis = Matrix3(box.x/nx, box.y/ny, box.z/nz);
	origin = -0.5f*box;

	init();
	zero();
}

// The box gives the system geometry.
// The grid point numbers define the resolution.
BaseGrid::BaseGrid(Matrix3 box, int nx0, int ny0, int nz0) {
	nx = nx0;
	ny = ny0;
	nz = nz0;

	// Tile the grid into the system box.
	if (nx <= 0) nx = 1;
	if (ny <= 0) ny = 1;
	if (nz <= 0) nz = 1;
	basis = Matrix3(box.ex()/nx, box.ey()/ny, box.ez()/nz);
	origin = -0.5f*(box.ex() + box.ey() + box.ez());

	init();
	zero();
}

// The box gives the system geometry.
// dx is the approx. resolution.
// The grid spacing is always a bit larger than dx.
BaseGrid::BaseGrid(Matrix3 box, Vector3 origin0, float dx) {
	dx = fabs(dx);
	
	// Tile the grid into the system box.
	// The grid spacing is always a bit larger than dx.
	nx = int(floor(box.ex().length()/dx))-1;
	ny = int(floor(box.ey().length()/dx))-1;
	nz = int(floor(box.ez().length()/dx))-1;
	if (nx <= 0) nx = 1;
	if (ny <= 0) ny = 1;
	if (nz <= 0) nz = 1;

	basis = Matrix3(box.ex()/nx, box.ey()/ny, box.ez()/nz);
	origin = origin0;

	init();
	zero();
}

// The box gives the system geometry.
// dx is the approx. resolution.
// The grid spacing is always a bit smaller than dx.
BaseGrid::BaseGrid(Matrix3 box, float dx) {
	dx = fabs(dx);
	
	// Tile the grid into the system box.
	// The grid spacing is always a bit smaller than dx.
	nx = int(ceilf(box.ex().length()/dx));
	ny = int(ceilf(box.ey().length()/dx));
	nz = int(ceilf(box.ez().length()/dx));
	if (nx <= 0) nx = 1;
	if (ny <= 0) ny = 1;
	if (nz <= 0) nz = 1;

	basis = Matrix3(box.ex()/nx, box.ey()/ny, box.ez()/nz);
	origin = -0.5f*(box.ex() + box.ey() + box.ez());

	init();
	zero();
}

// Make an exact copy of a grid.
BaseGrid::BaseGrid(const BaseGrid& g) {
	nx = g.nx;
	ny = g.ny;
	nz = g.nz;
	basis = g.basis;
	origin = g.origin;
	
	init();
	for (int i = 0; i < size; i++) val[i] = g.val[i];
}

BaseGrid BaseGrid::mult(const BaseGrid& g) {
	for (int i = 0; i < size; i++) val[i] *= g.val[i];
	return *this;
}

BaseGrid& BaseGrid::operator=(const BaseGrid& g) {
	delete[] val;
	val = NULL;
	nx = g.nx;
	ny = g.ny;
	nz = g.nz;
	basis = g.basis;
	origin = g.origin;
	
	init();
	for (int i = 0; i < size; i++) val[i] = g.val[i];

	return *this;
}


// Make a copy of a grid, but at a different resolution.
BaseGrid::BaseGrid(const BaseGrid& g, int nx0, int ny0, int nz0) : nx(nx0),  ny(ny0), nz(nz0) {
	if (nx <= 0) nx = 1;
	if (ny <= 0) ny = 1;
	if (nz <= 0) nz = 1;

	// Tile the grid into the box of the template grid.
	Matrix3 box = g.getBox();
	basis = Matrix3(box.ex()/nx, box.ey()/ny, box.ez()/nz);

	origin = g.origin;
	init();

	// Do an interpolation to obtain the values.
	for (int i = 0; i < size; i++) {
		Vector3 r = getPosition(i);
		val[i] = g.interpolatePotential(r);
	}
}

// Read a grid from a file.
BaseGrid::BaseGrid(const char* fileName) {
		 // Open the file.
	FILE* inp = fopen(fileName,"r");
	if (inp == NULL) {
		printf("ERROR BaseGrid::BaseGrid Couldn't open file %s.\n",fileName);
		exit(-1);
	}
	//printf("Reading dx file %s...\n", fileName);
	
	size = 0;
	nx = 0;
	ny = 0;
	nz = 0;
	basis = Matrix3(1.0f);
	origin = Vector3(0.0f);    

	int n = 0;
	float x, y, z;
	char line[STRLEN];
	int p, nRead;
	int deltaCount = 0;
	Vector3 base[3];
	while (fgets(line, STRLEN, inp) != NULL) {
		// Ignore comments.
		int len = strlen(line);
		if (line[0] == '#') continue;
		if (len < 2) continue;
	
		if (isInt(line[0]) && n < size) {
// Read grid values.
nRead = sscanf(line, "%f %f %f", &x, &y, &z);
if (size > 0) {
	switch(nRead) {
	case 1:
		val[n] = x;
		n++;
		if (n != size) {
			printf("ERROR BaseGrid::BaseGrid Improperly formatted dx file %s.\n", fileName);
			printf("line `%s'\n", line);
		}
		break;
	case 2:
		val[n] = x;
		val[n+1] = y;
		n += 2;
		if (n != size) {
			printf("ERROR BaseGrid::BaseGrid Improperly formatted dx file %s.\n", fileName);
			printf("line `%s'\n", line);
		}
		break;
	case 3:
		val[n] = x;
		val[n+1] = y;
		val[n+2] = z;
		n += 3;
		break;
	}
}
		} else if (len > 5) {
// Read the grid parameters.
char start[6];
for (int i = 0; i < 5; i++) start[i] = line[i];
start[5] = '\0';

if(strcmp("origi", start) == 0) {
	// Get an origin line.
	p = firstSpace(line, STRLEN);
	sscanf(&(line[p+1]), "%f %f %f", &x, &y, &z);
	origin = Vector3(x, y, z);
	//printf("Origin: %.12g %.12g %.12g\n", x, y, z);
} else if(strcmp("delta", start) == 0) {
	// Get a delta matrix line.
	p = firstSpace(line, STRLEN);
	sscanf(&(line[p+1]), "%f %f %f", &x, &y, &z);
	base[deltaCount] = Vector3(x, y, z);
	//printf("Delta %d: %.12g %.12g %.12g\n", deltaCount, x, y, z);
	if (deltaCount < 2) deltaCount = deltaCount + 1;
} else if(strcmp("objec", start) == 0) {
	//printf("%s", line);
	// Get the system dimensions.
	if (line[7] != '1') continue;
	int read = sscanf(line, "object 1 class gridpositions counts %d %d %d\n", &nx, &ny, &nz);
	//printf("Size: %d %d %d\n", nx, ny, nz);
	if (read == 3) {
		size = nx*ny*nz;
		nynz = ny*nz;
		val = new float[size];
		zero();
	}
}
		}
	}
	fclose(inp);

	basis = Matrix3(base[0], base[1], base[2]);
	basisInv = basis.inverse();
	if (size == 0 || n != size) {
		printf("ERROR BaseGrid::BaseGrid Improperly formatted dx file %s.\n",fileName);
		printf("declared size: %d, items: %d\n", size, n);
		printf("first value: %10g, final value: %.10g\n", val[0], val[n-1]);
		exit(-1);
	}
}  

// Write without comments.
void BaseGrid::write(const char* fileName) const {
	write(fileName, "");
}

// Writes the grid as a file in the dx format.
void BaseGrid::write(const char* fileName, const char* comments) const {
	// Open the file.
	FILE* out = fopen(fileName,"w");
	if (out == NULL) {
		printf("ERROR BaseGrid::write Couldn't open file %s.\n",fileName);
		exit(-1);
	}

	// Write the header.
	fprintf(out, "# %s\n", comments);
	fprintf(out, "object 1 class gridpositions counts %d %d %d\n", nx, ny, nz);
	fprintf(out, "origin %.12g %.12g %.12g\n", origin.x, origin.y, origin.z);
	fprintf(out, "delta %.12g %.12g %.12g\n", basis.exx, basis.eyx, basis.ezx);
	fprintf(out, "delta %.12g %.12g %.12g\n", basis.exy, basis.eyy, basis.ezy);
	fprintf(out, "delta %.12g %.12g %.12g\n", basis.exz, basis.eyz, basis.ezz);
	fprintf(out, "object 2 class gridconnections counts %d %d %d\n", nx, ny, nz);
	fprintf(out, "object 3 class array type float rank 0 items %d data follows\n", size);
	
	// Write the data.
	int penultima = 3*(size/3);
	int mod = size - penultima;

	int i;
	for (i = 0; i < penultima; i+=3) {
		fprintf(out, "%.12g %.12g %.12g\n", val[i], val[i+1], val[i+2]);
	}
	if (mod == 1) {
		fprintf(out, "%.12g\n", val[size-1]);
	} else if (mod == 2) {
		fprintf(out, "%.12g %.12g\n", val[size-2], val[size-1]);
	}
	fclose(out);
}

// Writes the grid data as a single column in the order:
// nx ny nz ox oy oz dxx dyx dzx dxy dyy dzy dxz dyz dzz val0 val1 val2 ...
void BaseGrid::writeData(const char* fileName) {
	// Open the file.
	FILE* out = fopen(fileName,"w");
	if (out == NULL) {
		printf("Couldn't open file %s.\n",fileName);
		exit(-1);
	}

	fprintf(out, "%d\n%d\n%d\n", nx, ny, nz);
	fprintf(out, "%.12g\n%.12g\n%.12g\n", origin.x, origin.y, origin.z);
	fprintf(out, "%.12g\n%.12g\n%.12g\n", basis.exx, basis.eyx, basis.ezx);
	fprintf(out, "%.12g\n%.12g\n%.12g\n", basis.exx, basis.eyx, basis.ezx);
	fprintf(out, "%.12g\n%.12g\n%.12g\n", basis.exx, basis.eyx, basis.ezx);

	for (int i = 0; i < size; i++) fprintf(out, "%.12g\n", val[i]);
	fclose(out);
}

// Write the valies in a single column.
void BaseGrid::writePotential(const char* fileName) const {
	FILE* out = fopen(fileName, "w");
	for (int i = 0; i < size; i++) fprintf(out, "%.12g\n", val[i]);
	fclose(out);
}

BaseGrid::~BaseGrid() {
	if (val != NULL)
		delete[] val;
}

void BaseGrid::zero() {
	for (int i = 0; i < size; i++) val[i] = 0.0f;
}

bool BaseGrid::setValue(int j, float v) {
	if (j < 0 || j >= size) return false;
	val[j] = v;
	return true;
}

bool BaseGrid::setValue(int ix, int iy, int iz, float v) {
	if (ix < 0 || ix >= nx) return false;
	if (iy < 0 || iy >= ny) return false;
	if (iz < 0 || iz >= nz) return false;
	int j = iz + iy*nz + ix*ny*nz;

	val[j] = v;
	return true;
}

float BaseGrid::getValue(int j) const {
	if (j < 0 || j >= size) return 0.0f;
	return val[j];
}

float BaseGrid::getValue(int ix, int iy, int iz) const {
	if (ix < 0 || ix >= nx) return 0.0f;
	if (iy < 0 || iy >= ny) return 0.0f;
	if (iz < 0 || iz >= nz) return 0.0f;
	
	int j = iz + iy*nz + ix*ny*nz;
	return val[j];
}

Vector3 BaseGrid::getPosition(int ix, int iy, int iz) const {
	return basis.transform(Vector3(ix, iy, iz)) + origin;
}

Vector3 BaseGrid::getPosition(int j) const {
	int iz = j%nz;
	int iy = (j/nz)%ny;
	int ix = j/(nz*ny);

	return basis.transform(Vector3(ix, iy, iz)) + origin;
}

// Does the point r fall in the grid?
// Obviously this is without periodic boundary conditions.
bool BaseGrid::inGrid(Vector3 r) const {
	Vector3 l = basisInv.transform(r-origin);

	if (l.x < 0.0f || l.x >= nx) return false;
	if (l.y < 0.0f || l.y >= ny) return false;
	if (l.z < 0.0f || l.z >= nz) return false;
	return true;
}

bool BaseGrid::inGridInterp(Vector3 r) const {
	Vector3 l = basisInv.transform(r-origin);

	if (l.x < 2.0f || l.x >= nx-3.0f) return false;
	if (l.y < 2.0f || l.y >= ny-3.0f) return false;
	if (l.z < 2.0f || l.z >= nz-3.0f) return false;
	return true;
}

Vector3 BaseGrid::transformTo(Vector3 r) const {
	return basisInv.transform(r-origin);
}
Vector3 BaseGrid::transformFrom(Vector3 l) const {
	return basis.transform(l) + origin;
}

IndexList BaseGrid::index(int j) const {
	int iz = j%nz;
	int iy = (j/nz)%ny;
	int ix = j/(nz*ny);
	IndexList ret;
	ret.add(ix);
	ret.add(iy);
	ret.add(iz);
	return ret;
}
int BaseGrid::indexX(int j) const { return j/(nz*ny); }
int BaseGrid::indexY(int j) const { return (j/nz)%ny; }
int BaseGrid::indexZ(int j) const { return j%nz; }
int BaseGrid::index(int ix, int iy, int iz) const { return iz + iy*nz + ix*ny*nz; }

int BaseGrid::index(Vector3 r) const {
	Vector3 l = basisInv.transform(r-origin);
	
	int ix = int(floor(l.x));
	int iy = int(floor(l.y));
	int iz = int(floor(l.z));

	ix = wrap(ix, nx);
	iy = wrap(iy, ny);
	iz = wrap(iz, nz);
	
	return iz + iy*nz + ix*ny*nz;
}

int BaseGrid::nearestIndex(Vector3 r) const {
	Vector3 l = basisInv.transform(r-origin);
	
	int ix = int(floorf(l.x + 0.5f));
	int iy = int(floorf(l.y + 0.5f));
	int iz = int(floorf(l.z + 0.5f));

	ix = wrap(ix, nx);
	iy = wrap(iy, ny);
	iz = wrap(iz, nz);
	
	return iz + iy*nz + ix*ny*nz;
}

// A matrix defining the basis for the entire system.
Matrix3 BaseGrid::getBox() const {
	return Matrix3(nx*basis.ex(), ny*basis.ey(), nz*basis.ez());
} 
// The longest diagonal of the system.
Vector3 BaseGrid::getExtent() const {
	return basis.transform(Vector3(nx,ny,nz));
}
// The longest diagonal of the system.
float BaseGrid::getDiagonal() const {
	return getExtent().length();
}
// The position farthest from the origin.
Vector3 BaseGrid::getDestination() const {
	return basis.transform(Vector3(nx,ny,nz)) + origin;
}
// The center of the grid.
Vector3 BaseGrid::getCenter() const {
	return basis.transform(Vector3(0.5f*nx,0.5f*ny,0.5f*nz)) + origin;
}
// The volume of a single cell.
float BaseGrid::getCellVolume() const {
	return fabs(basis.det());
}
// The volume of the entire system.
float BaseGrid::getVolume() const {
	return getCellVolume()*size;
}
Vector3 BaseGrid::getCellDiagonal() const {
	return basis.ex() + basis.ey() + basis.ez();
}

// Add a fixed value to the grid.
void BaseGrid::shift(float s) {
	for (int i = 0; i < size; i++) val[i] += s;
}

// Multiply the grid by a fixed value.
void BaseGrid::scale(float s) {
	for (int i = 0; i < size; i++) val[i] *= s;
}

// Get the mean of the entire grid.
float BaseGrid::mean() const {
	float sum = 0.0f;
	for (int i = 0; i < size; i++) sum += val[i];
	return sum/size;
}

// Compute the average profile along an axis.
// Assumes that the grid axis with index "axis" is aligned with the world axis of index "axis".
void BaseGrid::averageProfile(const char* fileName, int axis) {
	FILE* out = fopen(fileName,"w");
	if (out == NULL) {
		printf("Couldn't open file %s.\n",fileName);
		exit(-1);
	}

	int dir0 = wrap(axis, 3);
	int dir1 = (axis+1)%3;
	int dir2 = (axis+2)%3;

	int jump[3];
	jump[0] = nynz;
	jump[1] = nz;
	jump[2] = 1;

	int n[3];
	n[0] = nx;
	n[1] = ny;
	n[2] = nz;
 
	for (int i0 = 0; i0 < n[dir0]; i0++) {
		float sum = 0;

		for (int i1 = 0; i1 < n[dir1]; i1++) {
			for (int i2 = 0; i2 < n[dir2]; i2++) {
				int j = i0*jump[dir0] + i1*jump[dir1] + i2*jump[dir2];
				sum += val[j];
			}
		}
		
		float v = sum/(n[dir1]*n[dir2]);
		float x = 0.0f;
		switch (dir0) {
		case 0:
			x = origin.x + i0*basis.exx;
			break;
		case 1:
			x = origin.y + i0*basis.eyy;
			break;
		case 2:
			x = origin.z + i0*basis.ezz;
			break;
		}
		fprintf(out, "%0.10g %0.10g\n", x, v);
	}

	fclose(out);
}

// Get the potential at the closest node.
float BaseGrid::getPotential(Vector3 pos) const {
	// Find the nearest node.
	int j = nearestIndex(pos);

	return val[j];
}

bool BaseGrid::crop(int x0, int y0, int z0, int x1, int y1, int z1, bool keep_origin) {
	if (x0 < 0 || x0 >= 2 * nx) x0 = 0;
	if (y0 < 0 || y0 >= 2 * ny) y0 = 0;
	if (z0 < 0 || z0 >= 2 * nz) z0 = 0;
	if (x1 < 0 || x1 >= 2 * nx) x1 = 2 * nx - 1;
	if (y1 < 0 || y1 >= 2 * ny) y1 = 2 * ny - 1;
	if (z1 < 0 || z1 >= 2 * nz) z1 = 2 * nz - 1;
	printf("Cropping to (%d, %d, %d) -> (%d, %d, %d)\n", x0, y0, z0, x1, y1, z1);

	if (x0 >= x1 || y0 >= y1 || z0 >= z1)
		return false;

	int new_nx = x1 - x0 + 1;
	int new_ny = y1 - y0 + 1;
	int new_nz = z1 - z0 + 1;
	int new_size = new_nx * new_ny * new_nz;
	float *new_val = new float[new_size];

	int ind = 0;
	int nynz = ny * nz;
	for (int i = x0; i < x1; i++)
		for (int j = y0; j < y1; j++)
			for (int k = z0; k < z1; k++) {
				int ind1 = k + j * nz + i * nynz;
				new_val[ind++] = val[ind1];
			}

	if (!keep_origin)
		origin += basis.transform(Vector3(x0, y0, z0));
	nx = new_nx;
	ny = new_ny;
	nz = new_nz;
	size = new_size;
	delete[] val;
	val = new_val;

	return true;
}

// Added by Rogan for times when simpler calculations are required.
float BaseGrid::interpolatePotentialLinearly(Vector3 pos) const {
	// Find the home node.
	Vector3 l = basisInv.transform(pos - origin);
	int homeX = int(floorf(l.x));
	int homeY = int(floorf(l.y));
	int homeZ = int(floorf(l.z));
	
	// Get the array jumps.
	int jump[3];
	jump[0] = nz*ny;
	jump[1] = nz;
	jump[2] = 1;

	// Shift the indices in the home array.
	int home[3];
	home[0] = homeX;
	home[1] = homeY;
	home[2] = homeZ;

	// Get the grid dimensions.
	int g[3];
	g[0] = nx;
	g[1] = ny;
	g[2] = nz;

	// Get the interpolation coordinates.
	float w[3];
	w[0] = l.x - homeX;
	w[1] = l.y - homeY;
	w[2] = l.z - homeZ;

	// Find the values at the neighbors.
	float g1[2][2][2];
	for (int ix = 0; ix < 2; ix++) {
		for (int iy = 0; iy < 2; iy++) {
			for (int iz = 0; iz < 2; iz++) {
				// Wrap around the periodic boundaries. 
				int jx = ix + home[0];
				jx = wrap(jx, g[0]);
				int jy = iy + home[1];
				jy = wrap(jy, g[1]);
				int jz = iz + home[2];
				jz = wrap(jz, g[2]);
				
				int ind = jz*jump[2] + jy*jump[1] + jx*jump[0];
				g1[ix][iy][iz] = val[ind];
			}
		}
	}

	// Mix along x.
	float g2[2][2];
	for (int iy = 0; iy < 2; iy++) {
		for (int iz = 0; iz < 2; iz++) {
			// p = w[0] * g[0][iy][iz] + (1-w[0]) * g[1][iy][iz]
			g2[iy][iz] = (1.0f-w[0])*g1[0][iy][iz] + w[0]*g1[1][iy][iz];
		}
	}

	// Mix along y.
	float g3[2];
	for (int iz = 0; iz < 2; iz++) {
		g3[iz] = (1.0f-w[1])*g2[0][iz] + w[1]*g2[1][iz];
	}

	// DEBUG
	//printf("(0,0,0)=%.1f (0,0,1)=%.1f (0,1,0)=%.1f (0,1,1)=%.1f (1,0,0)=%.1f (1,0,1)=%.1f (1,1,0)=%.1f (1,1,1)=%.1f ",
	//   g1[0][0][0], g1[0][0][1], g1[0][1][0], g1[0][1][1], g1[1][0][0], g1[1][0][1], g1[1][1][0], g1[1][1][1] );
	//printf ("%.2f\n",(1.0-w[2])*g3[0] + w[2]*g3[1]);

	// Mix along z
	return (1.0f-w[2])*g3[0] + w[2]*g3[1];
}



Vector3 BaseGrid::wrapDiffNearest(Vector3 r) const {
	Vector3 l = basisInv.transform(r);
	l.x = wrapDiff(l.x, nx);
	l.y = wrapDiff(l.y, ny);
	l.z = wrapDiff(l.z, nz);

	float length2 = basis.transform(l).length2();

	for (int dx = -1; dx <= 1; dx++) {
		for (int dy = -1; dy <= 1; dy++) {
			for (int dz = -1; dz <= 1; dz++) {
				//if (dx == 0 && dy == 0 && dz == 0) continue;
				Vector3 tmp = Vector3(l.x+dx*nx, l.y+dy*ny, l.z+dz*nz);
				if (basis.transform(tmp).length2() < length2) {
					l = tmp;
					length2 = basis.transform(l).length2();
				}
			}
		}
	}

	return basis.transform(l);
}


// Includes the home node.
// indexBuffer must have a size of at least 27.
void BaseGrid::getNeighbors(int j, int* indexBuffer) const {
	int jx = indexX(j);
	int jy = indexY(j);
	int jz = indexZ(j);

	int k = 0;
	for (int ix = -1; ix <= 1; ix++) {
		for (int iy = -1; iy <= 1; iy++) {
			for (int iz = -1; iz <= 1; iz++) {
				int ind = wrap(jz+iz,nz) + nz*wrap(jy+iy,ny) + nynz*wrap(jx+ix,nx);
				indexBuffer[k] = ind;
				k++;
			}
		}
	}
}

// Get the values at the neighbors of a node.
// Note that homeX, homeY, and homeZ do not need to be wrapped,
// since we do it here.
void BaseGrid::getNeighborValues(NeighborList* neigh, int homeX, int homeY, int homeZ) const {
	for (int ix = -1; ix <= 1; ix++) {
		for (int iy = -1; iy <= 1; iy++) {
			for (int iz = -1; iz <= 1; iz++) {
				int ind = wrap(homeZ+iz,nz) + nz*wrap(homeY+iy,ny) + nynz*wrap(homeX+ix,nx);
				neigh->v[ix+1][iy+1][iz+1] = val[ind];
			}
		}
	}
}  
