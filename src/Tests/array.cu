#include "hip/hip_runtime.h"
#include <float.h>
#include <iostream>
#include <cstdio>

// #include "useful.h"
#include "SignalManager.h"
#include "Types.h"
#include <hip/hip_runtime.h>
#include <nvfunctional>

#include <catch2/catch_test_macros.hpp>
#include <catch2/benchmark/catch_benchmark.hpp>
#include <catch2/matchers/catch_matchers_floating_point.hpp>

namespace Tests::TestArray {
    // enum BinaryOp_t { ADD, CROSS, DOT, SUB, FINAL };
    // BinaryOp_t& operator++(BinaryOp_t& op) { return op = static_cast<BinaryOp_t>( 1+static_cast<int>(op) ); }

    // std::string get_binary_op_name( BinaryOp_t op ) {
    // 	switch (op) {
    // 	case ADD:
    // 	    return "add";
    // 	case SUB:
    // 	    return "subtract";
    // 	case CROSS:
    // 	    return "cross";
    // 	case DOT:
    // 	    return "dot";
    // 	}
    // 	return std::string(""); // (static_cast<int>(op)));
    // }

    // template<typename R, typename T, typename U>
    // __host__ __device__ nvstd::function<R(T,U)> get_binary_op_func( BinaryOp_t op) {
    // 	switch (op) {
    // 	case ADD:
    // 	    return [] (T a, U b) {return static_cast<R>(b+a);};
    // 	case SUB:
    // 	    return [] (T a, U b) {return static_cast<R>(b-a);};
    // 	case CROSS:
    // 	    return [] (T a, U b) {return static_cast<R>(b.cross(a));};
    // 	case DOT:
    // 	    return [] (T a, U b) {return static_cast<R>(b.dot(a));};
    // 	default:
    // 	    assert(false);
    // 	}
    // 	return [] (T a, U b) {return static_cast<R>(b+a);};
    // }

    // template<typename R, typename T, typename U>
    // __global__ void binary_op_test_kernel( BinaryOp_t op, R* result, T in1, U in2 ) {
    // 	nvstd::function<R(T,U)> fn = get_binary_op_func<R,T,U>(op);
    // 	if (blockIdx.x == 0) {
    // 	    *result = fn(in1,in2);
    // 	}
    // }

    // template<typename T, typename U>
    // void check_vectors_equal( T&& cpu, U&& gpu) {
    // 	CHECK( type_name<decltype(cpu)>() == type_name<decltype(gpu)>() ); // should be unneccesary
    // 	CHECK( cpu.x == gpu.x );
    // 	CHECK( cpu.y == gpu.y );
    // 	CHECK( cpu.z == gpu.z );
    // 	CHECK( cpu.w == gpu.w );
    // }

    // template<typename A, typename B>
    // void run_tests() {
    // 	using T = Vector3_t<A>;
    // 	using U = Vector3_t<B>;
    // 	using R = std::common_type_t<T,U>;
    
    // 	T v1(1,1.005,0);
    // 	U v2(0,2,0);
    // 	R *gpu_result_d, gpu_result, cpu_result;
    // 	hipMalloc((void **)&gpu_result_d, sizeof(R));

    // 	for (BinaryOp_t op = ADD; op < FINAL; ++op) {
    // 	    INFO( get_binary_op_name( op ) );
    // 	    binary_op_test_kernel<R,T,U><<<1,1>>>(op, gpu_result_d, v1, v2);
    // 	    hipMemcpy(&gpu_result, gpu_result_d, sizeof(R), hipMemcpyDeviceToHost);
    // 	    hipDeviceSynchronize();
	
    // 	    // Get cpu_result
    // 	    cpu_result = (get_binary_op_func<R,T,U>(op))(v1,v2);

    // 	    // Check consistency
    // 	    check_vectors_equal(cpu_result, gpu_result);
    // 	}
    // 	hipFree(gpu_result_d);
    // }

    // template <typename T>
    // void print_enable_if_value_helper(std::true_type) {
    // 	std::cout << "has_copy_to_cuda is true" << std::endl;
    // }

    // template <typename T>
    // void print_enable_if_value_helper(std::false_type) {
    // 	std::cout << "has_copy_to_cuda is false" << std::endl;
    // }

    // template <typename T>
    // void print_enable_if_value_helper(std::true_type) {
    // 	std::cout << "has_copy_to_cuda is true" << std::endl;
    // }

    // template <typename T>
    // void print_enable_if_value_helper(std::false_type) {
    // 	std::cout << "has_copy_to_cuda is false" << std::endl;
    // }

    // template <typename T>
    // void print_enable_if_value() {
    // 	print_enable_if_value_helper<has_copy_to_cuda<T>>(typename has_copy_to_cuda<T>::type{});
    // }

    
    template<typename T> __host__ __device__ 
    void print_it(T x) { printf("Unsupported type\n"); }
    template<> __host__ __device__
    void print_it(const int x) { printf("int %d\n", x); }
    template<> __host__ __device__
    void print_it(const long int x) { printf("long int %ld\n", x); }
    template<> __host__ __device__
    void print_it(const float x) { printf("float %f\n", x); }
    template<> __host__ __device__
    void print_it(const double x) { printf("double %lf\n", x); }
    template<> __host__ __device__
    void print_it(const Vector3&& x) { x.print(); }
    template<> __host__ __device__
    void print_it(const Vector3& x) { x.print(); }
    
    template <typename T>
    void print_enable_if_value() {
	if (has_copy_to_cuda<T>::value) {
	    std::cout << "has_copy_to_cuda is true" << std::endl;
	} else {
	    std::cout << "has_copy_to_cuda is false" << std::endl;
	}
    }

    template<typename T>
    Array<T> allocate_array_host(size_t num) {
	Array<T> arr(num);
	return arr;
    }

    template<typename T>
    Array<T>* allocate_array_device(size_t num) {
	Array<T> arr(num);
	return arr.copy_to_cuda();
    }

    template<typename T>
    T* allocate_plain_array_host(size_t num) {
	T* arr = new T[num];
	return arr;
    }
    template<typename T>
    T* allocate_plain_array_device(size_t num) {
	T* arr = allocate_plain_array_host<T>(num);
	T* arr_d;
	size_t sz = sizeof(T)*num;
	gpuErrchk(hipMalloc(&arr_d, sz));
	gpuErrchk(hipMemcpy(arr_d, arr, sz, hipMemcpyHostToDevice));
	delete[] arr;
	return arr_d;
    }
    
    template<typename T>
    HOST DEVICE void inline _copy_helper(size_t& idx, T* __restrict__ out, const T* __restrict__ inp) {
	out[idx] = inp[idx];
    }

    // HOST DEVICE void inline _copy_helper(size_t& idx, float* __restrict__ out, const float* __restrict__ inp) {
    // 	out[idx] = inp[idx];
    // }
    template<typename T>
    HOST DEVICE void inline _copy_helper(size_t& idx, Array<T>* __restrict__ out, const Array<T>* __restrict__ inp) {
	(*out)[idx] = (*inp)[idx];
    }

    
    template<typename T>
    __global__ void copy_kernel(size_t num, T* __restrict__ out, const T* __restrict__ inp) {
	for (size_t i = threadIdx.x+blockIdx.x*blockDim.x; i < num; i+=blockDim.x*gridDim.x) {
	    _copy_helper(i, out, inp);
	}
    }

    template<typename T>
    void call_copy_kernel(size_t num, T* __restrict__ out, const T* __restrict__ inp, size_t block_size=256) {
	copy_kernel<<<block_size,1,0>>>(num, out, inp);
	gpuErrchk( hipDeviceSynchronize() );
    }
    
    // Array<T> _copy_array_cuda(size_t num) {
    // 	Array<T> arr(num);
    // 	return arr;
    // }

    
    TEST_CASE( "Test Array assignment and copy_to_cuda", "[Array]" ) {
	{
	    // Creation and copy assignment
	    Array<Vector3> a = allocate_array_host<Vector3>(10);
	}

	{
	    // Allocation and deallocation
	    VectorArr a(10);
	    a[0] = Vector3(1);
	    // a[0].print();
	    // a[1].print();
	    a[3] = Vector3(3);
	    // a[3].print();

	    VectorArr* a_d = a.copy_to_cuda();
	    VectorArr b(0);
	    VectorArr* b_d = b.copy_to_cuda();
	    VectorArr a_d_h = a_d->copy_from_cuda(a_d);
	    VectorArr b_d_h = b_d->copy_from_cuda(b_d);
		    
	    // a_d_h[0].print();
	    // a_d_h[1].print();
	    // a_d_h[3].print();

	    REQUIRE( a[1] == a_d_h[1] );
	    REQUIRE( a[3] == a_d_h[3] );

	    VectorArr::remove_from_cuda(a_d);
	    VectorArr::remove_from_cuda(b_d);

	    print_enable_if_value<int>();  // Replace VectorArr with your actual type
	    print_enable_if_value<Vector3>();  // Replace VectorArr with your actual type
	    print_enable_if_value<VectorArr>();  // Replace VectorArr with your actual type
	    print_enable_if_value<Array<VectorArr>>();  // Replace VectorArr with your actual type
	    
	    // b_d_h[0].print();
	}
    }
    TEST_CASE( "Test Assigment and copying of Arrays of Arrays and copy_to_cuda", "[Array]" ) {
	{
	    // Allocation and deallocation
	    // printf("Creating v1(10)\n");
	    VectorArr v1(10);
	    for (int i = 0; i < v1.size(); ++i) {
		v1[i] = Vector3(i+1);
	    }
 	    // printf("Creating v2(20)\n");
	    VectorArr v2(20);
	    for (int i = 0; i < v2.size(); ++i) {
		v2[i] = Vector3(10*i+1);
	    }
	    
	    // printf("Creating a(2)\n");
	    Array<VectorArr> a(3);
	    a[0] = v1;
	    a[1] = v2;
	    // a[1] = std::move(v2);

	    Array<VectorArr>* a_d = a.copy_to_cuda();
	    Array<VectorArr> a_d_h = a_d->copy_from_cuda(a_d);
	    
	    
	    REQUIRE( a[0][1] == a_d_h[0][1] );
	    // REQUIRE( a[0][5] == a_d_h[0][5] );

	    a_d->remove_from_cuda(a_d);
	}
    }

    TEST_CASE( "Test sending Arrays", "[Array]" ) {
	{
	    // Allocation and deallocation
	    // printf("Creating v1(10)\n");
	    Resource loc = Resource{Resource::GPU,0};
	    
	    VectorArr v1(10);
	    for (int i = 0; i < v1.size(); ++i) {
		v1[i] = Vector3(i+1);
	    }
	    VectorArr v2(20);
	    for (int i = 0; i < v2.size(); ++i) {
		v2[i] = Vector3(10*i+1);
	    }
	    
	    Array<VectorArr> a(3);
	    a[0] = v1;
	    a[1] = v2;
	    // a[1] = std::move(v2);

	    Proxy<Array<VectorArr>> a_d = send(loc, a);
	    // Array<VectorArr> a_d_h = a_d->copy_from_cuda(a_d);
	    
	    // REQUIRE( a[0][1] == a_d_h[0][1] );
	    // REQUIRE( a[0][5] == a_d_h[0][5] );
	    printf("Removing...\n");
	    a.remove_from_cuda(a_d.addr); // TODO: generalize

	}
    }
    
    TEST_CASE( "Test Assigment and copying of Arrays of Arrays of Arrays", "[Array]" ) {
	{
	    // Allocation and deallocation
	    // printf("Creating v1(10)\n");
	    VectorArr v1(10);
	    for (int i = 0; i < v1.size(); ++i) {
		v1[i] = Vector3(i+1);
	    }
 	    // printf("Creating v2(20)\n");
	    VectorArr v2(20);
	    for (int i = 0; i < v2.size(); ++i) {
		v2[i] = Vector3(10*i+1);
	    }
	    
	    // printf("Creating a(3)\n");
	    Array<VectorArr> a(3);
	    a[0] = v1;
	    a[1] = v2;

	    Array<Array<VectorArr>> b(3);
	    b[0] = a;
	    b[2] = std::move(a);

	    Array<Array<VectorArr>>* b_d = b.copy_to_cuda();
	    Array<Array<VectorArr>> b_d_h = b_d->copy_from_cuda(b_d);
	    	    
	    REQUIRE( b[0][0][0] == b_d_h[0][0][0] );
	    b_d->remove_from_cuda(b_d);
	}
    }

    //Benchmark showing that Array<Vector3> performs similarly to plain array for device copy, at least 
    /*
    TEST_CASE( "Test performance copying Array vs plain arrays", "[Array]" ) {
	size_t num = 100000;
	float* inp3 = allocate_plain_array_device<float>(3*num);
	float* out3 = allocate_plain_array_device<float>(3*num);

	float* inp4 = allocate_plain_array_device<float>(4*num);
	float* out4 = allocate_plain_array_device<float>(4*num);

	float4* inpF4 = allocate_plain_array_device<float4>(num);
	float4* outF4 = allocate_plain_array_device<float4>(num);

	Array<Vector3>* inpV = allocate_array_device<Vector3>(num);
	Array<Vector3>* outV = allocate_array_device<Vector3>(num);

	// call_copy_kernel(3*num, out3, inp3);
	// call_copy_kernel(4*num, out4, inp4);
	// call_copy_kernel(num, outV, inpV);
	BENCHMARK("Call 3x num float copy") {
	    call_copy_kernel(3*num, out3, inp3);
	};
	BENCHMARK("Call num Vector3 copy") {
	    call_copy_kernel(num, outV, inpV);
	};
	BENCHMARK("Call num float4 copy") {
	    call_copy_kernel(num, outF4, inpF4);
	};
	BENCHMARK("Call 3x num float copy (repeat)") {
	    call_copy_kernel(3*num, out3, inp3);
	};
	BENCHMARK("Call 4x num float copy") {
	    call_copy_kernel(4*num, out4, inp4);
	};
	BENCHMARK("Call num Vector3 copy (repeat)") {
	    call_copy_kernel(num, outV, inpV);
	};
	BENCHMARK("Call num float4 copy (repeat)") {
	    call_copy_kernel(num, outF4, inpF4);
	};
    }
    // */
}
