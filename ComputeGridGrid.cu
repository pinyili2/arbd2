#include "hip/hip_runtime.h"
// Included in RigidBodyController.cu
#include "ComputeGridGrid.cuh"
#include "RigidBodyGrid.h"
#include "CudaUtil.cuh"

//RBTODO handle periodic boundaries
//RBTODO: add __restrict__, benchmark (Q: how to restrict member data?)
__global__
void computeGridGridForce(const RigidBodyGrid* rho, const RigidBodyGrid* u,
													const Matrix3 basis_rho, const Matrix3 basis_u_inv,
													const Vector3 origin_rho_minus_origin_u,
													Vector3 * retForce, Vector3 * retTorque) {

	extern __shared__ Vector3 s[];
	Vector3 *force = s;
	Vector3 *torque = &s[NUMTHREADS];

  // RBTODO: http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops
	const int tid = threadIdx.x;
	const int r_id = blockIdx.x * blockDim.x + threadIdx.x;

	force[tid] = Vector3(0.0f);
	torque[tid] = Vector3(0.0f);
	if (r_id < rho->getSize()) { // skip threads with nothing to do
		// RBTODO: reduce registers used;
		//   commenting out interpolateForceD still uses ~40 registers
		//   -- the innocuous-looking fn below is responsible; consumes ~17 registers!
		Vector3 r_pos= rho->getPosition(r_id); /* i,j,k value of voxel */

		r_pos = basis_rho.transform( r_pos ) + origin_rho_minus_origin_u; /* real space */
		const Vector3 u_ijk_float = basis_u_inv.transform( r_pos );

		// RBTODO What about non-unit delta?
		/* Vector3 tmpf  = Vector3(0.0f); */
		/* float tmpe = 0.0f; */
		/* const ForceEnergy fe = ForceEnergy( tmpf, tmpe); */
		const ForceEnergy fe = u->interpolateForceDLinearly( u_ijk_float ); /* in coord frame of u */
		force[tid] = fe.f;

		const float r_val = rho->val[r_id]; /* maybe move to beginning of function?  */
		force[tid] = basis_u_inv.transpose().transform( r_val*force[tid] ); /* transform to lab frame, with correct scaling factor */

		// Calculate torque about origin_u in the lab frame
		torque[tid] = r_pos.cross(force[tid]);
	}

	// Reduce force and torques
	// http://www.cuvilib.com/Reduction.pdf
	// RBTODO optimize further, perhaps
	__syncthreads();
	for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
		if (tid < offset) {
			int oid = tid + offset;
			force[tid] = force[tid] + force[oid];
			torque[tid] = torque[tid] + torque[oid];
		}
		__syncthreads();
	}

	if (tid == 0) {
		retForce[blockIdx.x] = force[0];
		retTorque[blockIdx.x] = torque[0];
	}
}

__global__
void computePartGridForce(const Vector3* __restrict__ pos, Vector3* particleForce,
				const int num, const int* __restrict__ particleIds, 
				const RigidBodyGrid* __restrict__ u,
				const Matrix3 basis_u_inv, const Vector3 origin_u,
				Vector3* __restrict__ retForce, Vector3* __restrict__ retTorque) {

	extern __shared__ Vector3 s[];
	Vector3 *force = s;
	Vector3 *torque = &s[NUMTHREADS];
  
	const int tid = threadIdx.x;
	const int i = blockIdx.x * blockDim.x + threadIdx.x;

	force[tid] = Vector3(0.0f);
	torque[tid] = Vector3(0.0f);
	if (i < num) {
		const int& id = particleIds[i];
		Vector3 p = pos[id] - origin_u;
		// TODO: wrap to center of u
		const Vector3 u_ijk_float = basis_u_inv.transform( p );
		const ForceEnergy fe = u->interpolateForceDLinearly( u_ijk_float ); /* in coord frame of u */
		force[tid] = fe.f;
		force[tid] = basis_u_inv.transpose().transform( force[tid] ); /* transform to lab frame */
		atomicAdd( &particleForce[id], force[tid] ); // apply force to particle
		
		// Calculate torque about origin_u in the lab frame
		torque[tid] = p.cross(force[tid]);				// RBTODO: test if sign is correct!
	}

	// Reduce force and torques
	__syncthreads();
	for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
		if (tid < offset) {
			int oid = tid + offset;
			force[tid] = force[tid] + force[oid];
			torque[tid] = torque[tid] + torque[oid];
		}
		__syncthreads();
	}

	if (tid == 0) {
		retForce[blockIdx.x] = force[0];
		retTorque[blockIdx.x] = torque[0];
	}
}


__global__
void printRigidBodyGrid(const RigidBodyGrid* rho) {
  printf("Printing an RB of size %d\n",rho->size);
  for (int i=0; i < rho->size; i++)
	printf("  val[%d] = %f\n", i, rho->val[i]);
}
