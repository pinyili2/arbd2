#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include "RigidBody.h"
#include "RigidBodyType.h"
#include "RigidBodyController.h"
#include "Configuration.h"
#include "ComputeGridGrid.cuh"

#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>
#include <gsl/gsl_math.h>

#include "Debug.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), __FILE__, line);
      if (abort) exit(code);
   }
}


RigidBody::RigidBody(String name, const Configuration& cref, const RigidBodyType& tref, RigidBodyController* RBCref) 
    : name(name), c(&cref), t(&tref), RBC(RBCref), impulse_to_momentum(4.1867999435271e4) /*impulse_to_momentum(4.184e8f)*/ { init(); }
RigidBody::RigidBody(const RigidBody& rb)
    : name(rb.name), c(rb.c), t(rb.t), RBC(rb.RBC), impulse_to_momentum(4.1867999435271e4)/*impulse_to_momentum(4.184e8f)*/ { init(); }
void RigidBody::init() {
	// units "(kcal_mol/AA) * ns" "dalton AA/ns" * 4.184e+08	
	timestep = c->timestep;
	Temp = c->temperature * 0.0019872065f;
	// RBTODO: use temperature grids
	// tempgrid = c->temperatureGrid;
	position = t->initPos;

	// Orientation matrix that brings vector from the RB frame to the lab frame
	orientation = t->initRot;

        momentum = t->initMomentum;
        angularMomentum = t->initAngularMomentum;

	// Memory allocation for forces between particles and grids 
	const int& numGrids = t->numPotGrids;
	if (numGrids > 0) {
	    numParticles = new int[numGrids];
	    particles_d = new int*[numGrids];
	    particleForceStreams = new const hipStream_t*[numGrids];

	    for (int i = 0; i < numGrids; ++i) {
		numParticles[i] = -1;
		const int& n = t->numParticles[i];
		if (n > 0) {
		    // gpuErrchk(hipMalloc( &particles_d[i], 0.5*sizeof(int)*n )); // not sure why 0.5 was here; prolly bug
		    gpuErrchk(hipMalloc( &particles_d[i], sizeof(int)*n )); // TODO: dynamically allocate memory as needed
		}
	    }
	} else {
	    numParticles = NULL;
	}
}

GPUManager RigidBody::gpuman = GPUManager();

//Boltzmann distribution
void RigidBody::Boltzmann(unsigned long int seed)
{

    gsl_rng *gslcpp_rng = gsl_rng_alloc(gsl_rng_default);
    //std::srand(time(NULL));
    gsl_rng_set (gslcpp_rng, seed);

    double sigma[4] = { sqrt(t->mass*Temp) * 2.046167135,sqrt(t->inertia.x*Temp) * 2.046167135, sqrt(t->inertia.y*Temp) * 2.046167135, sqrt(t->inertia.z*Temp) * 2.046167135 };

    //Vector3 rando = getRandomGaussVector();
    momentum = Vector3(gsl_ran_gaussian(gslcpp_rng,sigma[0]),gsl_ran_gaussian(gslcpp_rng,sigma[0]), gsl_ran_gaussian(gslcpp_rng,sigma[0]));

    angularMomentum.x = gsl_ran_gaussian(gslcpp_rng,sigma[1]);
    angularMomentum.y = gsl_ran_gaussian(gslcpp_rng,sigma[2]);
    angularMomentum.z = gsl_ran_gaussian(gslcpp_rng,sigma[3]);
    printf("%f\n", Temp);
    printf("%f\n", Temperature());
    gsl_rng_free(gslcpp_rng);
}

RigidBody::~RigidBody() {
	const int& numGrids = t->numPotGrids;
	for (int i = 0; i < numGrids; ++i) {
		const int& n = t->numParticles[i];
		if (n > 0) {
			gpuErrchk(hipFree( particles_d[i] ));
		}
	}
	if (numParticles != NULL) {
		delete[] numParticles;
		delete[] particles_d;
		delete[] particleForceStreams;
	}
}

int RigidBody::appendNumParticleBlocks( std::vector<int>* blocks ) {
    int ret = 0;
    const int& numGrids = t->numPotGrids;
    for (int i = 0; i < numGrids; ++i) {
	numParticles[i] = -1;
	const int& n = t->numParticles[i];
	const int nb = (n/NUMTHREADS)+1; // max number of blocks
	if (n > 0) {
	    blocks->push_back(nb);
	    ret += nb;
	}
    }
    return ret;
}

void RigidBody::addForce(Force f) { 
	force += f; 
} 
void RigidBody::addTorque(Force torq) {
	torque += torq; 
}
void RigidBody::addEnergy(float e)
{
    energy += e;
}
void RigidBody::updateParticleList(Vector3* pos_d, BaseGrid* sys_d) {
	for (int i = 0; i < t->numPotGrids; ++i) {
		numParticles[i] = 0;
		int& tnp = t->numParticles[i];
		if (tnp > 0) {
		    int idx = t->potential_grid_idx[i];
			Vector3 gridCenter = t->RBC->grids[idx].getCenter();
			float cutoff = gridCenter.length();
			cutoff += t->RBC->grids[idx].getRadius();
			cutoff += c->pairlistDistance; 
		   
			int* tmp_d;
			gpuErrchk(hipMalloc( &tmp_d, sizeof(int) ));
			gpuErrchk(hipMemcpy( tmp_d, &numParticles[i], sizeof(int), hipMemcpyHostToDevice ));

			int nb = floor(tnp/NUMTHREADS) + 1;
#if __CUDA_ARCH__ >= 300
			createPartlist<<<nb,NUMTHREADS>>>(pos_d, tnp, t->particles_d[i],
							tmp_d, particles_d[i],
							gridCenter + position, cutoff*cutoff, sys_d);
#else
			createPartlist<<<nb,NUMTHREADS,NUMTHREADS/WARPSIZE>>>(pos_d, tnp, t->particles_d[i],
							tmp_d, particles_d[i],
							gridCenter + position, cutoff*cutoff, sys_d);
#endif			
			gpuErrchk(hipMemcpy(&numParticles[i], tmp_d, sizeof(int), hipMemcpyDeviceToHost ));
			gpuErrchk(hipFree( tmp_d ));
		}
	}
}

void RigidBody::callGridParticleForceKernel(Vector3* pos_d, Vector3* force_d, int s, float* energy, bool get_energy, int scheme, BaseGrid* sys, BaseGrid* sys_d, ForceEnergy* forcestorques_d, const std::vector<int>& forcestorques_offset, int& fto_idx) {
	// Apply the force and torque on the rigid body, and forces on particles
	
	// RBTODO: performance: consolidate CUDA stream management
	// loop over potential grids 
	for (int i = 0; i < t->numPotGrids; ++i) {
		if (numParticles[i] <= 0) continue;
		// const int nb = 500;
		/*
		  r: postion of particle in real space
		  B: grid Basis
		  o: grid origin
		  R: rigid body orientation
		  c: rigid body center

		  B': R.B 
		  c': R.o + c
		*/

		const hipStream_t& stream = gpuman.get_next_stream();
		particleForceStreams[i] = &stream;

		int idx = t->potential_grid_idx[i];

		Vector3 c =  getOrientation()*t->RBC->grids[idx].getOrigin() + getPosition();
		Matrix3 B = (getOrientation()*t->RBC->grids[idx].getBasis()).inverse();
		
		const int nb = (numParticles[i]/NUMTHREADS)+1;		
		computePartGridForce<<< nb, NUMTHREADS, NUMTHREADS*2*sizeof(ForceEnergy), stream >>>(
			pos_d, force_d, numParticles[i], particles_d[i],
			&t->RBC->grids_d[idx],
			B, c, forcestorques_d+forcestorques_offset[fto_idx++], energy, get_energy, scheme, sys_d);
	}
}

void RigidBody::applyGridParticleForces(BaseGrid* sys, ForceEnergy* forcestorques, const std::vector<int>& forcestorques_offset, int& fto_idx) {
	// loop over potential grids 
	for (int i = 0; i < t->numPotGrids; ++i) {
		if (numParticles[i] <= 0) continue;
		const int nb = (numParticles[i]/NUMTHREADS)+1;
		int idx = t->potential_grid_idx[i];
		Vector3 c =  getOrientation()*t->RBC->grids[idx].getOrigin() + getPosition();

		// Sum and apply forces and torques
		//Vector3 f = Vector3(0.0f);
		ForceEnergy f = ForceEnergy(0.f,0.f);
		Vector3 torq = Vector3(0.0f);
		for (int k = 0; k < nb; ++k) {
		    int j = forcestorques_offset[fto_idx]+2*k;
		    f = f + forcestorques[j];
		    torq = torq + forcestorques[j+1].f;
		}
		++fto_idx;
	        //why the force points are at the origin of the potential?	
		torq = -torq + (sys->wrapDiff(getPosition()-c)).cross( f.f ); 
		addForce( -f.f );
		addTorque( torq );
                addEnergy( f.e );
	}
}

	/*===========================================================================\
	| Following "Algorithm for rigid-body Brownian dynamics" Dan Gordon, Matthew |
	|   Hoyles, and Shin-Ho Chung                                                |
	|   http://langevin.anu.edu.au/publications/PhysRevE_80_066703.pdf           |
	|                                                                            |
	|                                                                            |
	| BUT: assume diagonal friction tensor and no Wiener process / stochastic    |
	|   calculus then this is just the same as for translation                   |
	|                                                                            |
	|   < T_i(t) T_i(t) > = 2 kT friction inertia                                |
	|                                                                            |
	|   friction / kt = Diff                                                     |
	\===========================================================================*/
void RigidBody::addLangevin(Vector3 w1, Vector3 w2) 
{
    Vector3 transForceCoeff = Vector3::element_sqrt( 2. * Temp * t->mass*t->transDamping / timestep );
    Vector3  rotTorqueCoeff = Vector3::element_sqrt( 2. * Temp * Vector3::element_mult( t->inertia,t->rotDamping) / timestep );

    Force f = Vector3::element_mult(transForceCoeff,w1) -
              Vector3::element_mult(t->transDamping, orientation.transpose()*momentum) * 10000;
    
    Force torq = Vector3::element_mult(rotTorqueCoeff,w2) -
                 Vector3::element_mult(t->rotDamping, angularMomentum) * 10000;

    f = orientation * f;
    torq = orientation * torq;

    addForce(f);
    addTorque(torq);
}

  /*==========================================================================\
	| from: Dullweber, Leimkuhler, Maclachlan. Symplectic splitting methods for |
	| rigid body molecular dynamics. JCP 107. (1997)                            |
	| http://jcp.aip.org/resource/1/jcpsa6/v107/i15/p5840_s1                    |
	\==========================================================================*/
void RigidBody::integrateDLM(int startFinishAll) 
{
    Vector3 trans; // = *p_trans;
    //Matrix3 rot = Matrix3(1); // = *p_rot;

    if ( isnan(force.x) || isnan(torque.x) ) 
    {   
        // NaN check
        printf("Rigid Body force or torque was NaN!\n");
        exit(-1);
    }

    if (startFinishAll == 0 || startFinishAll == 2) 
    {
        // propogate momenta by half step
        momentum += 0.5f * timestep * force * impulse_to_momentum;
        angularMomentum += 0.5f * timestep * (orientation.transpose()*torque) * impulse_to_momentum;
    } 
    else if (startFinishAll == 1)
    {
        position += timestep * momentum / t->mass * 1e4; // update CoM a full timestep
        // update orientations a full timestep
        Matrix3 R; // represents a rotation about a principle axis
        R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x * 1e4); // R1
        applyRotation(R);

        R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y * 1e4); // R2
        applyRotation(R);
                        
        R = Rz(    timestep * angularMomentum.z / t->inertia.z * 1e4); // R3
        applyRotation(R);
                        
        R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y * 1e4); // R4
        applyRotation(R);

        R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x * 1e4); // R5
        applyRotation(R);               
        // TODO make this periodic
        // printf("det: %.12f\n", orientation.det());
        orientation = orientation.normalized();
        // orientation = orientation/orientation.det();
        // printf("det2: %.12f\n", orientation.det());
        // orientation = orientation/orientation.det(); // TODO: see if this can be somehow eliminated (wasn't in original DLM algorithm...)
    }
}
/* Following:
Brownian Dynamics Simulation of Rigid Particles of Arbitrary Shape in External Fields
Miguel X. Fernandes, José García de la Torre
*/

//Chris original implementation for Brownian motion
void RigidBody::integrate(int startFinishAll)
{
    // UNITS
    // Temp: kcal_mol
    // t->transDamping: (kcal_mol/AA) / (amu AA/ns)
    // t->mass: amu
    // diffusion: AA**2/ns

    //if (startFinishAll == 1) return;

    //Matrix3 rot = Matrix3(1); // = *p_rot;

    if ( isnan(force.x) || isnan(torque.x) ) 
    {
        printf("Rigid Body force or torque was NaN!\n");
        exit(-1);
    }
    //float Temp = 1;
    Vector3 diffusion    = Temp / (t->transDamping*t->mass); // TODO: assign diffusion in config file, or elsewhere
    //Vector3 diffusion    = Temp / (t->transDamping*t->mass);
    Vector3 rotDiffusion = Temp / (Vector3::element_mult(t->rotDamping,t->inertia));

    Vector3 rando  = getRandomGaussVector();
    Vector3 offset = Vector3::element_mult( (diffusion / Temp), orientation.transpose() * force ) * timestep +
                     Vector3::element_mult( Vector3::element_sqrt( 2.0f * diffusion * timestep), rando) ;

    position += orientation*offset;

    rando = getRandomGaussVector();
    Vector3 rotationOffset = Vector3::element_mult( (rotDiffusion / Temp) , orientation.transpose() * torque * timestep) +
                             Vector3::element_mult( Vector3::element_sqrt( 2.0f * rotDiffusion * timestep), rando );

    // Consider whether a DLM-like decomposition of rotations is needed for time-reversibility
    orientation = orientation * (Rz(rotationOffset.z * 0.5) * Ry(rotationOffset.y * 0.5) * Rx(rotationOffset.x)
                              *  Ry(rotationOffset.y * 0.5) * Rz(rotationOffset.z * 0.5));
    //orientation = orientation * Rz(rotationOffset.z) * Ry(rotationOffset.y) * Rx(rotationOffset.x);
    orientation = orientation.normalized();
}
 
float RigidBody::Temperature()
{
    return (momentum.length2() / t->mass + 
            angularMomentum.x * angularMomentum.x / t->inertia.x + 
            angularMomentum.y * angularMomentum.y / t->inertia.y + 
            angularMomentum.z * angularMomentum.z / t->inertia.z) * 0.50 / Temp * (2.388458509e-1);
}

void RigidBody::applyRotation(const Matrix3& R) {
	angularMomentum = R * angularMomentum;
	// According to DLM, but rotations work the wrong way; I think DLM update is wrong
	// orientation = orientation * R.transpose(); 

	// This makes sense: apply a rotation in the body frame followed by a transformation from body to lab frame
	// Also works in statistical test
	// Consistent with www.archer.ac.uk/documentation/white-papers/lammps-elba/lammps-ecse.pdf
	orientation = orientation * R; 
        orientation.normalized();	
}

// Rotations about axes
// for very small angles 10^-8, cos^2+sin^2 != 1 
// concerned about the accumulation of errors in non-unitary transformations!
Matrix3 RigidBody::Rx(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		1.0f, 0.0f, 0.0f,
		0.0f,  cos, -sin,
		0.0f,  sin,  cos);
}
Matrix3 RigidBody::Ry(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		cos,  0.0f,  sin,
		0.0f, 1.0f, 0.0f,
		-sin, 0.0f,  cos);
}
Matrix3 RigidBody::Rz(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		cos,  -sin, 0.0f,
		sin,   cos, 0.0f,
		0.0f, 0.0f, 1.0f);
}
Matrix3 RigidBody::eulerToMatrix(const Vector3 e) {
	// convert euler angle input to rotation matrix
	// http://en.wikipedia.org/wiki/Rotation_formalisms_in_three_dimensions#Conversion_formulae_between_formalisms
	return Rz(e.z) * Ry(e.y) * Rx(e.x);
}
