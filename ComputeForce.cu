#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////
// Brownian dynamics base class
// Author: Jeff Comer <jcomer2@illinois.edu>

#include "ComputeForce.h"
#include "ComputeForce.cuh"
#include <hip/hip_runtime_api.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

hipEvent_t start, stop;

void runSort(int2 *d1, int *d2, float *key,
				int2 *scratch1, int  *scratch2, float *scratchKey,
				unsigned int count);

ComputeForce::ComputeForce(int num, const BrownianParticleType part[],
													 int numParts, const BaseGrid* g, float switchStart,
													 float switchLen, float electricConst,
													 int fullLongRange, int numBonds, int numTabBondFiles,
													 int numExcludes, int numAngles, int numTabAngleFiles,
													 int numDihedrals, int numTabDihedralFiles,
													 int numReplicas) :
		num(num), numParts(numParts), sys(g), switchStart(switchStart),
		switchLen(switchLen), electricConst(electricConst),
		cutoff2((switchLen + switchStart) * (switchLen + switchStart)),
		decomp(g->getBox(), g->getOrigin(), switchStart + switchLen, numReplicas),
		numBonds(numBonds), numTabBondFiles(numTabBondFiles),
		numExcludes(numExcludes), numAngles(numAngles),
		numTabAngleFiles(numTabAngleFiles), numDihedrals(numDihedrals),
		numTabDihedralFiles(numTabDihedralFiles), numReplicas(numReplicas) {
	// Allocate the parameter tables.
	decomp_d = NULL;

	tableEps = new float[numParts * numParts];
	tableRad6 = new float[numParts * numParts];
	tableAlpha = new float[numParts * numParts];

	const size_t tableSize = sizeof(float) * numParts * numParts;
	gpuErrchk(hipMalloc(&tableEps_d, tableSize));
	gpuErrchk(hipMalloc(&tableRad6_d, tableSize));
	gpuErrchk(hipMalloc(&tableAlpha_d, tableSize));
	gpuErrchk(hipMalloc(&sys_d, sizeof(BaseGrid)));
	gpuErrchk(hipMemcpyAsync(sys_d, sys, sizeof(BaseGrid), hipMemcpyHostToDevice));
	// Form the parameter tables.
	makeTables(part);

	gpuErrchk(hipMemcpyAsync(tableAlpha_d, tableAlpha, tableSize,
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableEps_d, tableEps, tableSize,
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableRad6_d, tableRad6, tableSize,
			hipMemcpyHostToDevice));

	// Create the potential table
	tablePot = new TabulatedPotential*[numParts * numParts];
	tablePot_addr = new TabulatedPotential*[numParts * numParts];
	for (int i = 0; i < numParts*numParts; i++) {
		tablePot_addr[i] = NULL;
		tablePot[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tablePot_d, sizeof(TabulatedPotential*) * numParts * numParts));

	// Create the bond table
	tableBond = new TabulatedPotential*[numTabBondFiles];
	tableBond_addr = new TabulatedPotential*[numTabBondFiles];
	for (int i = 0; i < numTabBondFiles; i++) {
		tableBond_addr[i] = NULL;
		tableBond[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableBond_d, sizeof(TabulatedPotential*) * numTabBondFiles));

	// Create the angle table
	tableAngle = new TabulatedAnglePotential*[numTabAngleFiles];
	tableAngle_addr = new TabulatedAnglePotential*[numTabAngleFiles];
	for (int i = 0; i < numTabAngleFiles; i++) {
		tableAngle_addr[i] = NULL;
		tableAngle[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableAngle_d, sizeof(TabulatedAnglePotential*) * numTabAngleFiles));

	// Create the dihedral table
	tableDihedral = new TabulatedDihedralPotential*[numTabDihedralFiles];
	tableDihedral_addr = new TabulatedDihedralPotential*[numTabDihedralFiles];
	for (int i = 0; i < numTabDihedralFiles; i++) {
		tableDihedral_addr[i] = NULL;
		tableDihedral[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableDihedral_d, sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles));

	//Calculate the number of blocks the grid should contain
	gridSize =  num / NUM_THREADS + 1;

	// Create and allocate the energy arrays
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * num));
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

ComputeForce::~ComputeForce() {
	delete[] tableEps;
	delete[] tableRad6;
	delete[] tableAlpha;
	hipFree(tableEps_d);
	hipFree(tableAlpha_d);
	hipFree(tableRad6_d);

	for (int j = 0; j < numParts * numParts; ++j)
		delete tablePot[j];
	delete[] tablePot;
	delete[] tablePot_addr;

	for (int j = 0; j < numTabBondFiles; ++j)
		delete tableBond[j];
	delete[] tableBond;
	delete[] tableBond_addr;
	gpuErrchk(hipFree(tableBond_d));

	for (int j = 0; j < numTabAngleFiles; ++j)
		if (tableAngle[j] != NULL)
			delete tableAngle[j];
	delete[] tableAngle;
	delete[] tableAngle_addr;

	if(type_d != NULL)
	{
		gpuErrchk(hipFree(tableAngle_d));

		gpuErrchk(hipFree(energies_d));

		gpuErrchk(hipFree(sys_d));

		gpuErrchk( hipFree(pos_d) );
		gpuErrchk( hipFree(forceInternal_d) );
		gpuErrchk( hipFree(type_d) );
		gpuErrchk( hipFree(bonds_d) );
		gpuErrchk( hipFree(bondMap_d) );
		gpuErrchk( hipFree(excludes_d) );
		gpuErrchk( hipFree(excludeMap_d) );
		gpuErrchk( hipFree(angles_d) );
		gpuErrchk( hipFree(dihedrals_d) );
		gpuErrchk( hipFree(bondList_d) );
	}
}

void ComputeForce::updateNumber(int newNum) {
	if (newNum == num or newNum < 0) return;

	// Set the new number.
	num = newNum;

	// Reallocate the neighbor list.
	//delete[] neigh;
	//neigh = new IndexList[num];
	decompose();

	printf("updateNumber() called\n");
	// Reallocate CUDA arrays

	// Recalculate the number of blocks in the grid
	gridSize = 0;
	while ((int)sqrt(NUM_THREADS) * gridSize < num)
		++gridSize;

	gpuErrchk(hipFree(energies_d));
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * gridSize));
}

void ComputeForce::makeTables(const BrownianParticleType part[]) {
	for (int i = 0; i < numParts; ++i) {
		const BrownianParticleType& pi = part[i];
		for (int j = 0; j < numParts; ++j) {
			const BrownianParticleType& pj = part[j];
			int ind = i * numParts + j;
			tableEps[ind] = sqrtf(pi.eps * pj.eps);
			float r = pi.radius + pj.radius;
			tableRad6[ind] = r * r * r * r * r * r;
			tableAlpha[ind] = electricConst * pi.charge * pj.charge;
		}
	}
}

bool ComputeForce::addTabulatedPotential(String fileName, int type0, int type1) {
	if (type0 < 0 or type0 >= numParts) return false;
	if (type1 < 0 or type1 >= numParts) return false;

	int ind = type0 + type1 * numParts;
	int ind1 = type1 + type0 * numParts;

	// If an entry already exists for this particle type, delete it
	if (tablePot[ind] != NULL) {
		delete tablePot[ind];
		gpuErrchk(hipFree(tablePot_addr[ind]));
		tablePot[ind] = NULL;
		tablePot_addr[ind] = NULL;
	}
	if (tablePot[ind1] != NULL) {
		gpuErrchk(hipFree(tablePot_addr[ind1]));
		delete tablePot[ind1];
		tablePot[ind1] = NULL;
		tablePot_addr[ind1] = NULL;
	}

	tablePot[ind] = new TabulatedPotential(fileName);
	tablePot[ind]->truncate(switchStart, sqrtf(cutoff2), 0.0f);
	tablePot[ind1] = new TabulatedPotential(*(tablePot[ind]));

	TabulatedPotential* t = new TabulatedPotential(*tablePot[ind]);

	// Copy tablePot[ind] to the device
	float *v0, *v1, *v2, *v3;
	size_t sz_n = sizeof(float) * tablePot[ind]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tablePot[ind]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tablePot[ind]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tablePot[ind]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tablePot[ind]->v3, sz_n, hipMemcpyHostToDevice));
	t->v0 = v0; t->v1 = v1;
	t->v2 = v2; t->v3 = v3;
	gpuErrchk(hipMalloc(&tablePot_addr[ind], sizeof(TabulatedPotential)));
	gpuErrchk(hipMemcpy(tablePot_addr[ind], t, sizeof(TabulatedPotential), hipMemcpyHostToDevice));
	t->v0 = NULL; t->v1 = NULL;
	t->v2 = NULL; t->v3 = NULL;
	delete t;
	/** Same thing for ind1 **/
	t = new TabulatedPotential(*tablePot[ind1]);
	sz_n = sizeof(float) * tablePot[ind1]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tablePot[ind1]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tablePot[ind1]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tablePot[ind1]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tablePot[ind1]->v3, sz_n, hipMemcpyHostToDevice));
	t->v0 = v0; t->v1 = v1;
	t->v2 = v2; t->v3 = v3;
	gpuErrchk(hipMalloc(&tablePot_addr[ind1], sizeof(TabulatedPotential)));
	gpuErrchk(hipMemcpy(tablePot_addr[ind1], t, sizeof(TabulatedPotential), hipMemcpyHostToDevice));
	t->v0 = NULL; t->v1 = NULL;
	t->v2 = NULL; t->v3 = NULL;
	delete t;
	gpuErrchk(hipMemcpy(tablePot_d, tablePot_addr,
			sizeof(TabulatedPotential*) * numParts * numParts, hipMemcpyHostToDevice));

	return true;
}

bool ComputeForce::addBondPotential(String fileName, int ind, Bond bonds[])
{
	if (tableBond[ind] != NULL) {
		delete tableBond[ind];
		gpuErrchk(hipFree(tableBond_addr[ind]));
		tableBond[ind] = NULL;
		tableBond_addr[ind] = NULL;
	}
	tableBond[ind] = new TabulatedPotential(fileName);
	tableBond[ind]->truncate(switchStart, sqrtf(cutoff2), 0.0f);

	for (int i = 0; i < numBonds; ++i)
		if (bonds[i].fileName == fileName)
			bonds[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(bonds_d, bonds, sizeof(Bond) * numBonds, hipMemcpyHostToDevice));

	// Copy tableBond[ind] to the device
	float *v0, *v1, *v2, *v3;
	size_t sz_n = sizeof(float) * tableBond[ind]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tableBond[ind]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tableBond[ind]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tableBond[ind]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tableBond[ind]->v3, sz_n, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&tableBond_addr[ind], sizeof(TabulatedPotential)));
	TabulatedPotential t = TabulatedPotential(*tableBond[ind]);
	t.v0 = v0; t.v1 = v1;
	t.v2 = v2; t.v3 = v3;
	gpuErrchk(hipMemcpyAsync(tableBond_addr[ind], &t,
			sizeof(TabulatedPotential), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(tableBond_d, tableBond_addr,
			sizeof(TabulatedPotential*) * numTabBondFiles, hipMemcpyHostToDevice));
	t.v0 = NULL; t.v1 = NULL;
	t.v2 = NULL; t.v3 = NULL;
	return true;
}

bool ComputeForce::addAnglePotential(String fileName, int ind, Angle* angles) {
	if (tableAngle[ind] != NULL) {
		delete tableAngle[ind];
		gpuErrchk(hipFree(tableAngle_addr[ind]));
		tableAngle[ind] = NULL;
		tableAngle_addr[ind] = NULL;
	}

	tableAngle[ind] = new TabulatedAnglePotential(fileName);
	TabulatedAnglePotential *t = new TabulatedAnglePotential(*tableAngle[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableAngle[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableAngle[ind]->pot, sizeof(float) * size, hipMemcpyHostToDevice));
	t->pot = pot;
	gpuErrchk(hipMalloc(&tableAngle_addr[ind], sizeof(TabulatedAnglePotential)));
	gpuErrchk(hipMemcpy(tableAngle_addr[ind], t, sizeof(TabulatedAnglePotential), hipMemcpyHostToDevice));
	t->pot = NULL;
	delete t;

	gpuErrchk(hipMemcpyAsync(tableAngle_d, tableAngle_addr,
			sizeof(TabulatedAnglePotential*) * numTabAngleFiles, hipMemcpyHostToDevice));

	for (int i = 0; i < numAngles; i++)
		if (angles[i].fileName == fileName)
			angles[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpy(angles_d, angles, sizeof(Angle) * numAngles,
			hipMemcpyHostToDevice));
	return true;
}

bool ComputeForce::addDihedralPotential(String fileName, int ind, Dihedral dihedrals[])
{
	for (int i = 0; i < numDihedrals; i++)
		if (dihedrals[i].fileName == fileName)
			dihedrals[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(dihedrals_d, dihedrals, sizeof(Dihedral) * numDihedrals,
			hipMemcpyHostToDevice));

	if (tableDihedral[ind] != NULL) {
		delete tableDihedral[ind];
		gpuErrchk(hipFree(tableDihedral_addr[ind]));
		tableDihedral[ind] = NULL;
		tableDihedral_addr[ind] = NULL;
	}

	tableDihedral[ind] = new TabulatedDihedralPotential(fileName);
	TabulatedDihedralPotential t = TabulatedDihedralPotential(*tableDihedral[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableDihedral[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableDihedral[ind]->pot,
			sizeof(float) * size, hipMemcpyHostToDevice));
	t.pot = pot;

	gpuErrchk(hipMalloc(&tableDihedral_addr[ind], sizeof(TabulatedDihedralPotential)));
	gpuErrchk(hipMemcpyAsync(tableDihedral_addr[ind], &t,
			sizeof(TabulatedDihedralPotential), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(tableDihedral_d, tableDihedral_addr,
			sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles, hipMemcpyHostToDevice));
	t.pot = NULL;
	return true;
}

void ComputeForce::decompose() {
	gpuErrchk( hipProfilerStart() );
	// Reset the cell decomposition.
	bool newDecomp = false;
	if (decomp_d)
		hipFree(decomp_d);
	else
		newDecomp = true;
		
	decomp.decompose_d(pos_d, num);
	decomp_d = decomp.copyToCUDA();

	// Update pairlists using cell decomposition (not sure this is really needed or good) 
	//RBTODO updatePairlists<<< nBlocks, NUM_THREADS >>>(pos_d, num, numReplicas, sys_d, decomp_d);	


	/* size_t free, total; */
	/* { */
	/* 	hipMemGetInfo(&free,&total); */
	/* 	printf("Free memory: %zu / %zu\n", free, total); */
	/* } */
	
	// initializePairlistArrays
	int nCells = decomp.nCells.x * decomp.nCells.y * decomp.nCells.z;
	int blocksPerCell = 10;
	if (newDecomp) {
		// RBTODO: free memory elsewhere
		// allocate device data
		// initializePairlistArrays<<< 1, 32 >>>(10*nCells*blocksPerCell);
		const int maxPairs = 1<<25;
		gpuErrchk(hipMalloc(&numPairs_d,       sizeof(int)));

		gpuErrchk(hipMalloc(&pairLists_d,      sizeof(int2)*maxPairs));
		gpuErrchk(hipMalloc(&pairTabPotType_d, sizeof(int)*maxPairs));

		gpuErrchk(hipDeviceSynchronize());
	}

	
	/* hipMemGetInfo(&free,&total); */
	/* printf("Free memory: %zu / %zu\n", free, total); */
	
	const int NUMTHREADS = 128;
	//const size_t nBlocks = (num * numReplicas) / NUM_THREADS + 1;
	const size_t nBlocks = nCells*blocksPerCell;

	/* clearPairlists<<< 1, 32 >>>(pos, num, numReplicas, sys_d, decomp_d); */
	/* gpuErrchk(hipDeviceSynchronize()); */
	/* pairlistTest<<< nBlocks, NUMTHREADS >>>(pos, num, numReplicas, */
	/* 																					 sys_d, decomp_d, nCells, blocksPerCell, */
	/* 																					 numPairs_d, pairListListI_d, pairListListJ_d); */
	/* gpuErrchk(hipDeviceSynchronize());	 */

	int tmp = 0;
	gpuErrchk(hipMemcpyAsync(numPairs_d, &tmp,	sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());
	
	float pairlistdist2 = (sqrt(cutoff2) + 2.0f);
	pairlistdist2 = pairlistdist2*pairlistdist2;
	
	createPairlists<<< 2048, 64 >>>(pos_d, num, numReplicas, sys_d, decomp_d, nCells, numPairs_d, pairLists_d, numParts, type_d, pairTabPotType_d, pairlistdist2);
	/* createPairlistsOld<<< nBlocks, NUMTHREADS >>>(pos, num, numReplicas, */
	/* 																					 sys_d, decomp_d, nCells, blocksPerCell, */
	/* 																					 numPairs_d, pairLists_d, */
	/* 																					 numParts, type, pairTabPotType_d, pairlistdist2); */

	gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
	// if (false)
	{ // sort pairlist
		int numPairs;
		gpuErrchk(hipMemcpyAsync( &numPairs, numPairs_d, sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
		printf("here, %d pairs\n", numPairs);
		/* runSort(pairLists_d, pairTabPotType_d, pairDists_d, */
		/* 				pairLists_s, pairTabPotType_s, pairDists_s, */
		/* 				numPairs); */
		/* printf("done\n"); */
		
		/* // RBTODO: sort pairListInd as well!!! (i.e. roll your own sort!) */
		/* // thrust::sort_by_key( pairDists_d, pairDists_d+numPairs_d, pairLists_d ); */
		/* // thrust::sort_by_key( pairDists_d, pairDists_d+numPairs_d, pairLists_d ); */
		/* gpuErrchk(hipDeviceSynchronize()); /\* RBTODO: sync needed here? *\/ */
	}
}

IndexList ComputeForce::decompDim() const {
	IndexList ret;
	ret.add(decomp.getNx());
	ret.add(decomp.getNy());
	ret.add(decomp.getNz());
	return ret;
}

CellDecomposition ComputeForce::getDecomp() { return decomp; }

float ComputeForce::decompCutoff() { return decomp.getCutoff(); }

// TODO: Fix this
int* ComputeForce::neighborhood(Vector3 r) {
	// return decomp.getCell(r)->getNeighbors();
	return NULL;
}

float ComputeForce::computeFull(bool get_energy) {
	float energy = 0.0f;
	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeFullKernel<<< numBlocks, numThreads >>>(forceInternal_d, pos_d, type_d, tableAlpha_d,
		tableEps_d, tableRad6_d, num, numParts, sys_d, energies_d, gridSize,
		numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeSoftcoreFull(bool get_energy) {
	float energy = 0.0f;
	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeSoftcoreFullKernel<<<numBlocks, numThreads>>>(forceInternal_d, pos_d, type_d,
			tableEps_d, tableRad6_d, num, numParts, sys_d, energies_d, gridSize,
			numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		hipDeviceSynchronize();
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeElecFull(bool get_energy) {
	float energy = 0.0f;

	gridSize = num/NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeElecFullKernel<<<numBlocks, numThreads>>>(forceInternal_d, pos_d, type_d,
			tableAlpha_d, num, numParts, sys_d, energies_d, gridSize, numReplicas,
			get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}


float ComputeForce::compute(bool get_energy) {
	float energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeKernel<<<numBlocks, numThreads>>>(forceInternal_d, pos_d, type_d,
			tableAlpha_d, tableEps_d, tableRad6_d, num, numParts, sys_d,
			decomp_d, energies_d, switchStart, switchLen, gridSize, numReplicas,
			get_energy);

	gpuErrchk(hipDeviceSynchronize());
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

//MLog: added Bond* bondList to the list of passed in variables.
/*float ComputeForce::computeTabulated(Vector3* force, Vector3* pos, int* type,
		Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap,
		Angle* angles, Dihedral* dihedrals, bool get_energy, Bond* bondList) {*/
float ComputeForce::computeTabulated(bool get_energy) {
	float energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);
	
	// Call the kernel to calculate the forces
	// int nb = (decomp.nCells.x * decomp.nCells.y * decomp.nCells.z);
	// int nb = (1+(decomp.nCells.x * decomp.nCells.y * decomp.nCells.z)) * 75; /* RBTODO: number of pairLists */
	const int nb = 800;
	// printf("ComputeTabulated\n");
	gpuErrchk(hipDeviceSynchronize());

	// RBTODO: get_energy
	//if (get_energy)
	if (false) 
	{
		clearEnergies<<< nb, numThreads >>>(energies_d,num);
		gpuErrchk(hipDeviceSynchronize());
		computeTabulatedEnergyKernel<<< nb, numThreads >>>(forceInternal_d, pos_d, sys_d,
						cutoff2, numPairs_d, pairLists_d, pairTabPotType_d, tablePot_d, energies_d);
	}
	
	else
	{
		computeTabulatedKernel<<< nb, numThreads >>>(forceInternal_d, pos_d, sys_d,
						cutoff2, numPairs_d, pairLists_d, pairTabPotType_d, tablePot_d);
	}
	/* printPairForceCounter<<<1,32>>>(); */

	//Mlog: the commented function doesn't use bondList, uncomment for testing.
	//if(bondMap_d != NULL && tableBond_d != NULL)
	if(bondList_d != NULL && tableBond_d != NULL)

	{
	    //computeTabulatedBonds <<<numBlocks, numThreads>>> ( force, pos, num, numParts, sys_d, bonds, bondMap_d, numBonds, numReplicas, energies_d, get_energy, tableBond_d);
	computeTabulatedBonds <<<nb, numThreads>>> ( forceInternal_d, pos_d, sys_d, numReplicas*numBonds/2, bondList_d, tableBond_d);
	}

	if (angleList_d != NULL && tableAngle_d != NULL)
		computeTabulatedAngles<<<nb, numThreads>>>(forceInternal_d, pos_d, sys_d, numAngles*numReplicas, angleList_d, tableAngle_d);

	if (dihedralList_d != NULL && tableDihedral_d != NULL)
		computeTabulatedDihedrals<<<nb, numThreads>>>(forceInternal_d, pos_d, sys_d, numDihedrals*numReplicas, dihedralList_d, dihedralPotList_d, tableDihedral_d);


	// Calculate the energy based on the array created by the kernel
	// TODO: return energy
	// if (get_energy) {
	// 	gpuErrchk(hipDeviceSynchronize());
	// 	thrust::device_ptr<float> en_d(energies_d);
	// 	energy = thrust::reduce(en_d, en_d + num);
	// }

	return energy;
}

float ComputeForce::computeTabulatedFull(bool get_energy) {
	energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeTabulatedFullKernel<<< numBlocks, numThreads >>>(forceInternal_d, pos_d, type_d,	tablePot_d, tableBond_d, num, numParts, sys_d, bonds_d, bondMap_d, numBonds, excludes_d, excludeMap_d, numExcludes, energies_d, gridSize, numReplicas, get_energy, angles_d);
	gpuErrchk(hipDeviceSynchronize());

	computeAngles<<< numBlocks, numThreads >>>(forceInternal_d, pos_d, angles_d, tableAngle_d,
																						 numAngles, num, sys_d, energies_d,
																						 get_energy);
	gpuErrchk(hipDeviceSynchronize());
	computeDihedrals<<< numBlocks, numThreads >>>(forceInternal_d, pos_d, dihedrals_d,
																							  tableDihedral_d, numDihedrals,
																								num, sys_d, energies_d,
																								get_energy);
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

void ComputeForce::copyToCUDA(Vector3* forceInternal, Vector3* pos)
{
	const size_t tot_num = num * numReplicas;

	gpuErrchk(hipMalloc(&pos_d, sizeof(Vector3) * tot_num));
	gpuErrchk(hipMemcpyAsync(pos_d, pos, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&forceInternal_d, sizeof(Vector3) * num * numReplicas));
	gpuErrchk(hipMemcpyAsync(forceInternal_d, forceInternal, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));

	gpuErrchk(hipDeviceSynchronize());
}

void ComputeForce::copyToCUDA(int simNum, int *type, Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap, Angle* angles, Dihedral* dihedrals)
{
	// type_d
	gpuErrchk(hipMalloc(&type_d, sizeof(int) * num * simNum));
	gpuErrchk(hipMemcpyAsync(type_d, type, sizeof(int) * num * simNum, hipMemcpyHostToDevice));
	
	if (numBonds > 0)
	{
		// bonds_d
		gpuErrchk(hipMalloc(&bonds_d, sizeof(Bond) * numBonds));
		gpuErrchk(hipMemcpyAsync(bonds_d, bonds, sizeof(Bond) * numBonds, hipMemcpyHostToDevice));
		
		// bondMap_d
		gpuErrchk(hipMalloc(&bondMap_d, sizeof(int2) * num));
		gpuErrchk(hipMemcpyAsync(bondMap_d, bondMap, sizeof(int2) * num, hipMemcpyHostToDevice));
	}

	if (numExcludes > 0) {
		// excludes_d
		gpuErrchk(hipMalloc(&excludes_d, sizeof(Exclude) * numExcludes));
		gpuErrchk(hipMemcpyAsync(excludes_d, excludes, sizeof(Exclude) * numExcludes,
				hipMemcpyHostToDevice));
		
		// excludeMap_d
		gpuErrchk(hipMalloc(&excludeMap_d, sizeof(int2) * num));
		gpuErrchk(hipMemcpyAsync(excludeMap_d, excludeMap, sizeof(int2) * num,
				hipMemcpyHostToDevice));
	}

	if (numAngles > 0) {
		// angles_d
		gpuErrchk(hipMalloc(&angles_d, sizeof(Angle) * numAngles));
		gpuErrchk(hipMemcpyAsync(angles_d, angles, sizeof(Angle) * numAngles,
				hipMemcpyHostToDevice));
	}

	if (numDihedrals > 0) {
		// dihedrals_d
		gpuErrchk(hipMalloc(&dihedrals_d, sizeof(Dihedral) * numDihedrals));
		gpuErrchk(hipMemcpyAsync(dihedrals_d, dihedrals,
												 		  sizeof(Dihedral) * numDihedrals,
														 	hipMemcpyHostToDevice));
	}

	gpuErrchk(hipDeviceSynchronize());
}

// void ComputeForce::createBondList(int3 *bondList)
// {
// 	size_t size = (numBonds / 2) * numReplicas * sizeof(int3);
// 	gpuErrchk( hipMalloc( &bondList_d, size ) );
// 	gpuErrchk( hipMemcpyAsync( bondList_d, bondList, size, hipMemcpyHostToDevice) );

// 	for(int i = 0 ; i < (numBonds / 2) * numReplicas ; i++)
// 	{
// 		cout << "Displaying: bondList_d["<< i <<"].x = " << bondList[i].x << ".\n"
// 			<< "Displaying: bondList_d["<< i <<"].y = " << bondList[i].y << ".\n"
// 			<< "Displaying: bondList_d["<< i <<"].z = " << bondList[i].z << ".\n";

// 	}
// }

void ComputeForce::copyBondedListsToGPU(int3 *bondList, int4 *angleList, int4 *dihedralList, int *dihedralPotList) {

	
	size_t size;

	if (numBonds > 0) {
	size = (numBonds / 2) * numReplicas * sizeof(int3);
	gpuErrchk( hipMalloc( &bondList_d, size ) );
	gpuErrchk( hipMemcpyAsync( bondList_d, bondList, size, hipMemcpyHostToDevice) );
	}
	
	if (numAngles > 0) {
    size = numAngles * numReplicas * sizeof(int4);
    gpuErrchk( hipMalloc( &angleList_d, size ) );
    gpuErrchk( hipMemcpyAsync( angleList_d, angleList, size, hipMemcpyHostToDevice) );
	}
	
	if (numDihedrals > 0) {
    size = numDihedrals * numReplicas * sizeof(int4);
    gpuErrchk( hipMalloc( &dihedralList_d, size ) );
    gpuErrchk( hipMemcpyAsync( dihedralList_d, dihedralList, size, hipMemcpyHostToDevice) );

    size = numDihedrals * numReplicas * sizeof(int);
    gpuErrchk( hipMalloc( &dihedralPotList_d, size ) );
    gpuErrchk( hipMemcpyAsync( dihedralPotList_d, dihedralPotList, size, hipMemcpyHostToDevice) );
	}
}
