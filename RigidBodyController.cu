/* #ifndef MIN_DEBUG_LEVEL */
/* #define MIN_DEBUG_LEVEL 5 */
/* #endif */
/* #define DEBUGM */
/* #include "Debug.h" */

/* #include "RigidBody.h" */
#include "RigidBodyController.h"
#include "Configuration.h"
#include "RigidBodyType.h"
#include "RigidBodyGrid.h"
#include "ComputeGridGrid.cuh"

#include <hip/hip_runtime_api.h>

// #include <vector>
#include "Debug.h"

#include "RandomCPU.h"							/* RBTODO: fix this? */

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, String file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), __FILE__, line);
      if (abort) exit(code);
   }
}

/* #include <hip/hip_runtime.h> */
/* #include <hip/hip_runtime.h> */
/* #include <hiprand/hiprand_kernel.h> */

RigidBodyController::RigidBodyController(const Configuration& c, const char* outArg) :
	conf(c), outArg(outArg) {

	if (conf.numRigidTypes > 0) {
		copyGridsToDevice();
	}

	int numRB = 0;
	// grow list of rbs
	for (int i = 0; i < conf.numRigidTypes; i++) {			
		numRB += conf.rigidBody[i].num;
		std::vector<RigidBody> tmp;
		// RBTODO: change conf.rigidBody to conf.rigidBodyType
		const int jmax = conf.rigidBody[i].num;
		for (int j = 0; j < jmax; j++) {
			String name = conf.rigidBody[i].name;
			if (jmax > 1) {
				char tmp[128];
				snprintf(tmp, 128, "#%d", j);
				name.add( tmp );
			}
			RigidBody r(name, conf, conf.rigidBody[i]);
			tmp.push_back( r );
	}
		rigidBodyByType.push_back(tmp);
}

	random = new RandomCPU(conf.seed + 1); /* +1 to avoid using same seed as RandomCUDA */
	
	initializeForcePairs();
	initializeParticleLists();
}
RigidBodyController::~RigidBodyController() {
	for (int i = 0; i < rigidBodyByType.size(); i++)
		rigidBodyByType[i].clear();
	rigidBodyByType.clear();
	delete random;
}

void RigidBodyController::initializeForcePairs() {
	// Loop over all pairs of rigid body types
	//   the references here make the code more readable, but they may incur a performance loss
	RigidBodyForcePair::createStreams();
	printf("Initializing force pairs\n");
	for (int ti = 0; ti < conf.numRigidTypes; ti++) {
		RigidBodyType& t1 = conf.rigidBody[ti];
		for (int tj = ti; tj < conf.numRigidTypes; tj++) {
			RigidBodyType& t2 = conf.rigidBody[tj];


			const std::vector<String>& keys1 = t1.densityGridKeys; 
			const std::vector<String>& keys2 = t2.potentialGridKeys;

			printf("  Working on type pair ");
			t1.name.printInline(); printf(":"); t2.name.print();
			
			// Loop over all pairs of grid keys (e.g. "Elec")
			std::vector<int> gridKeyId1;
			std::vector<int> gridKeyId2;
			
			printf("  Grid keys %d:%d\n",keys1.size(),keys2.size());

			bool paired = false;
			for(int k1 = 0; k1 < keys1.size(); k1++) {
				for(int k2 = 0; k2 < keys2.size(); k2++) {
					printf("    checking grid keys ");
					keys1[k1].printInline(); printf(":"); keys2[k2].print();
					
					if ( keys1[k1] == keys2[k2] ) {
						gridKeyId1.push_back(k1);
						gridKeyId2.push_back(k2);
						paired = true;
					}
				}
			}
			
			if (paired) {
				// found matching keys => calculate force between all grid pairs
				std::vector<RigidBody>& rbs1 = rigidBodyByType[ti];
				std::vector<RigidBody>& rbs2 = rigidBodyByType[tj];

				// Loop over rigid bodies of these types
				for (int i = 0; i < rbs1.size(); i++) {
					for (int j = (ti==tj ? i+1 : 0); j < rbs2.size(); j++) {
						RigidBody* rb1 = &(rbs1[i]);
						RigidBody* rb2 = &(rbs2[j]);

						printf("    pushing RB force pair for %d:%d\n",i,j);
						RigidBodyForcePair fp = RigidBodyForcePair(&(t1),&(t2),rb1,rb2,gridKeyId1,gridKeyId2, false);
						gpuErrchk(hipDeviceSynchronize()); /* RBTODO: this should be extraneous */
						forcePairs.push_back( fp ); 
						printf("    done pushing RB force pair for %d:%d\n",i,j);
					}
				}
			}
		}
	}

	// add Pmfs (not a true pairwise RB interaction; hacky implementation)
	for (int ti = 0; ti < conf.numRigidTypes; ti++) {
		RigidBodyType& t1 = conf.rigidBody[ti];

		const std::vector<String>& keys1 = t1.densityGridKeys; 
		const std::vector<String>& keys2 = t1.pmfKeys;
		std::vector<int> gridKeyId1;
		std::vector<int> gridKeyId2;
		
		// Loop over all pairs of grid keys (e.g. "Elec")
		bool paired = false;
		for(int k1 = 0; k1 < keys1.size(); k1++) {
			for(int k2 = 0; k2 < keys2.size(); k2++) {
				if ( keys1[k1] == keys2[k2] ) {
					gridKeyId1.push_back(k1);
					gridKeyId2.push_back(k2);
					paired = true;
				}
			}
		}	
		if (paired) {
			// found matching keys => calculate force between all grid pairs
			std::vector<RigidBody>& rbs1 = rigidBodyByType[ti];
			
			// Loop over rigid bodies of these types
			for (int i = 0; i < rbs1.size(); i++) {
					RigidBody* rb1 = &(rbs1[i]);
					RigidBodyForcePair fp = RigidBodyForcePair(&(t1),&(t1),rb1,rb1,gridKeyId1,gridKeyId2, true);
					gpuErrchk(hipDeviceSynchronize()); /* RBTODO: this should be extraneous */
					forcePairs.push_back( fp ); 
			}
		}
	}

	// Initialize device data for RB force pairs after std::vector is done growing
	for (int i = 0; i < forcePairs.size(); i++)
		forcePairs[i].initialize();
			
}

void RigidBodyController::initializeParticleLists() {
	// Populate RigidBodyType.particles
	
	// TODO: ensure no duplicates in conf.partRigidBodyGrid[i]
	
    // Allocate RB type's numParticles array
	for (int rb = 0; rb < conf.numRigidTypes; ++rb) {
		RigidBodyType& t = conf.rigidBody[rb];
		t.numParticles = new int[t.numPotGrids];
		for (int i = 0; i < t.numPotGrids; ++i) t.numParticles[i] = 0;
	}		

	// Count the number of particles; Loop over particle types
	for (int i = 0; i < conf.numParts; ++i) {

		// Loop over rigid body grid names associated with particle type
		const std::vector<String>& gridNames = conf.partRigidBodyGrid[i];
		for (int j = 0; j < gridNames.size(); ++j) {

			// Loop over RB types
			for (int rb = 0; rb < conf.numRigidTypes; ++rb) {
				RigidBodyType& t = conf.rigidBody[rb];
				const std::vector<String>& keys = t.potentialGridKeys;

				// Loop over potential grids
				for(int k = 0; k < keys.size(); k++) {
					// printf("    checking grid keys ");
					if (gridNames[j] == keys[k])
						t.numParticles[k] += conf.numPartsOfType[i];
				}
			}
		}
	}

	// Allocate each particles array
	for (int rb = 0; rb < conf.numRigidTypes; ++rb) {
		RigidBodyType& t = conf.rigidBody[rb];
		t.particles = new int*[t.numPotGrids];
		for (int i = 0; i < t.numPotGrids; ++i) {
			t.particles[i] = new int[t.numParticles[i]];
			t.numParticles[i] = 0; // now use this as a counter 
		}
	}

	// Set the number of particles; Loop over particle types
	for (int i = 0; i < conf.numParts; ++i) {
		int tmp[conf.numPartsOfType[i]]; // temporary array holding particles of type i
		int currId = 0;
		for (int j = 0; j < conf.num; ++j) {
			if (conf.type[j] == i)
				tmp[currId++] = j;
		}
		
		// Loop over rigid body grid names associated with particle type
		const std::vector<String>& gridNames = conf.partRigidBodyGrid[i];
		for (int j = 0; j < gridNames.size(); ++j) {

			// Loop over RB types
			for (int rb = 0; rb < conf.numRigidTypes; ++rb) {
				RigidBodyType& t = conf.rigidBody[rb];
				const std::vector<String>& keys = t.potentialGridKeys;

				// Loop over potential grids
				for(int k = 0; k < keys.size(); k++) {
					// printf("    checking grid keys ");
					if (gridNames[j] == keys[k]) {
						memcpy( &(t.particles[k][t.numParticles[k]]), tmp, sizeof(int)*currId );
						t.numParticles[k] += currId;
					}
				}
			}
		}
	}

	// Initialize device data for RB force pairs after std::vector is done growing

	// for (int i = 0; i < forcePairs.size(); i++)
	// 	forcePairs[i].initialize();
			
}


void RigidBodyController::updateForces(Vector3* pos_d, Vector3* force_d, int s) {
	if (s <= 1)
		gpuErrchk( hipProfilerStart() );

	
	// clear old forces
	for (int i = 0; i < rigidBodyByType.size(); i++) {
		for (int j = 0; j < rigidBodyByType[i].size(); j++) {
			RigidBody& rb = rigidBodyByType[i][j];
			rb.clearForce();
			rb.clearTorque();
		}
	}

	// Grid–particle forces
	for (int i = 0; i < rigidBodyByType.size(); i++) {
		callGridParticleForceKernel( pos_d, force_d, conf.rigidBody[i], rigidBodyByType[i], s );
	}

	// Grid–Grid forces
	if (forcePairs.size() > 0) {
		
		for (int i=0; i < forcePairs.size(); i++)
			forcePairs[i].callGridForceKernel(i,s);

		// each kernel call is followed by async memcpy for previous; now get last
		RigidBodyForcePair* fp = RigidBodyForcePair::lastRbForcePair;
		fp->retrieveForcesForGrid( fp->lastRbGridID );
		fp->lastRbGridID = -1;

		// stream sync was slower than device sync
		/* for (int i = 0; i < NUMSTREAMS; i++) { */
		/* 	const hipStream_t &s = RigidBodyForcePair::stream[i]; */
		/* 	gpuErrchk(hipStreamSynchronize( s ));  */
		/* } */
		gpuErrchk(hipDeviceSynchronize());
	
		for (int i=0; i < forcePairs.size(); i++)
			forcePairs[i].processGPUForces();

	}
}
void RigidBodyController::integrate(int step) {
 	// tell RBs to integrate
	for (int i = 0; i < rigidBodyByType.size(); i++) {
		for (int j = 0; j < rigidBodyByType[i].size(); j++) {
			RigidBody& rb = rigidBodyByType[i][j];
			
			// thermostat
			rb.addLangevin( random->gaussian_vector(), random->gaussian_vector() );
		}
	}

	if ( step % conf.outputPeriod == 0 ) { /* PRINT & INTEGRATE */
		if (step == 0) {						// first step so only start this cycle
			print(step);
			for (int i = 0; i < rigidBodyByType.size(); i++) {
				for (int j = 0; j < rigidBodyByType[i].size(); j++) {
					RigidBody& rb = rigidBodyByType[i][j];
					rb.integrate(0);	
				}
			}
		} else {										// finish last cycle
			for (int i = 0; i < rigidBodyByType.size(); i++) {
				for (int j = 0; j < rigidBodyByType[i].size(); j++) {
					RigidBody& rb = rigidBodyByType[i][j];
					rb.integrate(1);	
				}
			}
			print(step);

			// start this cycle
			for (int i = 0; i < rigidBodyByType.size(); i++) {
				for (int j = 0; j < rigidBodyByType[i].size(); j++) {
					RigidBody& rb = rigidBodyByType[i][j];
					rb.integrate(0);	
				}
			}
		}
	} else {											/* INTEGRATE ONLY */
		if (step == 0) {						// first step so only start this cycle
			print(step);
			for (int i = 0; i < rigidBodyByType.size(); i++) {
				for (int j = 0; j < rigidBodyByType[i].size(); j++) {
					RigidBody& rb = rigidBodyByType[i][j];
					rb.integrate(0);	
				}
			}
		} else {										// integrate end of last step and start of this one
			for (int i = 0; i < rigidBodyByType.size(); i++) {
				for (int j = 0; j < rigidBodyByType[i].size(); j++) {
					RigidBody& rb = rigidBodyByType[i][j];
					rb.integrate(2);	
				}
			}
		}
	}
}

// allocate and initialize an array of stream handles
hipStream_t *RigidBodyForcePair::stream = (hipStream_t *) malloc(NUMSTREAMS * sizeof(hipStream_t));
int RigidBodyForcePair::nextStreamID = 0;	 /* used during stream init */
int RigidBodyForcePair::lastRbGridID = -1; /* used to schedule kernel interaction */
RigidBodyForcePair* RigidBodyForcePair::lastRbForcePair = NULL;

void RigidBodyForcePair::createStreams() {
	for (int i = 0; i < NUMSTREAMS; i++)
		gpuErrchk( hipStreamCreate( &(stream[i]) ) );
		// gpuErrchk( hipStreamCreateWithFlags( &(stream[i]) , hipStreamNonBlocking ) );
}
Vector3 RigidBodyForcePair::getOrigin1(const int i) {
	const int k1 = gridKeyId1[i];
	return rb1->getOrientation()*type1->densityGrids[k1].getOrigin() + rb1->getPosition();
}
Vector3 RigidBodyForcePair::getOrigin2(const int i) {
	const int k2 = gridKeyId2[i];
	if (!isPmf)
		return rb2->getOrientation()*type2->potentialGrids[k2].getOrigin() + rb2->getPosition();
	else
		return type2->rawPmfs[k2].getOrigin();
}		
Matrix3 RigidBodyForcePair::getBasis1(const int i) {
	const int k1 = gridKeyId1[i];
	return rb1->getOrientation()*type1->densityGrids[k1].getBasis();
}
Matrix3 RigidBodyForcePair::getBasis2(const int i) {
	const int k2 = gridKeyId2[i];
	if (!isPmf)
		return rb2->getOrientation()*type2->potentialGrids[k2].getBasis();
	else
		return type2->rawPmfs[k2].getBasis();
}

// RBTODO: bundle several rigidbodypair evaluations in single kernel call
void RigidBodyForcePair::callGridForceKernel(int pairId, int s) {
	// get the force/torque between a pair of rigid bodies
	/* printf("  Updating rbPair forces\n"); */
	const int numGrids = gridKeyId1.size();

	/* if (s%10 != 0) */
	/* 	pairId = -1000; */

	// RBTODO: precompute certain common transformations and pass in kernel call
	for (int i = 0; i < numGrids; i++) {
		const int nb = numBlocks[i];
		const int k1 = gridKeyId1[i];
		const int k2 = gridKeyId2[i];
		const hipStream_t &s = stream[streamID[i]];

		/*
			ijk: index of grid value
			r: postion of point ijk in real space
			B: grid Basis
			o: grid origin
			R: rigid body orientation
			c: rigid body center

			B': R.B 
			c': R.o + c

  		/.––––––––––––––––––.
	  	| r = R.(B.ijk+o)+c |
	  	| r = B'.ijk + c'   |
	  	`––––––––––––––––––./
		*/
		Matrix3 B1 = getBasis1(i);
		Vector3 c = getOrigin1(i) - getOrigin2(i);
		
		Matrix3 B2 = getBasis2(i).inverse();

		
		// RBTODO: get energy
		if (!isPmf) {								/* pair of RBs */
			computeGridGridForce<<< nb, numThreads, NUMTHREADS*2*sizeof(Vector3), s >>>
				(type1->rawDensityGrids_d[k1], type2->rawPotentialGrids_d[k2],
				 B1, B2, c,
				 forces_d[i], torques_d[i]);
		} else {										/* RB with a PMF */
			computeGridGridForce<<< nb, numThreads, NUMTHREADS*2*sizeof(Vector3), s >>>
				(type1->rawDensityGrids_d[k1], type2->rawPmfs_d[k2],
				 B1, B2, c,
				 forces_d[i], torques_d[i]);
		}
		// retrieveForcesForGrid(i); // this is slower than approach below, unsure why
		
		if (lastRbGridID >= 0)
			lastRbForcePair->retrieveForcesForGrid(lastRbGridID);
		lastRbForcePair = this;
		lastRbGridID = i;
	}
}
void RigidBodyController::callGridParticleForceKernel(Vector3* pos_d, Vector3* force_d,
				const RigidBodyType& t, std::vector<RigidBody>& rbs, int s) {
	// get the force/torque on a rigid body, and forces on particles
	
	// RBTODO: consolidate CUDA stream management
	for (int i = 0; i < t.numPotGrids; ++i) {
		if (t.numParticles[i] == 0) continue;

		for (int j = 0; j < rbs.size(); ++j) {
			// const int nb = 500;
			/*
			  r: postion of particle in real space
			  B: grid Basis
			  o: grid origin
			  R: rigid body orientation
			  c: rigid body center

			  B': R.B 
			  c': R.o + c
			*/
			// Matrix3 B1 = getBasis1(i);
			Vector3 c =  rbs[j].getOrientation()*t.potentialGrids[i].getOrigin() + rbs[j].getPosition();
			Matrix3 B = (rbs[j].getOrientation()*t.potentialGrids[i].getBasis()).inverse();
		
			// RBTODO: get energy
			const int nb = (t.numParticles[i]/NUMTHREADS)+1;

			// RBTODO: IMPORTANT: Improve this
			Vector3 forces[nb];
			Vector3 torques[nb];
			for (int k=0; k < nb; ++k) {
				forces[k] = Vector3(0.0f);
				torques[k] = Vector3(0.0f);
			}
			Vector3* forces_d;
			Vector3* torques_d;			
			gpuErrchk(hipMalloc(&forces_d, sizeof(Vector3)*nb));
			gpuErrchk(hipMalloc(&torques_d, sizeof(Vector3)*nb));
			gpuErrchk(hipMemcpy(forces_d, forces, sizeof(Vector3)*nb, hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(torques_d, torques, sizeof(Vector3)*nb, hipMemcpyHostToDevice));
			
			computePartGridForce<<< nb, NUMTHREADS, NUMTHREADS*2*sizeof(Vector3) >>>(
				pos_d, force_d, t.numParticles[i], t.particles[i],
				t.rawPotentialGrids_d[i],
				B, c, forces_d, torques_d);

			gpuErrchk(hipMemcpy(forces, forces_d, sizeof(Vector3)*nb, hipMemcpyDeviceToHost));
			gpuErrchk(hipMemcpy(torques, torques_d, sizeof(Vector3)*nb, hipMemcpyDeviceToHost));

			Vector3 f = Vector3(0.0f);
			Vector3 t = Vector3(0.0f);
			for (int k = 0; k < nb; ++k) {
				f = f + forces[k];
				t = t + torques[j];
			}
			
			t = -t - (rbs[j].getPosition()-c).cross( -f ); 
			rbs[j].addForce( -f );
			rbs[j].addTorque( t );
		}
	}
}

void RigidBodyForcePair::retrieveForcesForGrid(const int i) {
	// i: grid ID (less than numGrids)
	const hipStream_t &s = stream[streamID[i]];
	const int nb = numBlocks[i];

	gpuErrchk(hipMemcpyAsync(forces[i], forces_d[i], sizeof(Vector3)*nb,
														hipMemcpyDeviceToHost, s));
	gpuErrchk(hipMemcpyAsync(torques[i], torques_d[i], sizeof(Vector3)*nb,
														hipMemcpyDeviceToHost, s));
	
}
void RigidBodyForcePair::processGPUForces() {
	
	const int numGrids = gridKeyId1.size();
	Vector3 f = Vector3(0.0f);
	Vector3 t = Vector3(0.0f);

	for (int i = 0; i < numGrids; i++) {
		const int nb = numBlocks[i];

		Vector3 tmpF = Vector3(0.0f);
		Vector3 tmpT = Vector3(0.0f);
			
		for (int j = 0; j < nb; j++) {
			tmpF = tmpF + forces[i][j];
			tmpT = tmpT + torques[i][j];
		}
		
		// tmpT is the torque calculated about the origin of grid k2 (e.g. c2)
		//   so here we transform torque to be about rb1
		Vector3 o2 = getOrigin2(i);
		tmpT = tmpT - (rb1->getPosition() - o2).cross( tmpF ); 

		// sum forces and torques
		f = f + tmpF;
		t = t + tmpT;
	}
   
	rb1->addForce( f );
	rb1->addTorque( t );

	if (!isPmf) {
		const Vector3 t2 = -t + (rb2->getPosition()-rb1->getPosition()).cross( f );
		rb2->addForce( -f );
		rb2->addTorque( t2 );
	}

	// printf("force: %s\n", f.toString().val());
	// printf("torque: %s\n", t.toString().val());
	
}

void RigidBodyController::copyGridsToDevice() {
	// RBTODO: clean this function up
	RigidBodyType **rb_addr = new RigidBodyType*[conf.numRigidTypes];	/* temporary pointer to device pointer */

	gpuErrchk(hipMalloc(&rbType_d, sizeof(RigidBodyType*) * conf.numRigidTypes));
	// TODO: The above line fails when there is not enough memory. If it fails, stop.

	printf("Copying RBs\n");
	// Copy rigidbody types 
	// http://stackoverflow.com/questions/16024087/copy-an-object-to-device
 	for (int i = 0; i < conf.numRigidTypes; i++)
		conf.rigidBody[i].updateRaw();


	// density grids
 	for (int i = 0; i < conf.numRigidTypes; i++) {
		printf("Copying density grids of RB type %d\n",i);
		RigidBodyType& rb = conf.rigidBody[i];

		int ng = rb.numDenGrids;
		rb.rawDensityGrids_d = new RigidBodyGrid*[ng]; /* not sure this is needed */
		
		printf("  RigidBodyType %d: numGrids = %d\n", i, ng);		
		// copy grid data to device
		for (int gid = 0; gid < ng; gid++) { 
			RigidBodyGrid* g = &(rb.rawDensityGrids[gid]); // convenience
			// RigidBodyGrid* g_d = rb.rawDensityGrids_d[gid]; // convenience
			int len = g->getSize();
			float* tmpData;

			size_t sz = sizeof(RigidBodyGrid);
			gpuErrchk(hipMalloc((void **) &(rb.rawDensityGrids_d[gid]), sz));
			/* gpuErrchk(hipMemcpy(rb.rawDensityGrids_d[gid], g, */
			/* 										 sz, hipMemcpyHostToDevice)); */
			gpuErrchk(hipMemcpy(rb.rawDensityGrids_d[gid], &(rb.rawDensityGrids[gid]),
													 sz, hipMemcpyHostToDevice));

			// allocate grid data on device
			// copy temporary host pointer to device pointer
			// copy data to device through temporary host pointer
			sz = sizeof(float) * len;
			gpuErrchk(hipMalloc((void **) &tmpData, sz)); 
			// gpuErrchk(hipMemcpy( tmpData, g->val, sz, hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy( tmpData, rb.rawDensityGrids[gid].val, sz, hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy( &(rb.rawDensityGrids_d[gid]->val), &tmpData,
														sizeof(float*), hipMemcpyHostToDevice));
		}
  }

	for (int i = 0; i < conf.numRigidTypes; i++) {
		printf("Working on RB %d\n",i);
		RigidBodyType& rb = conf.rigidBody[i];

		int ng = rb.numPotGrids;
		rb.rawPotentialGrids_d = new RigidBodyGrid*[ng]; /* not 100% sure this is needed, possible memory leak */

		printf("  RigidBodyType %d: numGrids = %d\n", i, ng);		
		// copy potential grid data to device
		for (int gid = 0; gid < ng; gid++) { 
			RigidBodyGrid* g = &(rb.rawPotentialGrids[gid]); // convenience
			// RigidBodyGrid* g_d = rb.rawDensityGrids_d[gid]; // convenience
			int len = g->getSize();
			float* tmpData;
			// tmpData = new float*[len];

			size_t sz = sizeof(RigidBodyGrid);
			gpuErrchk(hipMalloc((void **) &(rb.rawPotentialGrids_d[gid]), sz));
			gpuErrchk(hipMemcpy( rb.rawPotentialGrids_d[gid], &(rb.rawPotentialGrids[gid]),
													 sz, hipMemcpyHostToDevice ));

			// allocate grid data on device
			// copy temporary host pointer to device pointer
			// copy data to device through temporary host pointer
			sz = sizeof(float) * len;
			gpuErrchk(hipMalloc((void **) &tmpData, sz)); 
			// sz = sizeof(float) * len;
			gpuErrchk(hipMemcpy( tmpData, rb.rawPotentialGrids[gid].val, sz, hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy( &(rb.rawPotentialGrids_d[gid]->val), &tmpData,
														sizeof(float*), hipMemcpyHostToDevice));
			
				// RBTODO: why can't tmpData be deleted? 
			// delete[] tmpData;
		}
	}

	for (int i = 0; i < conf.numRigidTypes; i++) {
		printf("Copying PMFs for RB %d\n",i);
		RigidBodyType& rb = conf.rigidBody[i];

		int ng = rb.numPmfs;
		rb.rawPmfs_d = new RigidBodyGrid*[ng]; /* not 100% sure this is needed, possible memory leak */

		printf("  RigidBodyType %d: numPmfs = %d\n", i, ng);		

		// copy pmf grid data to device
		for (int gid = 0; gid < ng; gid++) { 
			RigidBodyGrid g = rb.rawPmfs[gid];
			int len = g.getSize();
			float* tmpData;
			// tmpData = new float*[len];

			size_t sz = sizeof(RigidBodyGrid);
			gpuErrchk(hipMalloc((void **) &(rb.rawPmfs_d[gid]), sz));
			gpuErrchk(hipMemcpy( rb.rawPmfs_d[gid], &g,
													 sz, hipMemcpyHostToDevice ));

			// allocate grid data on device
			// copy temporary host pointer to device pointer
			// copy data to device through temporary host pointer
			sz = sizeof(float) * len;
			gpuErrchk(hipMalloc((void **) &tmpData, sz)); 
			// sz = sizeof(float) * len;
			gpuErrchk(hipMemcpy( tmpData, rb.rawPmfs[gid].val, sz, hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy( &(rb.rawPmfs_d[gid]->val), &tmpData,
														sizeof(float*), hipMemcpyHostToDevice));
			
		}
	}
	
	gpuErrchk(hipDeviceSynchronize());
	printf("Done copying RBs\n");

	/* // DEBUG */
	/* RigidBodyType& rb = conf.rigidBody[0]; */
	/* printRigidBodyGrid<<<1,1>>>( rb.rawPotentialGrids_d[0] ); */
	/* gpuErrchk(hipDeviceSynchronize()); */
	/* printRigidBodyGrid<<<1,1>>>( rb.rawDensityGrids_d[0] ); */
	/* gpuErrchk(hipDeviceSynchronize()); */
}

void RigidBodyController::print(int step) {
	// modeled after outputExtendedData() in Controller.C
	if ( step >= 0 ) {
		// Write RIGID BODY trajectory file
		if ( step % conf.outputPeriod == 0 ) {
			if ( ! trajFile.rdbuf()->is_open() ) {
	      // open file
	      printf("OPENING RIGID BODY TRAJECTORY FILE\n");
				// RBTODO: backup_file(simParams->rigidBodyTrajectoryFile);

				char fname[140];
				strcpy(fname,outArg);
				strcat(fname, ".rb-traj");
	      trajFile.open(fname);
				
	      while (!trajFile) {
					/* if ( errno == EINTR ) {
						printf("Warning: Interrupted system call opening RIGIDBODY trajectory file, retrying.\n");
						trajFile.clear();
						trajFile.open(simParams->rigidBodyTrajectoryFile);
						continue;
					}
					*/ 
					//char err_msg[257];
					printf("Error opening RigidBody trajectory file %s",fname);
					exit(1);
	      }
	      trajFile << "# RigidBody trajectory file" << std::endl;
	      printLegend(trajFile);
			}
			printf("WRITING RIGID BODY COORDINATES AT STEP %d\n",step);
			printData(step,trajFile);
			trajFile.flush();    
		}
    
		// Write restart File
		/* if ( simParams->restartFrequency && */
		/* 		 ((step % simParams->restartFrequency) == 0) && */
		/* 		 (step != simParams->firstTimestep) )	{ */
		if ( step % conf.outputPeriod == 0 && step != 0 ){
			printf("RIGID BODY: WRITING RESTART FILE AT STEP %d\n", step);
			char fname[140];
			strcpy(fname,outArg);
			strcat(fname, ".rigid");
			// RBTODO: NAMD_backup_file(fname,".old"); /*  */
			std::ofstream restartFile(fname);
			while (!restartFile) {
				/* RBTODO 
	      if ( errno == EINTR ) {
					printf("Warning: Interrupted system call opening rigid body restart file, retrying.\n");
					restartFile.clear();
					restartFile.open(fname);
					continue;
	      }
				*/
	      printf("Error opening rigid body restart file %s",fname);
	      exit(1); // NAMD_err(err_msg);
			}
			restartFile << "# RigidBody restart file" << std::endl;
			printLegend(restartFile);
			printData(step,restartFile);
			if (!restartFile) {
	      printf("Error writing rigid body restart file %s",fname);
	      exit(-1); // NAMD_err(err_msg);
			} 
		}
	}
}
void RigidBodyController::printLegend(std::ofstream &file) {
        file << "#$LABELS step RigidBodyKey"
		 << " posX  posY  posZ"
		 << " rotXX rotXY rotXZ"
		 << " rotYX rotYY rotYZ"
		 << " rotZX rotZY rotZZ"
		 << " velX  velY  velZ"
		 << " angVelX angVelY angVelZ" << std::endl;
}
void RigidBodyController::printData(int step,std::ofstream &file) {
	// tell RBs to integrate
	for (int i = 0; i < rigidBodyByType.size(); i++) {
		for (int j = 0; j < rigidBodyByType[i].size(); j++) {
			const RigidBody& rb = rigidBodyByType[i][j];
			
			Vector3 v =  rb.getPosition();
			Matrix3 t =  rb.getOrientation();
			file << step <<" "<< rb.getKey()
					 <<" "<< v.x <<" "<< v.y <<" "<< v.z;
			file <<" "<< t.exx <<" "<< t.exy <<" "<< t.exz
					 <<" "<< t.eyx <<" "<< t.eyy <<" "<< t.eyz
					 <<" "<< t.ezx <<" "<< t.ezy <<" "<< t.ezz;
			v = rb.getVelocity();
			file <<" "<< v.x <<" "<< v.y <<" "<< v.z;
			v = rb.getAngularVelocity();
			file <<" "<< v.x <<" "<< v.y <<" "<< v.z
					 << std::endl;
		}
	}
}

int RigidBodyForcePair::initialize() {
	printf("    Initializing (streams for) RB force pair...\n");

	const int numGrids = gridKeyId1.size();
	// RBTODO assert gridKeysIds are same size 

	// allocate memory for forces/torques
	for (int i = 0; i < numGrids; i++) {
		const int k1 = gridKeyId1[i];
		const int sz = type1->rawDensityGrids[k1].getSize();
		const int nb = sz / numThreads + ((sz % numThreads == 0) ? 0:1 );
		streamID.push_back( nextStreamID % NUMSTREAMS );
		nextStreamID++;

		numBlocks.push_back(nb);
		forces.push_back( new Vector3[nb] );
		torques.push_back( new Vector3[nb] );

		forces_d.push_back( new Vector3[nb] ); // RBTODO: correct?
		torques_d.push_back( new Vector3[nb] );

		// allocate device memory for numBlocks of torque, etc.
    // printf("      Allocating device memory for forces/torques\n");
		gpuErrchk(hipMalloc(&(forces_d[i]), sizeof(Vector3) * nb));
		gpuErrchk(hipMalloc(&(torques_d[i]), sizeof(Vector3) * nb));
	}
	gpuErrchk(hipDeviceSynchronize());
	// printf("    Done initializing RB force pair\n");
	return nextStreamID;
}

void RigidBodyForcePair::swap(RigidBodyForcePair& a, RigidBodyForcePair& b) {
	using std::swap;
	swap(a.type1, b.type1);
	swap(a.type2, b.type2);
	swap(a.rb1, b.rb1);
	swap(a.rb2, b.rb2);

	swap(a.gridKeyId1, b.gridKeyId1);
	swap(a.gridKeyId2, b.gridKeyId2);

	swap(a.numBlocks, b.numBlocks);

	swap(a.forces,    b.forces);
	swap(a.forces_d,  b.forces_d);
	swap(a.torques,   b.torques);
	swap(a.torques_d, b.torques_d);
}


RigidBodyForcePair::~RigidBodyForcePair() {
	printf("    Destructing RB force pair\n");
	const int numGrids = gridKeyId1.size();

	// printf("      numGrids = %d\n",numGrids);

	// RBTODO assert gridKeysIds are same size 

	// allocate memory for forces/torques
	if (streamID.size() > 0) {
		for (int i = 0; i < numGrids; i++) {
			const int k1 = gridKeyId1[i];
			const int nb = numBlocks[i];

			// free device memory for numBlocks of torque, etc.
			// printf("      Freeing device memory for forces/torques\n");
			gpuErrchk(hipFree( forces_d[i] ));	
			gpuErrchk(hipFree( torques_d[i] ));
		}
		gpuErrchk(hipDeviceSynchronize());
	}
	streamID.clear();
	numBlocks.clear();
	forces.clear();
	forces_d.clear();
	torques.clear();
	torques_d.clear();
}



