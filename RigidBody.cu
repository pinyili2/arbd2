#include "hip/hip_runtime.h"
/* #ifndef MIN_DEBUG_LEVEL */
/* #define MIN_DEBUG_LEVEL 5 */
/* #endif */
/* #include "Debug.h" */

#include <iostream>
#include <typeinfo>
#include "RigidBody.h"
#include "Configuration.h"

#include "Debug.h"


RigidBody::RigidBody(const Configuration& cref, RigidBodyType& tref)
	: c(&cref), t(&tref), impulse_to_momentum(0.0004184) {

	timestep = c->timestep;
	// RBTODO: fix this
	Temp = 295;
	// tempgrid = c->temperatureGrid;

	position = Vector3();

	// Orientation matrix that brings vector from the RB frame to the lab frame
	orientation = Matrix3(1.0f);
	
	momentum = Vector3() * t->mass; // lab frame
	/* DebugM(4, "velocity " << rbParams->velocity << "\n" << endi); */
	DebugM(4, "momentum " << momentum << "\n" << endi);

	angularMomentum = Vector3(); // rigid body frame
	angularMomentum.x *= t->inertia.x;
	angularMomentum.y *= t->inertia.y;
	angularMomentum.z *= t->inertia.z;

	/* isFirstStep = true; // this might not work flawlessly... */

	/* clearForce(); */
	/* clearTorque(); */
    
	/* DebugM(4, "RigidBody initial Force: " << force << "\n" << endi); */
}

void RigidBody::addForce(Force f) { 
	// DebugM(1, "RigidBody "<<key<<" adding f ("<<f<<") to Force " << force << "\n" << endi);    
	force += f; 
} 
void RigidBody::addTorque(Force torq) {
	// DebugM(1, "RigidBody adding t ("<<t<<") to torque " << torque << "\n" << endi);   
	torque += torq; 
}
RigidBody::~RigidBody() {}

	/*===========================================================================\
	| Following "Algorithm for rigid-body Brownian dynamics" Dan Gordon, Matthew |
	|   Hoyles, and Shin-Ho Chung                                                |
	|   http://langevin.anu.edu.au/publications/PhysRevE_80_066703.pdf           |
	|                                                                            |
	|                                                                            |
	| BUT: assume diagonal friction tensor and no Wiener process / stochastic    |
	|   calculus then this is just the same as for translation                   |
	|                                                                            |
	|   < T_i(t) T_i(t) > = 2 kT friction inertia                                |
	|                                                                            |
	|   friction / kt = Diff                                                     |
	\===========================================================================*/
void RigidBody::addLangevin(Vector3 w1, Vector3 w2) {
	// w1 and w2 should be standard normal distributions

	// in RB frame     
	Vector3 tmp = orientation.transpose()*momentum;
	Force f = Vector3::element_mult(t->transForceCoeff,w1) -
		Vector3::element_mult(t->transDamping, orientation.transpose()*momentum); 
    
	Force torq = Vector3::element_mult(t->rotTorqueCoeff,w2) -
		Vector3::element_mult(t->rotDamping, angularMomentum);

	f = orientation * f; // return to lab frame
	torq = orientation * torq;
    
	addForce(f);
	addTorque(torq);
}

  /*==========================================================================\
	| from: Dullweber, Leimkuhler, Maclachlan. Symplectic splitting methods for |
	| rigid body molecular dynamics. JCP 107. (1997)                            |
	| http://jcp.aip.org/resource/1/jcpsa6/v107/i15/p5840_s1                    |
	\==========================================================================*/
// void RigidBody::integrate(Vector3& old_trans, Matrix3& old_rot, int startFinishAll) {}
void RigidBody::integrate(int startFinishAll) {
	Vector3 trans; // = *p_trans;
	Matrix3 rot = Matrix3(1); // = *p_rot;

#ifdef DEBUGM
	switch (startFinishAll) {
	case 0: // start
		DebugM(2, "Rigid Body integrating start of cycle" << "\n" << endi);
	case 1: // finish
		DebugM(2, "Rigid Body integrating finish of cycle" << "\n" << endi);
	case 2: // finish and start
		DebugM(2, "Rigid Body integrating finishing last cycle, starting this one" << "\n" << endi);
	}    
#endif

	if ( isnan(force.x) || isnan(torque.x) ) { // NaN check
		printf("Rigid Body force was NaN!\n");
		exit(-1);
	}

	// torque = Vector(0,0,10); // debug
	Force tmpTorque = orientation.transpose()*torque; // bring to rigid body frame

	DebugM(3, "integrate" <<": force "<<force <<": velocity "<<getVelocity() << "\n" << endi);
	DebugM(3, "integrate" <<": torque "<<tmpTorque <<": orientationalVelocity "<<getAngularVelocity() << "\n" << endi);

	if (startFinishAll == 0 || startFinishAll == 1) {
		// propogate momenta by half step
		momentum += 0.5 * timestep * force * impulse_to_momentum;
		angularMomentum += 0.5 * timestep * tmpTorque * impulse_to_momentum;
	} else {
		// propogate momenta by a full timestep
		momentum += timestep * force * impulse_to_momentum;
		angularMomentum += timestep * tmpTorque * impulse_to_momentum;
	}

	DebugM(3, "  position before: " << position << "\n" << endi);

	if (startFinishAll == 0 || startFinishAll == 2) {
		// update positions
		// trans = Vector(0); if (false) {
		trans = timestep * momentum / t->mass;
		position += trans; // update CoM a full timestep
		// }

		// update orientations a full timestep
		Matrix3 R; // represents a rotation about a principle axis
		R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x ); // R1
		angularMomentum = R * angularMomentum;
		rot = R.transpose();
		DebugM(1, "R: " << R << "\n" << endi);
		DebugM(1, "Rot 1: " << rot << "\n" << endi);

		R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y ); // R2
		angularMomentum = R * angularMomentum;
		rot = rot * R.transpose();
		DebugM(1, "R: " << R << "\n" << endi);
		DebugM(1, "Rot 2: " << rot << "\n" << endi);

		R = Rz(    timestep * angularMomentum.z / t->inertia.z ); // R3
		angularMomentum = R * angularMomentum;
		rot = rot * R.transpose();
		DebugM(1, "R: " << R << "\n" << endi);
		DebugM(1, "Rot 3: " << rot << "\n" << endi);

		R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y ); // R4
		angularMomentum = R * angularMomentum;
		rot = rot * R.transpose();
		DebugM(1, "R: " << R << "\n" << endi);
		DebugM(1, "Rot 4: " << rot << "\n" << endi);

		R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x ); // R5
		angularMomentum = R * angularMomentum;
		rot = rot * R.transpose();
		DebugM(1, "R: " << R << "\n" << endi);
		DebugM(1, "Rot 5: " << rot << "\n" << endi);

		// DebugM(3,"TEST: " << Ry(0.01) <<"\n" << endi); // DEBUG
 
		// update actual orientation
		Matrix3 newOrientation = orientation*rot; // not 100% sure; rot could be in rb frame
		orientation = newOrientation;
		/* rot = rot.transpose(); */

		/* DebugM(2, "trans during: " << trans */
		/* 			 << "\n" << endi); */
		/* DebugM(2, "rot during: " << rot */
		/* 			 << "\n" << endi); */
    
		/* clearForce(); */
		/* clearTorque(); */
	
		/* old_trans = trans; */
		/* old_rot = rot; */
	}
	DebugM(3, "  position after: " << position << "\n" << endi);
}    

// Rotations about axes
// for very small angles 10^-8, cos^2+sin^2 != 1 
// concerned about the accumulation of errors in non-unitary transformations!
Matrix3 RigidBody::Rx(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	Matrix3 tmp;
	tmp.exx = 1; tmp.exy =   0; tmp.exz =    0;
	tmp.eyx = 0; tmp.eyy = cos; tmp.eyz = -sin;
	tmp.ezx = 0; tmp.ezy = sin; tmp.ezz =  cos;
	return tmp;
}
Matrix3 RigidBody::Ry(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	Matrix3 tmp;
	tmp.exx =  cos; tmp.exy = 0; tmp.exz = sin;
	tmp.eyx =    0; tmp.eyy = 1; tmp.eyz =   0;
	tmp.ezx = -sin; tmp.ezy = 0; tmp.ezz = cos;
	return tmp;
}
Matrix3 RigidBody::Rz(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	Matrix3 tmp;
	tmp.exx = cos; tmp.exy = -sin; tmp.exz = 0;
	tmp.eyx = sin; tmp.eyy =  cos; tmp.eyz = 0;
	tmp.ezx =   0; tmp.ezy =    0; tmp.ezz = 1;
	return tmp;
}
Matrix3 RigidBody::eulerToMatrix(const Vector3 e) {
	// convert euler angle input to rotation matrix
	// http://en.wikipedia.org/wiki/Rotation_formalisms_in_three_dimensions#Conversion_formulae_between_formalisms
	return Rz(e.z) * Ry(e.y) * Rx(e.x);
}
