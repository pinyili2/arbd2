#include "hip/hip_runtime.h"
#include "CUDA.h"

#ifdef USE_CUDA
// __global__ void BDIntegrate_kernel() {
//     if (threadIdx.x == 0) {
// 	printf("BDIntegrate_kernel()\n");
// 	IntegratorKernels::BDIntegrate();
//     }
// };

// void BDIntegrateCUDA::compute(Patch* p) {
//     printf("BDIntegrateCUDA::compute()\n");
//     BDIntegrate_kernel<<<1,32>>>();
// };

PatchCUDA::PatchCUDA() : Patch() {
    pos_force_d = momentum_d = rb_pos_d = rb_orient_d = rb_mom_d = rb_ang_mom_d = type_d = rb_type_d = nullptr;
}
#endif
