#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////
// Brownian dynamics base class
// Author: Jeff Comer <jcomer2@illinois.edu>

#include "ComputeForce.h"
#include "ComputeForce.cuh"
#include <hip/hip_runtime_api.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

hipEvent_t start, stop;

void runSort(int2 *d1, int *d2, float *key,
				int2 *scratch1, int  *scratch2, float *scratchKey,
				unsigned int count);

ComputeForce::ComputeForce(int num, const BrownianParticleType part[],
													 int numParts, const BaseGrid* g, float switchStart,
													 float switchLen, float electricConst,
													 int fullLongRange, int numBonds, int numTabBondFiles,
													 int numExcludes, int numAngles, int numTabAngleFiles,
													 int numDihedrals, int numTabDihedralFiles,
													 int numReplicas) :
		num(num), numParts(numParts), sys(g), switchStart(switchStart),
		switchLen(switchLen), electricConst(electricConst),
		cutoff2((switchLen + switchStart) * (switchLen + switchStart)),
		decomp(g->getBox(), g->getOrigin(), switchStart + switchLen, numReplicas),
		numBonds(numBonds), numTabBondFiles(numTabBondFiles),
		numExcludes(numExcludes), numAngles(numAngles),
		numTabAngleFiles(numTabAngleFiles), numDihedrals(numDihedrals),
		numTabDihedralFiles(numTabDihedralFiles), numReplicas(numReplicas) {
	// Allocate the parameter tables.
	decomp_d = NULL;

	tableEps = new float[numParts * numParts];
	tableRad6 = new float[numParts * numParts];
	tableAlpha = new float[numParts * numParts];

	const size_t tableSize = sizeof(float) * numParts * numParts;
	gpuErrchk(hipMalloc(&tableEps_d, tableSize));
	gpuErrchk(hipMalloc(&tableRad6_d, tableSize));
	gpuErrchk(hipMalloc(&tableAlpha_d, tableSize));
	gpuErrchk(hipMalloc(&sys_d, sizeof(BaseGrid)));
	gpuErrchk(hipMemcpyAsync(sys_d, sys, sizeof(BaseGrid), hipMemcpyHostToDevice));
	// Form the parameter tables.
	makeTables(part);

	gpuErrchk(hipMemcpyAsync(tableAlpha_d, tableAlpha, tableSize,
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableEps_d, tableEps, tableSize,
			hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableRad6_d, tableRad6, tableSize,
			hipMemcpyHostToDevice));

	// Create the potential table
	tablePot = new TabulatedPotential*[numParts * numParts];
	tablePot_addr = new TabulatedPotential*[numParts * numParts];
	for (int i = 0; i < numParts*numParts; i++) {
		tablePot_addr[i] = NULL;
		tablePot[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tablePot_d, sizeof(TabulatedPotential*) * numParts * numParts));

	// Create the bond table
	tableBond = new TabulatedPotential*[numTabBondFiles];
	tableBond_addr = new TabulatedPotential*[numTabBondFiles];
	for (int i = 0; i < numTabBondFiles; i++) {
		tableBond_addr[i] = NULL;
		tableBond[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableBond_d, sizeof(TabulatedPotential*) * numTabBondFiles));

	// Create the angle table
	tableAngle = new TabulatedAnglePotential*[numTabAngleFiles];
	tableAngle_addr = new TabulatedAnglePotential*[numTabAngleFiles];
	for (int i = 0; i < numTabAngleFiles; i++) {
		tableAngle_addr[i] = NULL;
		tableAngle[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableAngle_d, sizeof(TabulatedAnglePotential*) * numTabAngleFiles));

	// Create the dihedral table
	tableDihedral = new TabulatedDihedralPotential*[numTabDihedralFiles];
	tableDihedral_addr = new TabulatedDihedralPotential*[numTabDihedralFiles];
	for (int i = 0; i < numTabDihedralFiles; i++) {
		tableDihedral_addr[i] = NULL;
		tableDihedral[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableDihedral_d, sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles));

	//Calculate the number of blocks the grid should contain
	gridSize =  num / NUM_THREADS + 1;

	// Create and allocate the energy arrays
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * num));
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

ComputeForce::~ComputeForce() {
	delete[] tableEps;
	delete[] tableRad6;
	delete[] tableAlpha;
	hipFree(tableEps_d);
	hipFree(tableAlpha_d);
	hipFree(tableRad6_d);

	for (int j = 0; j < numParts * numParts; ++j)
		delete tablePot[j];
	delete[] tablePot;
	delete[] tablePot_addr;

	for (int j = 0; j < numTabBondFiles; ++j)
		delete tableBond[j];
	delete[] tableBond;
	delete[] tableBond_addr;
	gpuErrchk(hipFree(tableBond_d));

	for (int j = 0; j < numTabAngleFiles; ++j)
		if (tableAngle[j] != NULL)
			delete tableAngle[j];
	delete[] tableAngle;
	delete[] tableAngle_addr;
	gpuErrchk(hipFree(tableAngle_d));

	gpuErrchk(hipFree(energies_d));

	gpuErrchk(hipFree(sys_d));
}

void ComputeForce::updateNumber(Vector3* pos, int type[], int newNum) {
	if (newNum == num or newNum < 0) return;

	// Set the new number.
	num = newNum;

	// Reallocate the neighbor list.
	//delete[] neigh;
	//neigh = new IndexList[num];
	decompose(pos, type);

	printf("updateNumber() called\n");
	// Reallocate CUDA arrays

	// Recalculate the number of blocks in the grid
	gridSize = 0;
	while ((int)sqrt(NUM_THREADS) * gridSize < num)
		++gridSize;

	gpuErrchk(hipFree(energies_d));
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * gridSize));
}

void ComputeForce::makeTables(const BrownianParticleType part[]) {
	for (int i = 0; i < numParts; ++i) {
		const BrownianParticleType& pi = part[i];
		for (int j = 0; j < numParts; ++j) {
			const BrownianParticleType& pj = part[j];
			int ind = i * numParts + j;
			tableEps[ind] = sqrtf(pi.eps * pj.eps);
			float r = pi.radius + pj.radius;
			tableRad6[ind] = r * r * r * r * r * r;
			tableAlpha[ind] = electricConst * pi.charge * pj.charge;
		}
	}
}

bool ComputeForce::addTabulatedPotential(String fileName, int type0, int type1) {
	if (type0 < 0 or type0 >= numParts) return false;
	if (type1 < 0 or type1 >= numParts) return false;

	int ind = type0 + type1 * numParts;
	int ind1 = type1 + type0 * numParts;

	// If an entry already exists for this particle type, delete it
	if (tablePot[ind] != NULL) {
		delete tablePot[ind];
		gpuErrchk(hipFree(tablePot_addr[ind]));
		tablePot[ind] = NULL;
		tablePot_addr[ind] = NULL;
	}
	if (tablePot[ind1] != NULL) {
		gpuErrchk(hipFree(tablePot_addr[ind1]));
		delete tablePot[ind1];
		tablePot[ind1] = NULL;
		tablePot_addr[ind1] = NULL;
	}

	tablePot[ind] = new TabulatedPotential(fileName);
	tablePot[ind]->truncate(switchStart, sqrtf(cutoff2), 0.0f);
	tablePot[ind1] = new TabulatedPotential(*(tablePot[ind]));

	TabulatedPotential* t = new TabulatedPotential(*tablePot[ind]);

	// Copy tablePot[ind] to the device
	float *v0, *v1, *v2, *v3;
	size_t sz_n = sizeof(float) * tablePot[ind]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tablePot[ind]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tablePot[ind]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tablePot[ind]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tablePot[ind]->v3, sz_n, hipMemcpyHostToDevice));
	t->v0 = v0; t->v1 = v1;
	t->v2 = v2; t->v3 = v3;
	gpuErrchk(hipMalloc(&tablePot_addr[ind], sizeof(TabulatedPotential)));
	gpuErrchk(hipMemcpy(tablePot_addr[ind], t, sizeof(TabulatedPotential), hipMemcpyHostToDevice));
	t->v0 = NULL; t->v1 = NULL;
	t->v2 = NULL; t->v3 = NULL;
	delete t;
	/** Same thing for ind1 **/
	t = new TabulatedPotential(*tablePot[ind1]);
	sz_n = sizeof(float) * tablePot[ind1]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tablePot[ind1]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tablePot[ind1]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tablePot[ind1]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tablePot[ind1]->v3, sz_n, hipMemcpyHostToDevice));
	t->v0 = v0; t->v1 = v1;
	t->v2 = v2; t->v3 = v3;
	gpuErrchk(hipMalloc(&tablePot_addr[ind1], sizeof(TabulatedPotential)));
	gpuErrchk(hipMemcpy(tablePot_addr[ind1], t, sizeof(TabulatedPotential), hipMemcpyHostToDevice));
	t->v0 = NULL; t->v1 = NULL;
	t->v2 = NULL; t->v3 = NULL;
	delete t;
	gpuErrchk(hipMemcpy(tablePot_d, tablePot_addr,
			sizeof(TabulatedPotential*) * numParts * numParts, hipMemcpyHostToDevice));

	return true;
}

bool ComputeForce::addBondPotential(String fileName, int ind,
																		Bond bonds[], Bond bonds_d[]) {
	if (tableBond[ind] != NULL) {
		delete tableBond[ind];
		gpuErrchk(hipFree(tableBond_addr[ind]));
		tableBond[ind] = NULL;
		tableBond_addr[ind] = NULL;
	}
	tableBond[ind] = new TabulatedPotential(fileName);
	tableBond[ind]->truncate(switchStart, sqrtf(cutoff2), 0.0f);

	for (int i = 0; i < numBonds; ++i)
		if (bonds[i].fileName == fileName)
			bonds[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(bonds_d, bonds, sizeof(Bond) * numBonds, hipMemcpyHostToDevice));

	// Copy tableBond[ind] to the device
	float *v0, *v1, *v2, *v3;
	size_t sz_n = sizeof(float) * tableBond[ind]->n;
	gpuErrchk(hipMalloc(&v0, sz_n));
	gpuErrchk(hipMalloc(&v1, sz_n));
	gpuErrchk(hipMalloc(&v2, sz_n));
	gpuErrchk(hipMalloc(&v3, sz_n));
	gpuErrchk(hipMemcpyAsync(v0, tableBond[ind]->v0, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v1, tableBond[ind]->v1, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v2, tableBond[ind]->v2, sz_n, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(v3, tableBond[ind]->v3, sz_n, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&tableBond_addr[ind], sizeof(TabulatedPotential)));
	TabulatedPotential t = TabulatedPotential(*tableBond[ind]);
	t.v0 = v0; t.v1 = v1;
	t.v2 = v2; t.v3 = v3;
	gpuErrchk(hipMemcpyAsync(tableBond_addr[ind], &t,
			sizeof(TabulatedPotential), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpy(tableBond_d, tableBond_addr,
			sizeof(TabulatedPotential*) * numTabBondFiles, hipMemcpyHostToDevice));
	t.v0 = NULL; t.v1 = NULL;
	t.v2 = NULL; t.v3 = NULL;
	return true;
}

bool ComputeForce::addAnglePotential(String fileName, int ind, Angle* angles, Angle* angles_d) {
	if (tableAngle[ind] != NULL) {
		delete tableAngle[ind];
		gpuErrchk(hipFree(tableAngle_addr[ind]));
		tableAngle[ind] = NULL;
		tableAngle_addr[ind] = NULL;
	}

	tableAngle[ind] = new TabulatedAnglePotential(fileName);
	TabulatedAnglePotential *t = new TabulatedAnglePotential(*tableAngle[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableAngle[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableAngle[ind]->pot, sizeof(float) * size, hipMemcpyHostToDevice));
	t->pot = pot;
	gpuErrchk(hipMalloc(&tableAngle_addr[ind], sizeof(TabulatedAnglePotential)));
	gpuErrchk(hipMemcpy(tableAngle_addr[ind], t, sizeof(TabulatedAnglePotential), hipMemcpyHostToDevice));
	t->pot = NULL;
	delete t;

	gpuErrchk(hipMemcpyAsync(tableAngle_d, tableAngle_addr,
			sizeof(TabulatedAnglePotential*) * numTabAngleFiles, hipMemcpyHostToDevice));

	for (int i = 0; i < numAngles; i++)
		if (angles[i].fileName == fileName)
			angles[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpy(angles_d, angles, sizeof(Angle) * numAngles,
			hipMemcpyHostToDevice));
	return true;
}

bool ComputeForce::addDihedralPotential(String fileName, int ind,
																				Dihedral dihedrals[],
																				Dihedral dihedrals_d[]) {
	for (int i = 0; i < numDihedrals; i++)
		if (dihedrals[i].fileName == fileName)
			dihedrals[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(dihedrals_d, dihedrals, sizeof(Dihedral) * numDihedrals,
			hipMemcpyHostToDevice));

	if (tableDihedral[ind] != NULL) {
		delete tableDihedral[ind];
		gpuErrchk(hipFree(tableDihedral_addr[ind]));
		tableDihedral[ind] = NULL;
		tableDihedral_addr[ind] = NULL;
	}

	tableDihedral[ind] = new TabulatedDihedralPotential(fileName);
	TabulatedDihedralPotential t = TabulatedDihedralPotential(*tableDihedral[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableDihedral[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableDihedral[ind]->pot,
			sizeof(float) * size, hipMemcpyHostToDevice));
	t.pot = pot;

	gpuErrchk(hipMalloc(&tableDihedral_addr[ind], sizeof(TabulatedDihedralPotential)));
	gpuErrchk(hipMemcpyAsync(tableDihedral_addr[ind], &t,
			sizeof(TabulatedDihedralPotential), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(tableDihedral_d, tableDihedral_addr,
			sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles, hipMemcpyHostToDevice));
	t.pot = NULL;
	return true;
}

void ComputeForce::decompose(Vector3* pos, int type[]) {
	gpuErrchk( hipProfilerStart() );
	// Reset the cell decomposition.
	bool newDecomp = false;
	if (decomp_d)
		hipFree(decomp_d);
	else
		newDecomp = true;
		
	decomp.decompose_d(pos, num);
	decomp_d = decomp.copyToCUDA();

	// Update pairlists using cell decomposition (not sure this is really needed or good) 
	//RBTODO updatePairlists<<< nBlocks, NUM_THREADS >>>(pos_d, num, numReplicas, sys_d, decomp_d);	


	/* size_t free, total; */
	/* { */
	/* 	hipMemGetInfo(&free,&total); */
	/* 	printf("Free memory: %zu / %zu\n", free, total); */
	/* } */
	
	// initializePairlistArrays
	int nCells = decomp.nCells.x * decomp.nCells.y * decomp.nCells.z;
	int blocksPerCell = 10;
	if (newDecomp) {
		// RBTODO: free memory elsewhere
		// allocate device data
		// initializePairlistArrays<<< 1, 32 >>>(10*nCells*blocksPerCell);
		const int maxPairs = 1<<25;
		gpuErrchk(hipMalloc(&numPairs_d,       sizeof(int)));

		gpuErrchk(hipMalloc(&pairLists_d,      sizeof(int2)*maxPairs));
		gpuErrchk(hipMalloc(&pairTabPotType_d, sizeof(int)*maxPairs));

		gpuErrchk(hipDeviceSynchronize());
	}

	
	/* hipMemGetInfo(&free,&total); */
	/* printf("Free memory: %zu / %zu\n", free, total); */
	
	const int NUMTHREADS = 128;
	//const size_t nBlocks = (num * numReplicas) / NUM_THREADS + 1;
	const size_t nBlocks = nCells*blocksPerCell;

	/* clearPairlists<<< 1, 32 >>>(pos, num, numReplicas, sys_d, decomp_d); */
	/* gpuErrchk(hipDeviceSynchronize()); */
	/* pairlistTest<<< nBlocks, NUMTHREADS >>>(pos, num, numReplicas, */
	/* 																					 sys_d, decomp_d, nCells, blocksPerCell, */
	/* 																					 numPairs_d, pairListListI_d, pairListListJ_d); */
	/* gpuErrchk(hipDeviceSynchronize());	 */

	{
		int tmp = 0;
		gpuErrchk(hipMemcpyAsync(numPairs_d, &tmp,
															sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipDeviceSynchronize());
	}

	
	float pairlistdist2 = (sqrt(cutoff2) + 2.0f);
	pairlistdist2 = pairlistdist2*pairlistdist2;
	
	createPairlists<<< 2048, 64 >>>(pos, num, numReplicas,
					sys_d, decomp_d, nCells,
					numPairs_d, pairLists_d,
					numParts, type, pairTabPotType_d, pairlistdist2);
	/* createPairlistsOld<<< nBlocks, NUMTHREADS >>>(pos, num, numReplicas, */
	/* 																					 sys_d, decomp_d, nCells, blocksPerCell, */
	/* 																					 numPairs_d, pairLists_d, */
	/* 																					 numParts, type, pairTabPotType_d, pairlistdist2); */

	gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
	// if (false)
	{ // sort pairlist
		int numPairs;
		gpuErrchk(hipMemcpyAsync( &numPairs, numPairs_d, sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
		printf("here, %d pairs\n", numPairs);
		/* runSort(pairLists_d, pairTabPotType_d, pairDists_d, */
		/* 				pairLists_s, pairTabPotType_s, pairDists_s, */
		/* 				numPairs); */
		/* printf("done\n"); */
		
		/* // RBTODO: sort pairListInd as well!!! (i.e. roll your own sort!) */
		/* // thrust::sort_by_key( pairDists_d, pairDists_d+numPairs_d, pairLists_d ); */
		/* // thrust::sort_by_key( pairDists_d, pairDists_d+numPairs_d, pairLists_d ); */
		/* gpuErrchk(hipDeviceSynchronize()); /\* RBTODO: sync needed here? *\/ */
	}
}

IndexList ComputeForce::decompDim() const {
	IndexList ret;
	ret.add(decomp.getNx());
	ret.add(decomp.getNy());
	ret.add(decomp.getNz());
	return ret;
}

CellDecomposition ComputeForce::getDecomp() { return decomp; }

float ComputeForce::decompCutoff() { return decomp.getCutoff(); }

// TODO: Fix this
int* ComputeForce::neighborhood(Vector3 r) {
	// return decomp.getCell(r)->getNeighbors();
	return NULL;
}

float ComputeForce::computeFull(Vector3* force, Vector3* pos, int* type, bool get_energy) {
	float energy = 0.0f;
	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeFullKernel<<< numBlocks, numThreads >>>(force, pos, type, tableAlpha_d,
		tableEps_d, tableRad6_d, num, numParts, sys_d, energies_d, gridSize,
		numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeSoftcoreFull(Vector3* force, Vector3* pos, int* type, bool get_energy) {
	float energy = 0.0f;
	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeSoftcoreFullKernel<<<numBlocks, numThreads>>>(force, pos, type,
			tableEps_d, tableRad6_d, num, numParts, sys_d, energies_d, gridSize,
			numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		hipDeviceSynchronize();
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeElecFull(Vector3* force, Vector3* pos,
		int* type, bool get_energy) {
	float energy = 0.0f;

	gridSize = num/NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeElecFullKernel<<<numBlocks, numThreads>>>(force, pos, type,
			tableAlpha_d, num, numParts, sys_d, energies_d, gridSize, numReplicas,
			get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}


float ComputeForce::compute(Vector3 force[], Vector3 pos[], int type[], bool get_energy) {
	float energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeKernel<<<numBlocks, numThreads>>>(force, pos, type,
			tableAlpha_d, tableEps_d, tableRad6_d, num, numParts, sys_d,
			decomp_d, energies_d, switchStart, switchLen, gridSize, numReplicas,
			get_energy);

	gpuErrchk(hipDeviceSynchronize());
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeTabulated(Vector3* force, Vector3* pos, int* type,
		Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap,
		Angle* angles, Dihedral* dihedrals, bool get_energy) {
	float energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);
	
	// Call the kernel to calculate the forces
	// int nb = (decomp.nCells.x * decomp.nCells.y * decomp.nCells.z);
	// int nb = (1+(decomp.nCells.x * decomp.nCells.y * decomp.nCells.z)) * 75; /* RBTODO: number of pairLists */
	const int nb = 800;
	// printf("ComputeTabulated\n");
	
	if (get_energy) {
		clearEnergies<<< nb, numThreads >>>(energies_d,num);
		gpuErrchk(hipDeviceSynchronize());
		computeTabulatedEnergyKernel<<< nb, numThreads >>>(force, pos, type,
						tablePot_d, tableBond_d, sys_d,
						bonds, bondMap,	numBonds,	energies_d,	cutoff2,
						numPairs_d, pairLists_d, pairTabPotType_d);
	} else {
		computeTabulatedKernel<<< nb, numThreads >>>(force, pos, type,
						tablePot_d, tableBond_d, sys_d,
						bonds, bondMap,	numBonds, cutoff2,
						numPairs_d, pairLists_d, pairTabPotType_d);
	}
	/* printPairForceCounter<<<1,32>>>(); */
	/* gpuErrchk(hipDeviceSynchronize()); */

	computeAngles<<<numBlocks, numThreads>>>(force, pos, angles,
			tableAngle_d, numAngles, num, sys_d, energies_d, get_energy);

	computeDihedrals<<<numBlocks, numThreads>>>(force, pos, dihedrals,
			tableDihedral_d, numDihedrals, num, sys_d, energies_d, get_energy);

	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}

float ComputeForce::computeTabulatedFull(Vector3* force, Vector3* pos, int* type,
		Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap,
		Angle* angles, Dihedral* dihedrals, bool get_energy) {
	energy = 0.0f;

	gridSize = (num * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeTabulatedFullKernel<<< numBlocks, numThreads >>>(force, pos, type,
			tablePot_d, tableBond_d, num, numParts, sys_d, bonds, bondMap, numBonds,
			excludes, excludeMap, numExcludes, energies_d, gridSize, numReplicas,
			get_energy, angles);
	gpuErrchk(hipDeviceSynchronize());

	computeAngles<<< numBlocks, numThreads >>>(force, pos, angles, tableAngle_d,
																						 numAngles, num, sys_d, energies_d,
																						 get_energy);
	gpuErrchk(hipDeviceSynchronize());
	computeDihedrals<<< numBlocks, numThreads >>>(force, pos, dihedrals,
																							  tableDihedral_d, numDihedrals,
																								num, sys_d, energies_d,
																								get_energy);
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}
