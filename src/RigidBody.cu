#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include "RigidBody.h"
#include "RigidBodyType.h"
#include "RigidBodyController.h"
#include "Configuration.h"
#include "ComputeGridGrid.cuh"

#include "Debug.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), __FILE__, line);
      if (abort) exit(code);
   }
}


RigidBody::RigidBody(String name, const Configuration& cref, const RigidBodyType& tref, RigidBodyController* RBCref) 
    : name(name), c(&cref), t(&tref), RBC(RBCref), impulse_to_momentum(4.1867999435271e4) /*impulse_to_momentum(4.184e8f)*/ { init(); }
RigidBody::RigidBody(const RigidBody& rb)
    : name(rb.name), c(rb.c), t(rb.t), RBC(rb.RBC), impulse_to_momentum(4.1867999435271e4)/*impulse_to_momentum(4.184e8f)*/ { init(); }
void RigidBody::init() {
	// units "(kcal_mol/AA) * ns" "dalton AA/ns" * 4.184e+08	
	timestep = c->timestep;
	Temp = c->temperature * 0.0019872065f;
	// RBTODO: use temperature grids
	// tempgrid = c->temperatureGrid;
	position = t->initPos;

	// Orientation matrix that brings vector from the RB frame to the lab frame
	orientation = t->initRot;

        momentum = t->initMomentum;
        angularMomentum = t->initAngularMomentum;

	// Memory allocation for forces between particles and grids 
	const int& numGrids = t->numPotGrids;
	numParticles = new int[numGrids];
	particles_d = new int*[numGrids];
	particleForceStreams = new const hipStream_t*[numGrids];

	for (int i = 0; i < numGrids; ++i) {
	    numParticles[i] = -1;
		const int& n = t->numParticles[i];
		if (n > 0) {
		    // gpuErrchk(hipMalloc( &particles_d[i], 0.5*sizeof(int)*n )); // not sure why 0.5 was here; prolly bug
		        gpuErrchk(hipMalloc( &particles_d[i], sizeof(int)*n )); // TODO: dynamically allocate memory as needed
		}
	}
}

GPUManager RigidBody::gpuman = GPUManager();

//Boltzmann distribution
void RigidBody::Boltzmann()
{

    Vector3 rando = getRandomGaussVector();
    momentum = sqrt(t->mass*Temp) * 2.046167135 * rando;
    rando = getRandomGaussVector();
    angularMomentum.x = sqrt(t->inertia.x*Temp) * 2.046167135 * rando.x;
    angularMomentum.y = sqrt(t->inertia.y*Temp) * 2.046167135 * rando.y;
    angularMomentum.z = sqrt(t->inertia.z*Temp) * 2.046167135 * rando.z;

    printf("%f\n", Temperature());
}

RigidBody::~RigidBody() {
	const int& numGrids = t->numPotGrids;
	for (int i = 0; i < numGrids; ++i) {
		const int& n = t->numParticles[i];
		if (n > 0) {
			gpuErrchk(hipFree( particles_d[i] ));
		}
	}
	if (numParticles != NULL) {
		delete[] numParticles;
		delete[] particles_d;
		delete[] particleForceStreams;
	}
}

int RigidBody::appendNumParticleBlocks( std::vector<int>* blocks ) {
    int ret = 0;
    const int& numGrids = t->numPotGrids;
    for (int i = 0; i < numGrids; ++i) {
	numParticles[i] = -1;
	const int& n = t->numParticles[i];
	const int nb = (n/NUMTHREADS)+1; // max number of blocks
	if (n > 0) {
	    blocks->push_back(nb);
	    ret += nb;
	}
    }
    return ret;
}

void RigidBody::addForce(Force f) { 
	force += f; 
} 
void RigidBody::addTorque(Force torq) {
	torque += torq; 
}

void RigidBody::updateParticleList(Vector3* pos_d) {
	for (int i = 0; i < t->numPotGrids; ++i) {
		numParticles[i] = 0;
		int& tnp = t->numParticles[i];
		if (tnp > 0) {
			Vector3 gridCenter = t->potentialGrids[i].getCenter();
			float cutoff = gridCenter.length();
			cutoff += t->potentialGrids[i].getRadius();
			cutoff += c->pairlistDistance; 
		   
			int* tmp_d;
			gpuErrchk(hipMalloc( &tmp_d, sizeof(int) ));
			gpuErrchk(hipMemcpy( tmp_d, &numParticles[i], sizeof(int), hipMemcpyHostToDevice ));

			int nb = floor(tnp/NUMTHREADS) + 1;
#if __CUDA_ARCH__ >= 300
			createPartlist<<<nb,NUMTHREADS>>>(pos_d, tnp, t->particles_d[i],
							tmp_d, particles_d[i],
							gridCenter + position, cutoff*cutoff);
#else
			createPartlist<<<nb,NUMTHREADS,NUMTHREADS/WARPSIZE>>>(pos_d, tnp, t->particles_d[i],
							tmp_d, particles_d[i],
							gridCenter + position, cutoff*cutoff);
#endif			
			gpuErrchk(hipMemcpy(&numParticles[i], tmp_d, sizeof(int), hipMemcpyDeviceToHost ));
			gpuErrchk(hipFree( tmp_d ));
		}
	}
}
void RigidBody::callGridParticleForceKernel(Vector3* pos_d, Vector3* force_d, Vector3* forcestorques_d, const std::vector<int>& forcestorques_offset, int& fto_idx) {
	// Apply the force and torque on the rigid body, and forces on particles
	
	// RBTODO: performance: consolidate CUDA stream management
	// loop over potential grids 
	for (int i = 0; i < t->numPotGrids; ++i) {
		if (numParticles[i] <= 0) continue;
		// const int nb = 500;
		/*
		  r: postion of particle in real space
		  B: grid Basis
		  o: grid origin
		  R: rigid body orientation
		  c: rigid body center

		  B': R.B 
		  c': R.o + c
		*/

		const hipStream_t& stream = gpuman.get_next_stream();
		particleForceStreams[i] = &stream;

		Vector3 c =  getOrientation()*t->potentialGrids[i].getOrigin() + getPosition();
		Matrix3 B = (getOrientation()*t->potentialGrids[i].getBasis()).inverse();
		
		// RBTODO: get energy
		const int nb = (numParticles[i]/NUMTHREADS)+1;		
		computePartGridForce<<< nb, NUMTHREADS, NUMTHREADS*2*sizeof(Vector3), stream >>>(
			pos_d, force_d, numParticles[i], particles_d[i],
			t->rawPotentialGrids_d[i],
			B, c, forcestorques_d+forcestorques_offset[fto_idx++]);
	}
}

void RigidBody::applyGridParticleForces(Vector3* forcestorques, const std::vector<int>& forcestorques_offset, int& fto_idx) {
	// loop over potential grids 
	for (int i = 0; i < t->numPotGrids; ++i) {
		if (numParticles[i] <= 0) continue;
		const int nb = (numParticles[i]/NUMTHREADS)+1;
		Vector3 c =  getOrientation()*t->potentialGrids[i].getOrigin() + getPosition();

		// Sum and apply forces and torques
		Vector3 f = Vector3(0.0f);
		Vector3 torq = Vector3(0.0f);
		for (int k = 0; k < nb; ++k) {
		    int j = forcestorques_offset[fto_idx]+2*k;
			f = f + forcestorques[j];
			torq = torq + forcestorques[j+1];
		}
		++fto_idx;

		torq = -torq + (getPosition()-c).cross( f ); 
		addForce( -f );
		addTorque( torq );
	}
}

	/*===========================================================================\
	| Following "Algorithm for rigid-body Brownian dynamics" Dan Gordon, Matthew |
	|   Hoyles, and Shin-Ho Chung                                                |
	|   http://langevin.anu.edu.au/publications/PhysRevE_80_066703.pdf           |
	|                                                                            |
	|                                                                            |
	| BUT: assume diagonal friction tensor and no Wiener process / stochastic    |
	|   calculus then this is just the same as for translation                   |
	|                                                                            |
	|   < T_i(t) T_i(t) > = 2 kT friction inertia                                |
	|                                                                            |
	|   friction / kt = Diff                                                     |
	\===========================================================================*/
void RigidBody::addLangevin(Vector3 w1, Vector3 w2) 
{
    Vector3 transForceCoeff = Vector3::element_sqrt( 2. * Temp * t->mass*t->transDamping / timestep );
    Vector3  rotTorqueCoeff = Vector3::element_sqrt( 2. * Temp * Vector3::element_mult(t->inertia,t->rotDamping) / timestep );

    Force f = Vector3::element_mult(transForceCoeff,w1) * 2.046167337 -
              Vector3::element_mult(t->transDamping, orientation.transpose()*momentum) * 41867.999435; 
    
    Force torq = Vector3::element_mult(rotTorqueCoeff,w2) * 2.046167337 -
                 Vector3::element_mult(t->rotDamping, angularMomentum) * 41867.999435;

    f = orientation * f;
    torq = orientation * torq;

    addForce(f);
    addTorque(torq);
}

  /*==========================================================================\
	| from: Dullweber, Leimkuhler, Maclachlan. Symplectic splitting methods for |
	| rigid body molecular dynamics. JCP 107. (1997)                            |
	| http://jcp.aip.org/resource/1/jcpsa6/v107/i15/p5840_s1                    |
	\==========================================================================*/
void RigidBody::integrateDLM(int startFinishAll) 
{
    Vector3 trans; // = *p_trans;
    //Matrix3 rot = Matrix3(1); // = *p_rot;
    if ( isnan(force.x) || isnan(torque.x) ) 
    {   
        // NaN check
        printf("Rigid Body force or torque was NaN!\n");
        exit(-1);
    }

    if (startFinishAll == 0 || startFinishAll == 2) 
    {
        // propogate momenta by half step
        momentum += 0.5f * timestep * force * impulse_to_momentum;
        angularMomentum += 0.5f * timestep * (orientation.transpose()*torque) * impulse_to_momentum;
    } 
    else if (startFinishAll == 1)
    {
        position += timestep * momentum / t->mass * 1e4; // update CoM a full timestep
        // update orientations a full timestep
        Matrix3 R; // represents a rotation about a principle axis
        R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x * 1e4); // R1
        applyRotation(R);

        R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y * 1e4); // R2
        applyRotation(R);
                        
        R = Rz(    timestep * angularMomentum.z / t->inertia.z * 1e4); // R3
        applyRotation(R);
                        
        R = Ry(0.5*timestep * angularMomentum.y / t->inertia.y * 1e4); // R4
        applyRotation(R);

        R = Rx(0.5*timestep * angularMomentum.x / t->inertia.x * 1e4); // R5
        applyRotation(R);               
        // TODO make this periodic
        // printf("det: %.12f\n", orientation.det());
        orientation = orientation.normalized();
        // orientation = orientation/orientation.det();
        // printf("det2: %.12f\n", orientation.det());
        // orientation = orientation/orientation.det(); // TODO: see if this can be somehow eliminated (wasn't in original DLM algorithm...)
    }
}
/* Following:
Brownian Dynamics Simulation of Rigid Particles of Arbitrary Shape in External Fields
Miguel X. Fernandes, José García de la Torre
*/

//Chris original implementation for Brownian motion
void RigidBody::integrate(int startFinishAll)
{
    //if (startFinishAll == 1) return;

    Matrix3 rot = Matrix3(1); // = *p_rot;

    if ( isnan(force.x) || isnan(torque.x) ) 
    {
        printf("Rigid Body force or torque was NaN!\n");
        exit(-1);
    }
    //float Temp = 1;
    Vector3 diffusion    = Temp / (t->transDamping*t->mass); // TODO: assign diffusion in config file, or elsewhere
    //Vector3 diffusion    = Temp / (t->transDamping*t->mass);
    Vector3 rotDiffusion = Temp / (Vector3::element_mult(t->rotDamping,t->inertia));

    Vector3 rando  = getRandomGaussVector();
    Vector3 offset = Vector3::element_mult( (diffusion / Temp), force ) * timestep  * 418.679994353 +
                     Vector3::element_mult( Vector3::element_sqrt( 2.0f * diffusion * timestep * 418.679994353), rando) ;

    position += offset;

    rando = getRandomGaussVector();
    Vector3 rotationOffset = Vector3::element_mult( (rotDiffusion / Temp) , orientation.transpose() * torque * timestep) * 418.679994353 +
                             Vector3::element_mult( Vector3::element_sqrt( 2.0f * rotDiffusion * timestep * 418.679994353), rando );

    // Consider whether a DLM-like decomposition of rotations is needed for time-reversibility
    orientation = orientation * (Rz(rotationOffset.z * 0.5) * Ry(rotationOffset.y * 0.5) * Rx(rotationOffset.x)
                              *  Ry(rotationOffset.y * 0.5) * Rz(rotationOffset.z * 0.5));
    //orientation = orientation * Rz(rotationOffset.z) * Ry(rotationOffset.y) * Rx(rotationOffset.x);
    orientation = orientation.normalized();
}
 
float RigidBody::Temperature()
{
    return (momentum.length2() / t->mass + 
            angularMomentum.x * angularMomentum.x / t->inertia.x + 
            angularMomentum.y * angularMomentum.y / t->inertia.y + 
            angularMomentum.z * angularMomentum.z / t->inertia.z) * 0.50;
}

void RigidBody::applyRotation(const Matrix3& R) {
	angularMomentum = R * angularMomentum;
	// According to DLM, but rotations work the wrong way; I think DLM update is wrong
	// orientation = orientation * R.transpose(); 

	// This makes sense: apply a rotation in the body frame followed by a transformation from body to lab frame
	// Also works in statistical test
	// Consistent with www.archer.ac.uk/documentation/white-papers/lammps-elba/lammps-ecse.pdf
	orientation = orientation * R; 
        orientation.normalized();	
}

// Rotations about axes
// for very small angles 10^-8, cos^2+sin^2 != 1 
// concerned about the accumulation of errors in non-unitary transformations!
Matrix3 RigidBody::Rx(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		1.0f, 0.0f, 0.0f,
		0.0f,  cos, -sin,
		0.0f,  sin,  cos);
}
Matrix3 RigidBody::Ry(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		cos,  0.0f,  sin,
		0.0f, 1.0f, 0.0f,
		-sin, 0.0f,  cos);
}
Matrix3 RigidBody::Rz(BigReal t) {
	BigReal qt = 0.25*t*t;  // for approximate calculations of sin(t) and cos(t)
	BigReal cos = (1-qt)/(1+qt);
	BigReal sin = t/(1+qt);

	return Matrix3(
		cos,  -sin, 0.0f,
		sin,   cos, 0.0f,
		0.0f, 0.0f, 1.0f);
}
Matrix3 RigidBody::eulerToMatrix(const Vector3 e) {
	// convert euler angle input to rotation matrix
	// http://en.wikipedia.org/wiki/Rotation_formalisms_in_three_dimensions#Conversion_formulae_between_formalisms
	return Rz(e.z) * Ry(e.y) * Rx(e.x);
}
