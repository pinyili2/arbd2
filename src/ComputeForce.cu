#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////
// Brownian dynamics base class
// Author: Jeff Comer <jcomer2@illinois.edu>

#include "ComputeForce.h"
#include "ComputeForce.cuh"
#include "Configuration.h"
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>

#ifndef gpuErrchk
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#endif 

#define gpuKernelCheck() {kernelCheck( __FILE__, __LINE__); }
inline void kernelCheck(const char* file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::fprintf(stderr,"Error: %s in %s %d\n", hipGetErrorString(err),file, line);
        assert(1==2);
    }
    //gpuErrchk(hipDeviceSynchronize());
}

hipEvent_t start, stop;

GPUManager ComputeForce::gpuman = GPUManager();

void runSort(int2 *d1, int *d2, float *key,
				int2 *scratch1, int  *scratch2, float *scratchKey,
				unsigned int count);

ComputeForce::ComputeForce(const Configuration& c, const int numReplicas = 1) :
    num(c.num), numParts(c.numParts), num_rb_attached_particles(c.num_rb_attached_particles),
    sys(c.sys), switchStart(c.switchStart),
    switchLen(c.switchLen), electricConst(c.coulombConst),
    cutoff2((c.switchLen + c.switchStart) * (c.switchLen + c.switchStart)),
    decomp(c.sys->getBox(), c.sys->getOrigin(), c.switchStart + c.switchLen + c.pairlistDistance, numReplicas),
    numBonds(c.numBonds), numTabBondFiles(c.numTabBondFiles),
    numExcludes(c.numExcludes), numAngles(c.numAngles),
    numTabAngleFiles(c.numTabAngleFiles), numDihedrals(c.numDihedrals),
    numTabDihedralFiles(c.numTabDihedralFiles), numRestraints(c.numRestraints),
    numGroupSites(c.numGroupSites),
    numReplicas(numReplicas) {

	// Grow vectors for per-gpu device pointers
	for (int i = 0; i < gpuman.gpus.size(); ++i) {
	    int s = gpuman.gpus.size();
	    sys_d	= std::vector<BaseGrid*>(s);
	    tablePot_addr = std::vector<TabulatedPotential**>(s);
	    tablePot_d	= std::vector<TabulatedPotential**>(s);
	    pairLists_d = std::vector<int2*>(s);
	    pairLists_tex = std::vector<hipTextureObject_t>(s);
	    pairTabPotType_d = std::vector<int*>(s);
	    pairTabPotType_tex = std::vector<hipTextureObject_t>(s);
	    numPairs_d = std::vector<int*>(s);
	    pos_d = std::vector<Vector3*>(s);
	    pos_tex = std::vector<hipTextureObject_t>(s);
	    forceInternal_d = std::vector<Vector3*>(s);
	}

	// Allocate the parameter tables.
	decomp_d = NULL;

	pairlistdist2 = (sqrt(cutoff2) + c.pairlistDistance);
	pairlistdist2 *= pairlistdist2;

	int np2     = numParts*numParts;
	tableEps    = new float[np2];
	tableRad6   = new float[np2];
	tableAlpha  = new float[np2];

	const size_t tableSize = sizeof(float) * np2;
	gpuErrchk(hipMalloc(&tableEps_d, tableSize));
	gpuErrchk(hipMalloc(&tableRad6_d, tableSize));
	gpuErrchk(hipMalloc(&tableAlpha_d, tableSize));
	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    gpuman.use(i);
	    gpuErrchk(hipMalloc(&sys_d[i], sizeof(BaseGrid)));
	    gpuErrchk(hipMemcpyAsync(sys_d[i], sys, sizeof(BaseGrid), hipMemcpyHostToDevice));
	}
	gpuman.use(0);

	// Build the parameter tables.
	makeTables(c.part);

	gpuErrchk(hipMemcpyAsync(tableAlpha_d, tableAlpha, tableSize, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableEps_d, tableEps, tableSize, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyAsync(tableRad6_d, tableRad6, tableSize, hipMemcpyHostToDevice));

	// Create the potential table
	tablePot = new TabulatedPotential*[np2];
	for (int i = 0; i < np2; ++i) tablePot[i] = NULL;

	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    tablePot_addr[i] = new TabulatedPotential*[np2];
	    for (int j = 0; j < np2; ++j) tablePot_addr[i][j] = NULL;
	    gpuman.use(i);
	    gpuErrchk(hipMalloc(&tablePot_d[i], sizeof(TabulatedPotential*) * np2));
	}
	gpuman.use(0);

	// Create the bond table
	tableBond = new TabulatedPotential*[numTabBondFiles];
	tableBond_addr = new TabulatedPotential*[numTabBondFiles];
	bondList_d = NULL;
	tableBond_d = NULL;
	for (int i = 0; i < numTabBondFiles; i++) {
		tableBond_addr[i] = NULL;
		tableBond[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableBond_d, sizeof(TabulatedPotential*) * numTabBondFiles));

	// Create the angle table
	tableAngle = new TabulatedAnglePotential*[numTabAngleFiles];
	tableAngle_addr = new TabulatedAnglePotential*[numTabAngleFiles];
	angleList_d = NULL;
	tableAngle_d = NULL;
	for (int i = 0; i < numTabAngleFiles; i++) {
		tableAngle_addr[i] = NULL;
		tableAngle[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableAngle_d, sizeof(TabulatedAnglePotential*) * numTabAngleFiles));

	// Create the dihedral table
	tableDihedral = new TabulatedDihedralPotential*[numTabDihedralFiles];
	tableDihedral_addr = new TabulatedDihedralPotential*[numTabDihedralFiles];
	dihedralList_d = NULL;
	tableDihedral_d = NULL;
	for (int i = 0; i < numTabDihedralFiles; i++) {
		tableDihedral_addr[i] = NULL;
		tableDihedral[i] = NULL;
	}
	gpuErrchk(hipMalloc(&tableDihedral_d, sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles));

	{	// allocate device for pairlists
		// RBTODO: select maxpairs in better way; add assertion in kernel to avoid going past this
		const int maxPairs = 1<<25;
		for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
		    gpuman.use(i);
		    gpuErrchk(hipMalloc(&numPairs_d[i],       sizeof(int)));
		    gpuErrchk(hipMalloc(&pairLists_d[i],      sizeof(int2)*maxPairs));
		    // gpuErrchk(hipBindTexture(0, pairListsTex, pairLists_d[i], sizeof(int2)*maxPairs)); //Han-Yi
		    gpuErrchk(hipMalloc(&pairTabPotType_d[i], sizeof(int)*maxPairs));
		}

		// create texture object
		for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
		    gpuman.use(i);
		    hipResourceDesc resDesc;
		    memset(&resDesc, 0, sizeof(resDesc));
		    resDesc.resType = hipResourceTypeLinear;
		    resDesc.res.linear.devPtr = pairLists_d[i];
		    resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
		    resDesc.res.linear.desc.x = 32; // bits per channel
		    resDesc.res.linear.desc.y = 32; // bits per channel
		    resDesc.res.linear.sizeInBytes = maxPairs*sizeof(int2);

		    hipTextureDesc texDesc;
		    memset(&texDesc, 0, sizeof(texDesc));
		    texDesc.readMode = hipReadModeElementType;

		    // create texture object: we only have to do this once!
		    pairLists_tex[i]=0;
		    hipCreateTextureObject(&pairLists_tex[i], &resDesc, &texDesc, NULL);
		}

		// create texture object
		for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
		    gpuman.use(i);
		    hipResourceDesc resDesc;
		    memset(&resDesc, 0, sizeof(resDesc));
		    resDesc.resType = hipResourceTypeLinear;
		    resDesc.res.linear.devPtr = pairTabPotType_d[i];
		    resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
		    resDesc.res.linear.desc.x = 32; // bits per channel
		    resDesc.res.linear.sizeInBytes = maxPairs*sizeof(int);

		    hipTextureDesc texDesc;
		    memset(&texDesc, 0, sizeof(texDesc));
		    texDesc.readMode = hipReadModeElementType;

		    // create texture object: we only have to do this once!
		    pairTabPotType_tex[i] = 0;
		    hipCreateTextureObject(&pairTabPotType_tex[i], &resDesc, &texDesc, NULL);

		}
		gpuman.use(0);


                //Han-Yi Chou
                int nCells = decomp.nCells.x * decomp.nCells.y * decomp.nCells.z;
                //int* nCells_dev;
		if (nCells < MAX_CELLS_FOR_CELLNEIGHBORLIST) {
		    int3 *Cells_dev;
		    size_t sz = 27*nCells*sizeof(int);
		    gpuErrchk(hipMalloc(&CellNeighborsList, sz));
		    //gpuErrchk(hipMalloc(&nCells_dev,sizeof(int)));
		    gpuErrchk(hipMalloc(&Cells_dev,sizeof(int3)));
		    //gpuErrchk(hipMemcpy(nCells_dev,&nCells,1,hipMemcpyHostToDevice);
		    gpuErrchk(hipMemcpy(Cells_dev,&(decomp.nCells),sizeof(int3),hipMemcpyHostToDevice));
		    createNeighborsList<<<256,256>>>(Cells_dev,CellNeighborsList);
		    gpuErrchk(hipFree(Cells_dev));

		    // create texture object
		    {
			hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeLinear;
			resDesc.res.linear.devPtr = CellNeighborsList;
			resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
			resDesc.res.linear.desc.x = 32; // bits per channel
			resDesc.res.linear.sizeInBytes = sz;

			hipTextureDesc texDesc;
			memset(&texDesc, 0, sizeof(texDesc));
			texDesc.readMode = hipReadModeElementType;

			// create texture object: we only have to do this once!
			neighbors_tex=0;
			hipCreateTextureObject(&neighbors_tex, &resDesc, &texDesc, NULL);
		    }
		}
	}
	
	restraintIds_d = NULL;

	//Calculate the number of blocks the grid should contain
	gridSize =  (num+num_rb_attached_particles) / NUM_THREADS + 1;

	// Create and allocate the energy arrays
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * (num+num_rb_attached_particles+numGroupSites) * numReplicas));
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

ComputeForce::~ComputeForce() {
	delete[] tableEps;
	delete[] tableRad6;
	delete[] tableAlpha;
	gpuErrchk(hipFree(tableEps_d));
	gpuErrchk(hipFree(tableAlpha_d));
	gpuErrchk(hipFree(tableRad6_d));
	
	for (int i = 0; i < numParts; ++i) {
	    for (int j = i; j < numParts; ++j) {
		int ind = i+j*numParts;
		if (tablePot[ind] != NULL) {
		    for (std::size_t g = 0; g < gpuman.gpus.size(); ++g) {
			gpuman.use(g);
			tablePot_addr[g][ind]->free_from_cuda(tablePot_addr[g][ind]);
		    }
		    delete tablePot[ind];
		}
	    }
	}
	delete[] tablePot;
	for (auto& tpa : tablePot_addr) delete[] tpa;

	for (int j = 0; j < numTabBondFiles; ++j)
		delete tableBond[j];
	delete[] tableBond;
	delete[] tableBond_addr;
	gpuErrchk(hipFree(tableBond_d));

	for (int j = 0; j < numTabAngleFiles; ++j)
		if (tableAngle[j] != NULL)
			delete tableAngle[j];
	delete[] tableAngle;
	delete[] tableAngle_addr;

	if(type_d != NULL)
	{
		gpuErrchk(hipFree(tableAngle_d));

		gpuErrchk(hipFree(energies_d));

		gpuErrchk( hipFree(type_d) );
		if (numBonds > 0) {
			gpuErrchk( hipFree(bonds_d) );
			gpuErrchk( hipFree(bondMap_d) );
			gpuErrchk( hipFree(bondList_d) );
		}
		if (numAngles > 0) {
			gpuErrchk( hipFree(angles_d) );
			gpuErrchk( hipFree(angleList_d) );
		}
		if (numDihedrals > 0) {
			gpuErrchk( hipFree(dihedrals_d) );
			gpuErrchk( hipFree(dihedralList_d) );
			gpuErrchk( hipFree(dihedralPotList_d) );
		}
		if (numExcludes > 0) {
			gpuErrchk( hipFree(excludes_d) );
			gpuErrchk( hipFree(excludeMap_d) );
		}
		if (numRestraints > 0) {
			gpuErrchk( hipFree(restraintIds_d) );
			gpuErrchk( hipFree(restraintLocs_d) );
			gpuErrchk( hipFree(restraintSprings_d) );
		}
	}

	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    gpuErrchk(hipFree(forceInternal_d[i]) );
	    gpuErrchk(hipFree(sys_d[i]));
	    gpuErrchk(hipDestroyTextureObject(pos_tex[i]));
	    gpuErrchk(hipFree(pos_d[i]) );
	    gpuErrchk(hipFree(numPairs_d[i]));
	    gpuErrchk(hipDestroyTextureObject(pairLists_tex[i]));
	    gpuErrchk(hipFree(pairLists_d[i]));
	    gpuErrchk(hipDestroyTextureObject(pairTabPotType_tex[i]));
	    gpuErrchk(hipFree(pairTabPotType_d[i]));
	}
        gpuErrchk(hipDestroyTextureObject(neighbors_tex));
        gpuErrchk(hipFree( CellNeighborsList));

}

void ComputeForce::updateNumber(int newNum) {
	if (newNum == num or newNum < 0) return;

	// Set the new number.
	num = newNum;

	// Reallocate the neighbor list.
	//delete[] neigh;
	//neigh = new IndexList[num];
	decompose();

	printf("updateNumber() called\n");
	// Reallocate CUDA arrays

	// Recalculate the number of blocks in the grid
	gridSize = 0;
	while ((int)sqrt(NUM_THREADS) * gridSize < num+num_rb_attached_particles)
		++gridSize;

	gpuErrchk(hipFree(energies_d));
	gpuErrchk(hipMalloc(&energies_d, sizeof(float) * gridSize));
}

void ComputeForce::makeTables(const BrownianParticleType part[]) {
	for (int i = 0; i < numParts; ++i) {
		const BrownianParticleType& pi = part[i];
		for (int j = 0; j < numParts; ++j) {
			const BrownianParticleType& pj = part[j];
			int ind = i * numParts + j;
			tableEps[ind] = sqrtf(pi.eps * pj.eps);
			float r = pi.radius + pj.radius;
			tableRad6[ind] = r * r * r * r * r * r;
			tableAlpha[ind] = electricConst * pi.charge * pj.charge;
		}
	}
}

bool ComputeForce::addTabulatedPotential(String fileName, int type0, int type1) {
	if (type0 < 0 or type0 >= numParts) return false;
	if (type1 < 0 or type1 >= numParts) return false;

	int ind = type0 + type1 * numParts;
	int ind1 = type1 + type0 * numParts;

	// If an entry already exists for this particle type, delete it
	if (tablePot[ind] != NULL) {
	    for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
		gpuman.use(i);
		tablePot_addr[i][ind]->free_from_cuda(tablePot_addr[i][ind]);
		delete tablePot_addr[i][ind];
	    }
	    gpuman.use(0);
	    delete tablePot[ind];
	}
	// if (tablePot[ind1] != NULL) {
	//     // gpuErrchk(hipFree(tablePot_addr[ind1]));
	// 	delete tablePot[ind1];
	// 	// tablePot[ind1] = NULL;
	// 	// tablePot_addr[ind1] = NULL;
	// }

	tablePot[ind] = tablePot[ind1] = new TabulatedPotential(fileName);
	tablePot[ind]->truncate(switchStart, sqrtf(cutoff2), 0.0f);

	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    gpuman.use(i);
	    tablePot_addr[i][ind] = tablePot_addr[i][ind1] = tablePot[ind]->copy_to_cuda();
	    gpuErrchk(hipMemcpy(tablePot_d[i], tablePot_addr[i],
				 sizeof(TabulatedPotential*) * numParts * numParts, hipMemcpyHostToDevice));
	}
	gpuman.use(0);
	return true;
}

bool ComputeForce::addBondPotential(String fileName, int ind, Bond bonds[])
{
    // TODO: see if tableBond_addr can be removed
    if (tableBond[ind] != NULL) {
	delete tableBond[ind];
	// gpuErrchk(hipFree(tableBond_addr[ind])); //TODO free this a little more cleanly
    }

    tableBond[ind] = new TabulatedPotential(fileName);

	for (int i = 0; i < numBonds; ++i)
		if (bonds[i].fileName == fileName)
			bonds[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(bonds_d, bonds, sizeof(Bond) * numBonds, hipMemcpyHostToDevice));

	tableBond_addr[ind] = tableBond[ind]->copy_to_cuda();
	gpuErrchk(hipMemcpy(tableBond_d, tableBond_addr,
			     sizeof(TabulatedPotential*) * numTabBondFiles, hipMemcpyHostToDevice));
	return true;
}

bool ComputeForce::addAnglePotential(String fileName, int ind, Angle* angles) {
	if (tableAngle[ind] != NULL) {
		delete tableAngle[ind];
		gpuErrchk(hipFree(tableAngle_addr[ind]));
		tableAngle[ind] = NULL;
		tableAngle_addr[ind] = NULL;
	}

	tableAngle[ind] = new TabulatedAnglePotential(fileName);
	TabulatedAnglePotential *t = new TabulatedAnglePotential(*tableAngle[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableAngle[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableAngle[ind]->pot, sizeof(float) * size, hipMemcpyHostToDevice));
	t->pot = pot;
	gpuErrchk(hipMalloc(&tableAngle_addr[ind], sizeof(TabulatedAnglePotential)));
	gpuErrchk(hipMemcpy(tableAngle_addr[ind], t, sizeof(TabulatedAnglePotential), hipMemcpyHostToDevice));
	t->pot = NULL;
	delete t;

	gpuErrchk(hipMemcpyAsync(tableAngle_d, tableAngle_addr,
			sizeof(TabulatedAnglePotential*) * numTabAngleFiles, hipMemcpyHostToDevice));

	for (int i = 0; i < numAngles; i++)
		if (angles[i].fileName == fileName)
			angles[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpy(angles_d, angles, sizeof(Angle) * numAngles,
			hipMemcpyHostToDevice));
	return true;
}

bool ComputeForce::addDihedralPotential(String fileName, int ind, Dihedral dihedrals[])
{
	for (int i = 0; i < numDihedrals; i++)
		if (dihedrals[i].fileName == fileName)
			dihedrals[i].tabFileIndex = ind;

	gpuErrchk(hipMemcpyAsync(dihedrals_d, dihedrals, sizeof(Dihedral) * numDihedrals,
			hipMemcpyHostToDevice));

	if (tableDihedral[ind] != NULL) {
		delete tableDihedral[ind];
		gpuErrchk(hipFree(tableDihedral_addr[ind]));
		tableDihedral[ind] = NULL;
		tableDihedral_addr[ind] = NULL;
	}

	tableDihedral[ind] = new TabulatedDihedralPotential(fileName);
	TabulatedDihedralPotential t = TabulatedDihedralPotential(*tableDihedral[ind]);

	// Copy tableAngle[ind] to the device
	float *pot;
	int size = tableDihedral[ind]->size;
	gpuErrchk(hipMalloc(&pot, sizeof(float) * size));
	gpuErrchk(hipMemcpyAsync(pot, tableDihedral[ind]->pot,
			sizeof(float) * size, hipMemcpyHostToDevice));
	t.pot = pot;

	gpuErrchk(hipMalloc(&tableDihedral_addr[ind], sizeof(TabulatedDihedralPotential)));
	gpuErrchk(hipMemcpyAsync(tableDihedral_addr[ind], &t,
			sizeof(TabulatedDihedralPotential), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(tableDihedral_d, tableDihedral_addr,
			sizeof(TabulatedDihedralPotential*) * numTabDihedralFiles, hipMemcpyHostToDevice));
	t.pot = NULL;
	return true;
}

void ComputeForce::decompose() {
	//gpuErrchk( hipProfilerStart() );

	// Reset the cell decomposition.
	if (decomp_d != NULL)
        {
            hipFree(decomp_d);
            decomp_d = NULL;
	}	
	decomp.decompose_d(pos_d[0], num+num_rb_attached_particles);
	decomp_d = decomp.copyToCUDA();

	// Update pairlists using cell decomposition (not sure this is really needed or good) 
	//RBTODO updatePairlists<<< nBlocks, NUM_THREADS >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d);	

	/* size_t free, total; */
	/* { */
	/* 	hipMemGetInfo(&free,&total); */
	/* 	printf("Free memory: %zu / %zu\n", free, total); */
	/* } */
	
	// initializePairlistArrays
	int nCells = decomp.nCells.x * decomp.nCells.y * decomp.nCells.z;

	// int blocksPerCell = 10;

	
	/* hipMemGetInfo(&free,&total); */
	/* printf("Free memory: %zu / %zu\n", free, total); */
	
	// const int NUMTHREADS = 128;
	//const size_t nBlocks = (num * numReplicas) / NUM_THREADS + 1;
	// const size_t nBlocks = nCells*blocksPerCell;

	/* clearPairlists<<< 1, 32 >>>(pos, num, numReplicas, sys_d[0], decomp_d); */
	/* gpuErrchk(hipDeviceSynchronize()); */
	/* pairlistTest<<< nBlocks, NUMTHREADS >>>(pos, num, numReplicas, */
	/* 																					 sys_d[0], decomp_d, nCells, blocksPerCell, */
	/* 																					 numPairs_d[0], pairListListI_d, pairListListJ_d); */
	/* gpuErrchk(hipDeviceSynchronize());	 */

	int tmp = 0;
	gpuErrchk(hipMemcpyAsync(numPairs_d[0], &tmp,	sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());
	// printf("Pairlistdist: %f\n",sqrt(pairlistdist2));

#ifdef DEBUGEXCLUSIONS
	initExSum();
	gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
#endif
    //Han-Yi Chou bind texture
    //printf("%d\n", sizeof(Vector3));
    //gpuErrchk(hipBindTexture(0,  PosTex, pos_d[0],sizeof(Vector3)*num*numReplicas));
    //gpuErrchk(hipBindTexture(0,CellsTex, decomp_d->getCells_d(),sizeof(CellDecomposition::cell_t)*num*numReplicas));
   
//#if __CUDA_ARCH__ >= 300
	//createPairlists_debug<<< 2048, 64 >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2);
    //#ifdef NEW
   //for sm52
    //createPairlists<32,64,1><<< dim3(256,128,numReplicas),dim3(32,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], 
      //GTX 980
      //Han-Yi Chou 2017 my code
      
      #if __CUDA_ARCH__ >= 520
      createPairlists<64,64,8><<<dim3(128,128,numReplicas),dim3(64,1,1)>>>(pos_d[0], num+num_rb_attached_particles, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
                                                                             pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d,
									   excludeMap_d, numExcludes, pairlistdist2, pos_tex[0], neighbors_tex);
      #else //__CUDA_ARCH__ == 300
      createPairlists<64,64,8><<<dim3(256,256,numReplicas),dim3(64,1,1)>>>(pos_d[0], num+num_rb_attached_particles, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
                                                                           pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, 
                                                                           excludeMap_d, numExcludes, pairlistdist2, pos_tex[0], neighbors_tex);
      #endif
       
      gpuKernelCheck();
      gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */

      #ifdef USE_NCCL
      if (gpuman.gpus.size() > 1) {
	  // Currently we don't use numPairs_d[i] for i > 0... might be able to reduce data transfer with some kind nccl scatter, and in that case we'd prefer to use all numPairs_d[i]
	  gpuErrchk(hipMemcpy(&numPairs, numPairs_d[0], sizeof(int), hipMemcpyDeviceToHost));
	  gpuman.nccl_broadcast(0, pairTabPotType_d, pairTabPotType_d, numPairs, -1);
	  gpuman.nccl_broadcast(0, pairLists_d, pairLists_d, numPairs, -1);
      }
      gpuman.sync();
      #endif

    //createPairlists<64,64><<< dim3(256,128,numReplicas),dim3(64,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
    //                                                                  pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d,
    //                                                                  excludeMap_d, numExcludes, pairlistdist2);

    //#else
    //createPairlists_debug<<< 2048, 64 >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d, 
      //                            pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2);
    //#endif
//#else
	// Use shared memory for warp_bcast function
	//createPairlists<<< 2048, 64, 2048/WARPSIZE >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2);
    //#ifdef NEW
    //for sm52
    //createPairlists<32,64,1><<<dim3(256,128,numReplicas),dim3(32,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], 
      //GTX 980
      //createPairlists<64,64,8><<<dim3(128,128,numReplicas),dim3(64,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
        //GTX 680
        //createPairlists<64,64,8><<<dim3(256,256,numReplicas),dim3(64,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
        //                                                              pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, 
        //                                                              excludeMap_d, numExcludes, pairlistdist2);
    //createPairlists<64,64><<<dim3(256,128,numReplicas),dim3(64,1,1)>>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0],
    //                                                                  pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d,
    //                                                                  excludeMap_d, numExcludes, pairlistdist2);

    //#else
    //createPairlists<<< 2048, 64, 2048/WARPSIZE >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d,
      //                                             pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2, CellNeighborsList);
    //#endif

//#endif
#if 0
//////debug section			
	// DEBUGING
	gpuErrchk(hipMemcpy(&tmp, numPairs_d[0],	sizeof(int), hipMemcpyDeviceToHost));
	//printf("CreatePairlist found %d pairs\n",tmp);
        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk( hipProfilerStart() );

        // Reset the cell decomposition.
        if (decomp_d)
            hipFree(decomp_d);

        decomp.decompose_d(pos_d[0], num);
        decomp_d = decomp.copyToCUDA();

	gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
        int tmp1 = 0;
        gpuErrchk(hipMemcpyAsync(numPairs_d[0], &tmp1,     sizeof(int), hipMemcpyHostToDevice));
        gpuErrchk(hipDeviceSynchronize());
        // printf("Pairlistdist: %f\n",sqrt(pairlistdist2));

#ifdef DEBUGEXCLUSIONS
        initExSum();
        gpuErrchk(hipDeviceSynchronize()); /* RBTODO: sync needed here? */
#endif
        #if __CUDA_ARCH__ >= 300
        createPairlists_debug<<< 2048, 64 >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2);
#else
        // Use shared memory for warp_bcast function
        createPairlists_debug<<< 2048, 64, 2048/WARPSIZE >>>(pos_d[0], num, numReplicas, sys_d[0], decomp_d, nCells, numPairs_d[0], pairLists_d[0], numParts, type_d, pairTabPotType_d[0], excludes_d, excludeMap_d, numExcludes, pairlistdist2);
#endif
    gpuErrchk(hipMemcpy(&tmp1, numPairs_d[0],  sizeof(int), hipMemcpyDeviceToHost));
    printf("Difference CreatePairlist found %d pairs\n",tmp-tmp1);
    gpuErrchk(hipDeviceSynchronize());

#ifdef DEBUGEXCLUSIONS
	printf("Counted %d exclusions\n", getExSum());
#endif
#endif
}

IndexList ComputeForce::decompDim() const {
	IndexList ret;
	ret.add(decomp.getNx());
	ret.add(decomp.getNy());
	ret.add(decomp.getNz());
	return ret;
}

CellDecomposition ComputeForce::getDecomp() { return decomp; }

float ComputeForce::decompCutoff() { return decomp.getCutoff(); }

float ComputeForce::computeFull(bool get_energy) {
	float energy = 0.0f;
	gridSize = ((num+num_rb_attached_particles) * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeFullKernel<<< numBlocks, numThreads >>>(forceInternal_d[0], pos_d[0], type_d, tableAlpha_d,
		tableEps_d, tableRad6_d, num+num_rb_attached_particles, numParts, sys_d[0], energies_d, gridSize,
		numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num + num_rb_attached_particles + numGroupSites);
	}

	return energy;
}

float ComputeForce::computeSoftcoreFull(bool get_energy) {
	float energy = 0.0f;
	gridSize = ((num+num_rb_attached_particles) * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeSoftcoreFullKernel<<<numBlocks, numThreads>>>(forceInternal_d[0], pos_d[0], type_d,
			tableEps_d, tableRad6_d, num+num_rb_attached_particles, numParts, sys_d[0], energies_d, gridSize,
			numReplicas, get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		hipDeviceSynchronize();
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num + num_rb_attached_particles);
	}

	return energy;
}

float ComputeForce::computeElecFull(bool get_energy) {
	float energy = 0.0f;

	gridSize = num/NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeElecFullKernel<<<numBlocks, numThreads>>>(forceInternal_d[0], pos_d[0], type_d,
			tableAlpha_d, num, numParts, sys_d[0], energies_d, gridSize, numReplicas,
			get_energy);

	// Calculate energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num);
	}

	return energy;
}


float ComputeForce::compute(bool get_energy) {
	float energy = 0.0f;

	gridSize = ((num+num_rb_attached_particles) * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeKernel<<<numBlocks, numThreads>>>(forceInternal_d[0], pos_d[0], type_d,
			tableAlpha_d, tableEps_d, tableRad6_d, num+num_rb_attached_particles, numParts, sys_d[0],
			decomp_d, energies_d, switchStart, switchLen, gridSize, numReplicas,
			get_energy);

	gpuErrchk(hipDeviceSynchronize());
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num + num_rb_attached_particles + numGroupSites);
	}

	return energy;
}

//MLog: added Bond* bondList to the list of passed in variables.
/*float ComputeForce::computeTabulated(Vector3* force, Vector3* pos, int* type,
		Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap,
		Angle* angles, Dihedral* dihedrals, bool get_energy, Bond* bondList) {*/
float ComputeForce::computeTabulated(bool get_energy) {
	float energy = 0.0f;

	gridSize = ((num+num_rb_attached_particles) * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);
	
	// Call the kernel to calculate the forces
	// int nb = (decomp.nCells.x * decomp.nCells.y * decomp.nCells.z);
	// int nb = (1+(decomp.nCells.x * decomp.nCells.y * decomp.nCells.z)) * 75; /* RBTODO: number of pairLists */
	const int nb = 800;
	// printf("ComputeTabulated\n");

	// RBTODO: get_energy
	if (get_energy)
	//if (false) 
	{
		//clearEnergies<<< nb, numThreads >>>(energies_d,num);
		//gpuErrchk(hipDeviceSynchronize());
	        hipMemset((void*)energies_d, 0, sizeof(float)*(num+num_rb_attached_particles+numGroupSites)*numReplicas);
		computeTabulatedEnergyKernel<<< nb, numThreads >>>(forceInternal_d[0], pos_d[0], sys_d[0],
						cutoff2, numPairs_d[0], pairLists_d[0], pairTabPotType_d[0], tablePot_d[0], energies_d);
	}
	
	else
	{
	    // Copy positions from device 0 to all others

                //gpuErrchk(hipBindTexture(0,  PosTex, pos_d[0],sizeof(Vector3)*num*numReplicas));
		//computeTabulatedKernel<<< nb, numThreads >>>(forceInternal_d[0], pos_d[0], sys_d[0],

	    int ngpu = gpuman.gpus.size();
	    if (ngpu == 1) {
		int i = 0;
		computeTabulatedKernel<64><<< dim3(2048,1,1), dim3(64,1,1), 0, gpuman.gpus[i].get_next_stream() >>>
		    (forceInternal_d[i], sys_d[i], cutoff2, numPairs_d[i], pairLists_d[i], pairTabPotType_d[i], tablePot_d[i], pairLists_tex[i], pos_tex[i], pairTabPotType_tex[i]);

	    } else {
	    for (size_t i = 0; i < ngpu; ++i) {
		gpuman.use(i);
		int start =            floor( ((float) numPairs*i    )/ngpu );
		int end   = i < ngpu-1 ? floor( ((float) numPairs*(i+1))/ngpu ) : numPairs;
		
		if (i == ngpu-1) assert(end == numPairs);
		computeTabulatedKernel<64><<< dim3(2048,1,1), dim3(64,1,1), 0, gpuman.gpus[i].get_next_stream() >>>(forceInternal_d[i], sys_d[i],
														    cutoff2, pairLists_d[i], pairTabPotType_d[i], tablePot_d[i], pairLists_tex[i], pos_tex[i], pairTabPotType_tex[i], start, end-start);
                  gpuKernelCheck();
	    }
	    gpuman.use(0);
	    }
                //gpuErrchk(hipUnbindTexture(PosTex));
	}
	/* printPairForceCounter<<<1,32>>>(); */

	//Mlog: the commented function doesn't use bondList, uncomment for testing.
	//if(bondMap_d != NULL && tableBond_d != NULL)
	if(bondList_d != NULL && tableBond_d != NULL)

	{
	    //computeTabulatedBonds <<<numBlocks, numThreads>>> ( force, pos, num, numParts, sys_d[0], bonds, bondMap_d, numBonds, numReplicas, energies_d, get_energy, tableBond_d);
	//computeTabulatedBonds <<<nb, numThreads>>> ( forceInternal_d[0], pos_d[0], sys_d[0], numReplicas*numBonds/2, bondList_d, tableBond_d);
	  //if(get_energy)
              //hipMemset(bond_energy_d, 0, sizeof(float)*num);
		computeTabulatedBonds <<<nb, numThreads, 0, gpuman.get_next_stream()>>> ( forceInternal_d[0], pos_d[0], sys_d[0], numReplicas*numBonds/2, bondList_d, tableBond_d, energies_d, get_energy);
	}

	if (angleList_d != NULL && tableAngle_d != NULL)
        {
            //if(get_energy)
		//computeTabulatedAngles<<<nb, numThreads>>>(forceInternal_d[0], pos_d[0], sys_d[0], numAngles*numReplicas, angleList_d, tableAngle_d);
	    computeTabulatedAngles<<<nb, numThreads, 0, gpuman.get_next_stream()>>>(forceInternal_d[0], pos_d[0], sys_d[0], numAngles*numReplicas, angleList_d, tableAngle_d, energies_d, get_energy);
        }
	if (dihedralList_d != NULL && tableDihedral_d != NULL)
        {
            //if(get_energy)
		//computeTabulatedDihedrals<<<nb, numThreads>>>(forceInternal_d[0], pos_d[0], sys_d[0], numDihedrals*numReplicas, dihedralList_d, dihedralPotList_d, tableDihedral_d);
	    computeTabulatedDihedrals<<<nb, numThreads, 0, gpuman.get_next_stream()>>>(forceInternal_d[0], pos_d[0], sys_d[0], numDihedrals*numReplicas, 
                dihedralList_d, dihedralPotList_d, tableDihedral_d, energies_d, get_energy);
        }

	// TODO: Sum energy
	if (restraintIds_d != NULL )
	    computeHarmonicRestraints<<<1, numThreads, 0, gpuman.get_next_stream()>>>(forceInternal_d[0], pos_d[0], sys_d[0], numRestraints*numReplicas, restraintIds_d, restraintLocs_d, restraintSprings_d);
	

	// Calculate the energy based on the array created by the kernel
	// TODO: return energy
	/*if (get_energy) 
        {
            float e = 0.f;
	    gpuErrchk(hipDeviceSynchronize());
	    thrust::device_ptr<float> en_d(energies_d);
	    e = (thrust::reduce(en_d, en_d+num*numReplicas)) / numReplicas;
            std::fstream energy_file;
            energy_file.open("energy_config.txt", std::fstream::out | std::fstream::app);
            if(energy_file.is_open())
            {
                energy_file << "Configuation Energy: "  << e << " kcal/mol " << std::endl;
                energy_file.close();
            }
            else
            {
                std::cout << "Error in opening energ files\n";
            }
            energy = e;
        }*/
	return energy;
}

float ComputeForce::computeTabulatedFull(bool get_energy) {
	energy = 0.0f;

	gridSize = ((num+num_rb_attached_particles) * numReplicas) / NUM_THREADS + 1;
	dim3 numBlocks(gridSize, 1, 1);
	dim3 numThreads(NUM_THREADS, 1, 1);

	// Call the kernel to calculate forces
	computeTabulatedFullKernel<<< numBlocks, numThreads >>>(forceInternal_d[0], pos_d[0], type_d,	tablePot_d[0], tableBond_d, num+num_rb_attached_particles, numParts, sys_d[0], bonds_d, bondMap_d, numBonds, excludes_d, excludeMap_d, numExcludes, energies_d, gridSize, numReplicas, get_energy, angles_d);
	gpuErrchk(hipDeviceSynchronize());

	computeAngles<<< numBlocks, numThreads >>>(forceInternal_d[0], pos_d[0], angles_d, tableAngle_d,
																						 numAngles, num+num_rb_attached_particles, sys_d[0], energies_d,
																						 get_energy);
	gpuErrchk(hipDeviceSynchronize());
	computeDihedrals<<< numBlocks, numThreads >>>(forceInternal_d[0], pos_d[0], dihedrals_d,
																							  tableDihedral_d, numDihedrals,
																								num+num_rb_attached_particles, sys_d[0], energies_d,
																								get_energy);
	// Calculate the energy based on the array created by the kernel
	if (get_energy) {
		gpuErrchk(hipDeviceSynchronize());
		thrust::device_ptr<float> en_d(energies_d);
		energy = thrust::reduce(en_d, en_d + num + num_rb_attached_particles);
	}

	return energy;
}

void ComputeForce::copyToCUDA(Vector3* forceInternal, Vector3* pos)
{
    const size_t tot_num = (num+num_rb_attached_particles+numGroupSites) * numReplicas;

	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    gpuman.use(i);
	    gpuErrchk(hipMalloc(&pos_d[i], sizeof(Vector3) * tot_num));
	    //Han-Yi bind to the texture
	    hipResourceDesc resDesc;
	    memset(&resDesc, 0, sizeof(resDesc));
	    resDesc.resType = hipResourceTypeLinear;
	    resDesc.res.linear.devPtr = pos_d[i];
	    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	    resDesc.res.linear.desc.x = 32; // bits per channel
	    resDesc.res.linear.desc.y = 32; // bits per channel
	    resDesc.res.linear.desc.z = 32; // bits per channel
	    resDesc.res.linear.desc.w = 32; // bits per channel
	    resDesc.res.linear.sizeInBytes = tot_num*sizeof(float4);
	    
	    hipTextureDesc texDesc;
	    memset(&texDesc, 0, sizeof(texDesc));
	    texDesc.readMode = hipReadModeElementType;
	    
	    // create texture object: we only have to do this once!
	    pos_tex[i] = 0;
	    hipCreateTextureObject(&pos_tex[i], &resDesc, &texDesc, NULL);
	    gpuErrchk(hipDeviceSynchronize());
	}
	gpuman.use(0);

	gpuErrchk(hipMemcpyAsync(pos_d[0], pos, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));

	for (std::size_t i = 0; i < gpuman.gpus.size(); ++i) {
	    gpuman.use(i);
	    gpuErrchk(hipMalloc(&forceInternal_d[i], sizeof(Vector3) * tot_num));
	}
	gpuman.use(0);
	gpuErrchk(hipMemcpyAsync(forceInternal_d[0], forceInternal, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));

	gpuErrchk(hipDeviceSynchronize());
}
void ComputeForce::copyToCUDA(Vector3* forceInternal, Vector3* pos, Vector3* mom)
{
    const size_t tot_num = num * numReplicas;

        gpuErrchk(hipMalloc(&mom_d, sizeof(Vector3) * tot_num));
        gpuErrchk(hipMemcpyAsync(mom_d, mom, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));

	copyToCUDA(forceInternal,pos);
        gpuErrchk(hipDeviceSynchronize());
}
void ComputeForce::copyToCUDA(Vector3* forceInternal, Vector3* pos, Vector3* mom, float* random)
{
    const size_t tot_num = num * numReplicas;

        gpuErrchk(hipMalloc(&ran_d, sizeof(float) * tot_num));
        gpuErrchk(hipMemcpyAsync(ran_d, random, sizeof(float) * tot_num, hipMemcpyHostToDevice));

	copyToCUDA(forceInternal, pos, mom);
        gpuErrchk(hipDeviceSynchronize());
}

void ComputeForce::setForceInternalOnDevice(Vector3* f) {
    // const size_t tot_num = (num+numGroupSites) * numReplicas;
    assert(numGroupSites == 0); // IMD, the only feature using this function, is currently incompatible with group sites
    const size_t tot_num = num * numReplicas;
	gpuErrchk(hipMemcpy(forceInternal_d[0], f, sizeof(Vector3) * tot_num, hipMemcpyHostToDevice));
}

void ComputeForce::copyToCUDA(int simNum, int *type, Bond* bonds, int2* bondMap, Exclude* excludes, int2* excludeMap, Angle* angles, Dihedral* dihedrals, const Restraint* const restraints)
{
    assert(simNum == numReplicas); // Not sure why we have both of these things
    int tot_num_with_rb = (num+num_rb_attached_particles) * simNum;
    int tot_num_with_rb_group = (num+num_rb_attached_particles+numGroupSites) * simNum;
	// type_d
	gpuErrchk(hipMalloc(&type_d, sizeof(int) * tot_num_with_rb));
	gpuErrchk(hipMemcpyAsync(type_d, type, sizeof(int) * tot_num_with_rb, hipMemcpyHostToDevice));
	
	if (numBonds > 0)
	{
		// bonds_d
		gpuErrchk(hipMalloc(&bonds_d, sizeof(Bond) * numBonds));
		gpuErrchk(hipMemcpyAsync(bonds_d, bonds, sizeof(Bond) * numBonds, hipMemcpyHostToDevice));
		
		// bondMap_d
		gpuErrchk(hipMalloc(&bondMap_d, sizeof(int2) * tot_num_with_rb_group));
		gpuErrchk(hipMemcpyAsync(bondMap_d, bondMap, sizeof(int2) * tot_num_with_rb_group, hipMemcpyHostToDevice));
	}

	if (numExcludes > 0) {
	    // printf("Copying %d exclusions to the GPU\n", numExcludes);
	    
		// excludes_d
		gpuErrchk(hipMalloc(&excludes_d, sizeof(Exclude) * numExcludes));
		gpuErrchk(hipMemcpyAsync(excludes_d, excludes, sizeof(Exclude) * numExcludes,
				hipMemcpyHostToDevice));
		
		// excludeMap_d
		gpuErrchk(hipMalloc(&excludeMap_d, sizeof(int2) * tot_num_with_rb));
		gpuErrchk(hipMemcpyAsync(excludeMap_d, excludeMap, sizeof(int2) * tot_num_with_rb,
				hipMemcpyHostToDevice));
	}

	if (numAngles > 0) {
		// angles_d
		gpuErrchk(hipMalloc(&angles_d, sizeof(Angle) * numAngles));
		gpuErrchk(hipMemcpyAsync(angles_d, angles, sizeof(Angle) * numAngles,
				hipMemcpyHostToDevice));
	}

	if (numDihedrals > 0) {
		// dihedrals_d
		gpuErrchk(hipMalloc(&dihedrals_d, sizeof(Dihedral) * numDihedrals));
		gpuErrchk(hipMemcpyAsync(dihedrals_d, dihedrals,
												 		  sizeof(Dihedral) * numDihedrals,
														 	hipMemcpyHostToDevice));
	}

	if (numRestraints > 0) {
	    int restraintIds[numRestraints];
	    Vector3 restraintLocs[numRestraints];
	    float restraintSprings[numRestraints];
	    for (int i = 0; i < numRestraints; ++i) {
		restraintIds[i]     = restraints[i].id;
		restraintLocs[i]    = restraints[i].r0;
		restraintSprings[i] = restraints[i].k;
	    }

	    gpuErrchk(hipMalloc(&restraintIds_d, sizeof(int) * numRestraints));
	    gpuErrchk(hipMalloc(&restraintLocs_d, sizeof(Vector3) * numRestraints));
	    gpuErrchk(hipMalloc(&restraintSprings_d, sizeof(float) * numRestraints));
	    
	    gpuErrchk(hipMemcpyAsync(restraintIds_d, restraintIds,
				      sizeof(int)     * numRestraints, hipMemcpyHostToDevice));
	    gpuErrchk(hipMemcpyAsync(restraintLocs_d, restraintLocs,
				      sizeof(Vector3) * numRestraints, hipMemcpyHostToDevice));
	    gpuErrchk(hipMemcpyAsync(restraintSprings_d, restraintSprings,
				      sizeof(float)   * numRestraints, hipMemcpyHostToDevice));
	}	    

	gpuErrchk(hipDeviceSynchronize());
}

// void ComputeForce::createBondList(int3 *bondList)
// {
// 	size_t size = (numBonds / 2) * numReplicas * sizeof(int3);
// 	gpuErrchk( hipMalloc( &bondList_d, size ) );
// 	gpuErrchk( hipMemcpyAsync( bondList_d, bondList, size, hipMemcpyHostToDevice) );

// 	for(int i = 0 ; i < (numBonds / 2) * numReplicas ; i++)
// 	{
// 		cout << "Displaying: bondList_d["<< i <<"].x = " << bondList[i].x << ".\n"
// 			<< "Displaying: bondList_d["<< i <<"].y = " << bondList[i].y << ".\n"
// 			<< "Displaying: bondList_d["<< i <<"].z = " << bondList[i].z << ".\n";

// 	}
// }

void ComputeForce::copyBondedListsToGPU(int3 *bondList, int4 *angleList, int4 *dihedralList, int *dihedralPotList) {

	
	size_t size;

	if (numBonds > 0) {
	size = (numBonds / 2) * numReplicas * sizeof(int3);
	gpuErrchk( hipMalloc( &bondList_d, size ) );
	gpuErrchk( hipMemcpyAsync( bondList_d, bondList, size, hipMemcpyHostToDevice) );
	}
	
	if (numAngles > 0) {
    size = numAngles * numReplicas * sizeof(int4);
    gpuErrchk( hipMalloc( &angleList_d, size ) );
    gpuErrchk( hipMemcpyAsync( angleList_d, angleList, size, hipMemcpyHostToDevice) );
	}
	
	if (numDihedrals > 0) {
    size = numDihedrals * numReplicas * sizeof(int4);
    gpuErrchk( hipMalloc( &dihedralList_d, size ) );
    gpuErrchk( hipMemcpyAsync( dihedralList_d, dihedralList, size, hipMemcpyHostToDevice) );

    size = numDihedrals * numReplicas * sizeof(int);
    gpuErrchk( hipMalloc( &dihedralPotList_d, size ) );
    gpuErrchk( hipMemcpyAsync( dihedralPotList_d, dihedralPotList, size, hipMemcpyHostToDevice) );
	}
}
