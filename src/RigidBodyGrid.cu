#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// Grid base class that does just the basics.
// Author: Jeff Comer <jcomer2@illinois.edu>

#include "RigidBodyGrid.h"
#include <hip/hip_runtime.h>

#define STRLEN 512

	/*                               \
	| CONSTRUCTORS, DESTRUCTORS, I/O |
	\===============================*/

RigidBodyGrid::RigidBodyGrid() {
	RigidBodyGrid tmp(1,1,1);
	val = new float[1];
	*this = tmp;									// TODO: verify that this is OK
}

// The most obvious of constructors.
RigidBodyGrid::RigidBodyGrid(int nx0, int ny0, int nz0) {
	nx = abs(nx0);
	ny = abs(ny0);
	nz = abs(nz0);
	
	val = new float[nx*ny*nz];
	zero();
}

RigidBodyGrid::RigidBodyGrid(const BaseGrid& g) {
	nx = g.nx;
	ny = g.ny;
	nz = g.nz;
	
	val = new float[nx*ny*nz];
	for (int i = 0; i < nx*ny*nz; i++) val[i] = g.val[i];
}

// Make an exact copy of a grid.
RigidBodyGrid::RigidBodyGrid(const RigidBodyGrid& g) {
	nx = g.nx;
	ny = g.ny;
	nz = g.nz;
	
	val = new float[nx*ny*nz];
	for (int i = 0; i < nx*ny*nz; i++) val[i] = g.val[i];
}

RigidBodyGrid RigidBodyGrid::mult(const RigidBodyGrid& g) {
	for (int i = 0; i < nx*ny*nz; i++) val[i] *= g.val[i];
	return *this;
}

RigidBodyGrid& RigidBodyGrid::operator=(const RigidBodyGrid& g) {
	if(val!=NULL) 
            delete[] val;
	val = NULL;
	nx = g.nx;
	ny = g.ny;
	nz = g.nz;
	
	val = new float[nx*ny*nz];
	for (int i = 0; i < nx*ny*nz; i++) val[i] = g.val[i];

	return *this;
}

RigidBodyGrid::~RigidBodyGrid() {
	if (val != NULL)
        {
		delete[] val;
                val = NULL;
        }
}

void RigidBodyGrid::zero() {
	for (int i = 0; i < nx*ny*nz; i++) val[i] = 0.0f;
}

bool RigidBodyGrid::setValue(int j, float v) {
	if (j < 0 || j >= nx*ny*nz) return false;
	val[j] = v;
	return true;
}

bool RigidBodyGrid::setValue(int ix, int iy, int iz, float v) {
	if (ix < 0 || ix >= nx) return false;
	if (iy < 0 || iy >= ny) return false;
	if (iz < 0 || iz >= nz) return false;
	int j = iz + iy*nz + ix*ny*nz;

	val[j] = v;
	return true;
}

float RigidBodyGrid::getValue(int j) const {

	if (j < 0 || j >= nx*ny*nz) return 0.0f;
	return val[j];
/*
    Vector3 idx = getPosition(j)
    return getValue(idx.x,idx.y,idx.z);
*/
}

HOST DEVICE float RigidBodyGrid::getValue(int ix, int iy, int iz) const {
/*
           if(ix < 0) ix = 0;
           else if(ix >= nx) ix = nx -1;

           if(iy < 0) iy = 0;
           else if(iy >= ny) iy = ny-1;

           if(iz < 0) iz = 0;
           else if(iz >= nz) iz = nz-1;

           int j = iz + nz * (iy + ny * ix);
           return val[j];
*/

	if (ix < 0 || ix >= nx) return 0.0f;
	if (iy < 0 || iy >= ny) return 0.0f;
	if (iz < 0 || iz >= nz) return 0.0f;
	
	int j = iz + iy*nz + ix*ny*nz;
	return val[j];

}

Vector3 RigidBodyGrid::getPosition(const int j) const {
	/* const int iz = j%nz; */
	/* const int iy = (j/nz)%ny; */
	/* const int ix = j/(nz*ny); */
	const int jy = j/nz;
	const int jx = jy/ny;

	const int iz = j - jy*nz;
	const int iy = jy - jx*ny;
	// const int ix = jx;

	return Vector3(jx,iy,iz);
}

Vector3 RigidBodyGrid::getPosition(int j, Matrix3 basis, Vector3 origin) const {
	int iz = j%nz;
	int iy = (j/nz)%ny;
	int ix = j/(nz*ny);

	return basis.transform(Vector3(ix, iy, iz)) + origin;
}

IndexList RigidBodyGrid::index(int j) const {
	int iz = j%nz;
	int iy = (j/nz)%ny;
	int ix = j/(nz*ny);
	IndexList ret;
	ret.add(ix);
	ret.add(iy);
	ret.add(iz);
	return ret;
}
int RigidBodyGrid::indexX(int j) const { return j/(nz*ny); }
int RigidBodyGrid::indexY(int j) const { return (j/nz)%ny; }
int RigidBodyGrid::indexZ(int j) const { return j%nz; }
int RigidBodyGrid::index(int ix, int iy, int iz) const { return iz + iy*nz + ix*ny*nz; }

// Add a fixed value to the grid.
void RigidBodyGrid::shift(float s) {
	for (int i = 0; i < nx*ny*nz; i++) val[i] += s;
}

// Multiply the grid by a fixed value.
void RigidBodyGrid::scale(float s) {
	for (int i = 0; i < nx*ny*nz; i++) val[i] *= s;
}

/** interpolateForce() to be used on CUDA Device **/
DEVICE ForceEnergy RigidBodyGrid::interpolateForceD(const Vector3 l) const {
	Vector3 f;
	// Vector3 l = basisInv.transform(pos - origin);
	const int homeX = int(floor(l.x));
	const int homeY = int(floor(l.y));
	const int homeZ = int(floor(l.z));
	const float wx = l.x - homeX;
	const float wy = l.y - homeY;
	const float wz = l.z - homeZ;
	const float wx2 = wx*wx;

	/* f.x */
	float g3[3][4];
	for (int iz = 0; iz < 4; iz++) {
		float g2[2][4];
		const int jz = (iz + homeZ - 1);
		for (int iy = 0; iy < 4; iy++) {
			float v[4];
			const int jy = (iy + homeY - 1);
			for (int ix = 0; ix < 4; ix++) {
				const int jx = (ix + homeX - 1);
				const int ind = jz + jy*nz + jx*nz*ny;
				v[ix] = jz < 0 || jz >= nz || jy < 0 || jy >= ny || jx < 0 || jx >= nx ?
					0 : val[ind];
			}
			const float a3 = 0.5f*(-v[0] + 3.0f*v[1] - 3.0f*v[2] + v[3])*wx2;
			const float a2 = 0.5f*(2.0f*v[0] - 5.0f*v[1] + 4.0f*v[2] - v[3])*wx;
			const float a1 = 0.5f*(-v[0] + v[2]);
			g2[0][iy] = 3.0f*a3 + 2.0f*a2 + a1;				/* f.x (derivative) */
			g2[1][iy] = a3*wx + a2*wx + a1*wx + v[1]; /* f.y & f.z */
		}

		// Mix along y.
		{
			g3[0][iz] = 0.5f*(-g2[0][0] + 3.0f*g2[0][1] - 3.0f*g2[0][2] + g2[0][3])*wy*wy*wy +
				0.5f*(2.0f*g2[0][0] - 5.0f*g2[0][1] + 4.0f*g2[0][2] - g2[0][3])      *wy*wy +
				0.5f*(-g2[0][0] + g2[0][2])                                          *wy +
				g2[0][1];
		}

		{
			const float a3 = 0.5f*(-g2[1][0] + 3.0f*g2[1][1] - 3.0f*g2[1][2] + g2[1][3])*wy*wy;
			const float a2 = 0.5f*(2.0f*g2[1][0] - 5.0f*g2[1][1] + 4.0f*g2[1][2] - g2[1][3])*wy;
			const float a1 = 0.5f*(-g2[1][0] + g2[1][2]);
			g3[1][iz] = 3.0f*a3 + 2.0f*a2 + a1;						/* f.y */
			g3[2][iz] = a3*wy + a2*wy + a1*wy + g2[1][1]; /* f.z */
		}
	}

	// Mix along z.
	f.x = -0.5f*(-g3[0][0] + 3.0f*g3[0][1] - 3.0f*g3[0][2] + g3[0][3])*wz*wz*wz +
		-0.5f*(2.0f*g3[0][0] - 5.0f*g3[0][1] + 4.0f*g3[0][2] - g3[0][3])*wz*wz +
		-0.5f*(-g3[0][0] + g3[0][2])                                    *wz -
		g3[0][1];
	f.y = -0.5f*(-g3[1][0] + 3.0f*g3[1][1] - 3.0f*g3[1][2] + g3[1][3])*wz*wz*wz +
		-0.5f*(2.0f*g3[1][0] - 5.0f*g3[1][1] + 4.0f*g3[1][2] - g3[1][3])*wz*wz +
		-0.5f*(-g3[1][0] + g3[1][2])                                    *wz -
		g3[1][1];
	f.z = -1.5f*(-g3[2][0] + 3.0f*g3[2][1] - 3.0f*g3[2][2] + g3[2][3])*wz*wz -
		(2.0f*g3[2][0] - 5.0f*g3[2][1] + 4.0f*g3[2][2] - g3[2][3])      *wz -
		0.5f*(-g3[2][0] + g3[2][2]);
	float e = 0.5f*(-g3[2][0] + 3.0f*g3[2][1] - 3.0f*g3[2][2] + g3[2][3])*wz*wz*wz +
		0.5f*(2.0f*g3[2][0] - 5.0f*g3[2][1] + 4.0f*g3[2][2] - g3[2][3])    *wz*wz +
		0.5f*(-g3[2][0] + g3[2][2])                                        *wz +
		g3[2][1];
	
	return ForceEnergy(f,e);
}
//#define cubic
DEVICE ForceEnergy RigidBodyGrid::interpolateForceDLinearly(const Vector3& l) const {
//#ifdef cubic
//return interpolateForceD(l);
//#elif defined(cubic_namd)
//return interpolateForceDnamd(l);
//#else
	// Find the home node.
	const int homeX = int(floor(l.x));
	const int homeY = int(floor(l.y));
	const int homeZ = int(floor(l.z));

	Vector3 f;

	const float wx = l.x - homeX;
	const float wy = l.y - homeY;	
	const float wz = l.z - homeZ;

	float v[2][2][2];
	for (int iz = 0; iz < 2; iz++) {
		int jz = (iz + homeZ);
		for (int iy = 0; iy < 2; iy++) {
			int jy = (iy + homeY);
			for (int ix = 0; ix < 2; ix++) {
				int jx = (ix + homeX);
				int ind = jz + jy*nz + jx*nz*ny;
				v[ix][iy][iz] = jz < 0 || jz >= nz || jy < 0 || jy >= ny || jx < 0 || jx >= nx ?
					0 : val[ind];
			}
		}
	}

	float g3[3][2];
	for (int iz = 0; iz < 2; iz++) {
		float g2[2][2];
		for (int iy = 0; iy < 2; iy++) {
			g2[0][iy] = (v[1][iy][iz] - v[0][iy][iz]); /* f.x */
			g2[1][iy] = wx * (v[1][iy][iz] - v[0][iy][iz]) + v[0][iy][iz]; /* f.y & f.z */
		}
		// Mix along y.
		g3[0][iz] = wy * (g2[0][1] - g2[0][0]) + g2[0][0];
		g3[1][iz] = (g2[1][1] - g2[1][0]);
		g3[2][iz] = wy * (g2[1][1] - g2[1][0]) + g2[1][0];
	}
	// Mix along z.
	f.x = -(wz * (g3[0][1] - g3[0][0]) + g3[0][0]);
	f.y = -(wz * (g3[1][1] - g3[1][0]) + g3[1][0]);
	f.z = -      (g3[2][1] - g3[2][0]);
	float e = wz * (g3[2][1] - g3[2][0]) + g3[2][0];
	return ForceEnergy(f,e);
//#endif
}
DEVICE ForceEnergy RigidBodyGrid::interpolateForceDnamd(const Vector3& l) const
{
                Vector3 f;
                //const Vector3 l = basisInv.transform(pos - origin);

                const int homeX = int(floor(l.x));
                const int homeY = int(floor(l.y));
                const int homeZ = int(floor(l.z));
                const float wx = l.x - homeX;
                const float wy = l.y - homeY;
                const float wz = l.z - homeZ;

                Vector3 dg = Vector3(wx,wy,wz);

                int inds[3];
                inds[0] = homeX;
                inds[1] = homeY;
                inds[2] = homeZ;

                // TODO: handle edges

                // Compute b
                                   float b[64];    // Matrix of values at 8 box corners
                compute_b(b, inds);

                // Compute a
                                   float a[64];
                compute_a(a, b);

                // Calculate powers of x, y, z for later use
                                   // e.g. x[2] = x^2
                                                      float x[4], y[4], z[4];
                x[0] = 1; y[0] = 1; z[0] = 1;
                for (int j = 1; j < 4; j++) {
                    x[j] = x[j-1] * dg.x;
                    y[j] = y[j-1] * dg.y;
                    z[j] = z[j-1] * dg.z;
                }

                float e = compute_V(a, x, y, z);
                f = compute_dV(a, x, y, z);

                //f = basisInv.transpose().transform(f);
                return ForceEnergy(f,e);
        }

DEVICE float RigidBodyGrid::compute_V(float *a, float *x, float *y, float *z) const
        {
            float V = 0.0;
            long int ind = 0;
            for (int l = 0; l < 4; l++) {
                for (int k = 0; k < 4; k++) {
                    for (int j = 0; j < 4; j++) {
                        V += a[ind] * x[j] * y[k] * z[l];
                        ind++;
                    }
                }
            }
            return V;
        }
DEVICE Vector3 RigidBodyGrid::compute_dV(float *a, float *x, float *y, float *z) const
        {
            Vector3 dV = Vector3(0.0f);
            long int ind = 0;
            for (int l = 0; l < 4; l++) {
                for (int k = 0; k < 4; k++) {
                    for (int j = 0; j < 4; j++) {
                        if (j > 0) dV.x += a[ind] * j * x[j-1] * y[k]   * z[l];         // dV/dx
                        if (k > 0) dV.y += a[ind] * k * x[j]   * y[k-1] * z[l];         // dV/dy
                        if (l > 0) dV.z += a[ind] * l * x[j]   * y[k]   * z[l-1];       // dV/dz
                        ind++;
                    }
                }
            }
            return dV*(-1.f);
        }
DEVICE void RigidBodyGrid::compute_a(float *a, float *b) const
        {
            // Static sparse 64x64 matrix times vector ... nicer looking way than this?
            a[0] = b[0];
            a[1] = b[8];
            a[2] = -3*b[0] + 3*b[1] - 2*b[8] - b[9];
            a[3] = 2*b[0] - 2*b[1] + b[8] + b[9];
            a[4] = b[16];
            a[5] = b[32];
            a[6] = -3*b[16] + 3*b[17] - 2*b[32] - b[33];
            a[7] = 2*b[16] - 2*b[17] + b[32] + b[33];
            a[8] = -3*b[0] + 3*b[2] - 2*b[16] - b[18];
            a[9] = -3*b[8] + 3*b[10] - 2*b[32] - b[34];
            a[10] = 9*b[0] - 9*b[1] - 9*b[2] + 9*b[3] + 6*b[8] + 3*b[9] - 6*b[10] - 3*b[11]
                + 6*b[16] - 6*b[17] + 3*b[18] - 3*b[19] + 4*b[32] + 2*b[33] + 2*b[34] + b[35];
            a[11] = -6*b[0] + 6*b[1] + 6*b[2] - 6*b[3] - 3*b[8] - 3*b[9] + 3*b[10] + 3*b[11]
                - 4*b[16] + 4*b[17] - 2*b[18] + 2*b[19] - 2*b[32] - 2*b[33] - b[34] - b[35];
            a[12] = 2*b[0] - 2*b[2] + b[16] + b[18];
            a[13] = 2*b[8] - 2*b[10] + b[32] + b[34];
            a[14] = -6*b[0] + 6*b[1] + 6*b[2] - 6*b[3] - 4*b[8] - 2*b[9] + 4*b[10] + 2*b[11]
                - 3*b[16] + 3*b[17] - 3*b[18] + 3*b[19] - 2*b[32] - b[33] - 2*b[34] - b[35];
            a[15] = 4*b[0] - 4*b[1] - 4*b[2] + 4*b[3] + 2*b[8] + 2*b[9] - 2*b[10] - 2*b[11]
                + 2*b[16] - 2*b[17] + 2*b[18] - 2*b[19] + b[32] + b[33] + b[34] + b[35];
            a[16] = b[24];
            a[17] = b[40];
            a[18] = -3*b[24] + 3*b[25] - 2*b[40] - b[41];
            a[19] = 2*b[24] - 2*b[25] + b[40] + b[41];
            a[20] = b[48];
            a[21] = b[56];
            a[22] = -3*b[48] + 3*b[49] - 2*b[56] - b[57];
            a[23] = 2*b[48] - 2*b[49] + b[56] + b[57];
            a[24] = -3*b[24] + 3*b[26] - 2*b[48] - b[50];
            a[25] = -3*b[40] + 3*b[42] - 2*b[56] - b[58];
            a[26] = 9*b[24] - 9*b[25] - 9*b[26] + 9*b[27] + 6*b[40] + 3*b[41] - 6*b[42] - 3*b[43]
                + 6*b[48] - 6*b[49] + 3*b[50] - 3*b[51] + 4*b[56] + 2*b[57] + 2*b[58] + b[59];
            a[27] = -6*b[24] + 6*b[25] + 6*b[26] - 6*b[27] - 3*b[40] - 3*b[41] + 3*b[42] + 3*b[43]
                - 4*b[48] + 4*b[49] - 2*b[50] + 2*b[51] - 2*b[56] - 2*b[57] - b[58] - b[59];
            a[28] = 2*b[24] - 2*b[26] + b[48] + b[50];
            a[29] = 2*b[40] - 2*b[42] + b[56] + b[58];
            a[30] = -6*b[24] + 6*b[25] + 6*b[26] - 6*b[27] - 4*b[40] - 2*b[41] + 4*b[42] + 2*b[43]
                - 3*b[48] + 3*b[49] - 3*b[50] + 3*b[51] - 2*b[56] - b[57] - 2*b[58] - b[59];
            a[31] = 4*b[24] - 4*b[25] - 4*b[26] + 4*b[27] + 2*b[40] + 2*b[41] - 2*b[42] - 2*b[43]
                + 2*b[48] - 2*b[49] + 2*b[50] - 2*b[51] + b[56] + b[57] + b[58] + b[59];
            a[32] = -3*b[0] + 3*b[4] - 2*b[24] - b[28];
            a[33] = -3*b[8] + 3*b[12] - 2*b[40] - b[44];
            a[34] = 9*b[0] - 9*b[1] - 9*b[4] + 9*b[5] + 6*b[8] + 3*b[9] - 6*b[12] - 3*b[13]
                + 6*b[24] - 6*b[25] + 3*b[28] - 3*b[29] + 4*b[40] + 2*b[41] + 2*b[44] + b[45];
            a[35] = -6*b[0] + 6*b[1] + 6*b[4] - 6*b[5] - 3*b[8] - 3*b[9] + 3*b[12] + 3*b[13]
                - 4*b[24] + 4*b[25] - 2*b[28] + 2*b[29] - 2*b[40] - 2*b[41] - b[44] - b[45];
            a[36] = -3*b[16] + 3*b[20] - 2*b[48] - b[52];
            a[37] = -3*b[32] + 3*b[36] - 2*b[56] - b[60];
            a[38] = 9*b[16] - 9*b[17] - 9*b[20] + 9*b[21] + 6*b[32] + 3*b[33] - 6*b[36] - 3*b[37]
                + 6*b[48] - 6*b[49] + 3*b[52] - 3*b[53] + 4*b[56] + 2*b[57] + 2*b[60] + b[61];
            a[39] = -6*b[16] + 6*b[17] + 6*b[20] - 6*b[21] - 3*b[32] - 3*b[33] + 3*b[36] + 3*b[37]
                - 4*b[48] + 4*b[49] - 2*b[52] + 2*b[53] - 2*b[56] - 2*b[57] - b[60] - b[61];
            a[40] = 9*b[0] - 9*b[2] - 9*b[4] + 9*b[6] + 6*b[16] + 3*b[18] - 6*b[20] - 3*b[22]
                + 6*b[24] - 6*b[26] + 3*b[28] - 3*b[30] + 4*b[48] + 2*b[50] + 2*b[52] + b[54];
            a[41] = 9*b[8] - 9*b[10] - 9*b[12] + 9*b[14] + 6*b[32] + 3*b[34] - 6*b[36] - 3*b[38]
                + 6*b[40] - 6*b[42] + 3*b[44] - 3*b[46] + 4*b[56] + 2*b[58] + 2*b[60] + b[62];
            a[42] = -27*b[0] + 27*b[1] + 27*b[2] - 27*b[3] + 27*b[4] - 27*b[5] - 27*b[6] + 27*b[7]
                - 18*b[8] - 9*b[9] + 18*b[10] + 9*b[11] + 18*b[12] + 9*b[13] - 18*b[14] - 9*b[15]
                - 18*b[16] + 18*b[17] - 9*b[18] + 9*b[19] + 18*b[20] - 18*b[21] + 9*b[22] - 9*b[23]
                - 18*b[24] + 18*b[25] + 18*b[26] - 18*b[27] - 9*b[28] + 9*b[29] + 9*b[30] - 9*b[31]
                - 12*b[32] - 6*b[33] - 6*b[34] - 3*b[35] + 12*b[36] + 6*b[37] + 6*b[38] + 3*b[39]
                - 12*b[40] - 6*b[41] + 12*b[42] + 6*b[43] - 6*b[44] - 3*b[45] + 6*b[46] + 3*b[47]
                - 12*b[48] + 12*b[49] - 6*b[50] + 6*b[51] - 6*b[52] + 6*b[53] - 3*b[54] + 3*b[55]
                - 8*b[56] - 4*b[57] - 4*b[58] - 2*b[59] - 4*b[60] - 2*b[61] - 2*b[62] - b[63];
            a[43] = 18*b[0] - 18*b[1] - 18*b[2] + 18*b[3] - 18*b[4] + 18*b[5] + 18*b[6] - 18*b[7]
                + 9*b[8] + 9*b[9] - 9*b[10] - 9*b[11] - 9*b[12] - 9*b[13] + 9*b[14] + 9*b[15]
                + 12*b[16] - 12*b[17] + 6*b[18] - 6*b[19] - 12*b[20] + 12*b[21] - 6*b[22] + 6*b[23]
                + 12*b[24] - 12*b[25] - 12*b[26] + 12*b[27] + 6*b[28] - 6*b[29] - 6*b[30] + 6*b[31]
                + 6*b[32] + 6*b[33] + 3*b[34] + 3*b[35] - 6*b[36] - 6*b[37] - 3*b[38] - 3*b[39]
                + 6*b[40] + 6*b[41] - 6*b[42] - 6*b[43] + 3*b[44] + 3*b[45] - 3*b[46] - 3*b[47]
                + 8*b[48] - 8*b[49] + 4*b[50] - 4*b[51] + 4*b[52] - 4*b[53] + 2*b[54] - 2*b[55]
                + 4*b[56] + 4*b[57] + 2*b[58] + 2*b[59] + 2*b[60] + 2*b[61] + b[62] + b[63];
            a[44] = -6*b[0] + 6*b[2] + 6*b[4] - 6*b[6] - 3*b[16] - 3*b[18] + 3*b[20] + 3*b[22]
                - 4*b[24] + 4*b[26] - 2*b[28] + 2*b[30] - 2*b[48] - 2*b[50] - b[52] - b[54];
            a[45] = -6*b[8] + 6*b[10] + 6*b[12] - 6*b[14] - 3*b[32] - 3*b[34] + 3*b[36] + 3*b[38]
                - 4*b[40] + 4*b[42] - 2*b[44] + 2*b[46] - 2*b[56] - 2*b[58] - b[60] - b[62];
            a[46] = 18*b[0] - 18*b[1] - 18*b[2] + 18*b[3] - 18*b[4] + 18*b[5] + 18*b[6] - 18*b[7]
                + 12*b[8] + 6*b[9] - 12*b[10] - 6*b[11] - 12*b[12] - 6*b[13] + 12*b[14] + 6*b[15]
                + 9*b[16] - 9*b[17] + 9*b[18] - 9*b[19] - 9*b[20] + 9*b[21] - 9*b[22] + 9*b[23]
                + 12*b[24] - 12*b[25] - 12*b[26] + 12*b[27] + 6*b[28] - 6*b[29] - 6*b[30] + 6*b[31]
                + 6*b[32] + 3*b[33] + 6*b[34] + 3*b[35] - 6*b[36] - 3*b[37] - 6*b[38] - 3*b[39]
                + 8*b[40] + 4*b[41] - 8*b[42] - 4*b[43] + 4*b[44] + 2*b[45] - 4*b[46] - 2*b[47]
                + 6*b[48] - 6*b[49] + 6*b[50] - 6*b[51] + 3*b[52] - 3*b[53] + 3*b[54] - 3*b[55]
                + 4*b[56] + 2*b[57] + 4*b[58] + 2*b[59] + 2*b[60] + b[61] + 2*b[62] + b[63];
            a[47] = -12*b[0] + 12*b[1] + 12*b[2] - 12*b[3] + 12*b[4] - 12*b[5] - 12*b[6] + 12*b[7]
                - 6*b[8] - 6*b[9] + 6*b[10] + 6*b[11] + 6*b[12] + 6*b[13] - 6*b[14] - 6*b[15]
                - 6*b[16] + 6*b[17] - 6*b[18] + 6*b[19] + 6*b[20] - 6*b[21] + 6*b[22] - 6*b[23]
                - 8*b[24] + 8*b[25] + 8*b[26] - 8*b[27] - 4*b[28] + 4*b[29] + 4*b[30] - 4*b[31]
                - 3*b[32] - 3*b[33] - 3*b[34] - 3*b[35] + 3*b[36] + 3*b[37] + 3*b[38] + 3*b[39]
                - 4*b[40] - 4*b[41] + 4*b[42] + 4*b[43] - 2*b[44] - 2*b[45] + 2*b[46] + 2*b[47]
                - 4*b[48] + 4*b[49] - 4*b[50] + 4*b[51] - 2*b[52] + 2*b[53] - 2*b[54] + 2*b[55]
                - 2*b[56] - 2*b[57] - 2*b[58] - 2*b[59] - b[60] - b[61] - b[62] - b[63];
            a[48] = 2*b[0] - 2*b[4] + b[24] + b[28];
            a[49] = 2*b[8] - 2*b[12] + b[40] + b[44];
            a[50] = -6*b[0] + 6*b[1] + 6*b[4] - 6*b[5] - 4*b[8] - 2*b[9] + 4*b[12] + 2*b[13]
                - 3*b[24] + 3*b[25] - 3*b[28] + 3*b[29] - 2*b[40] - b[41] - 2*b[44] - b[45];
            a[51] = 4*b[0] - 4*b[1] - 4*b[4] + 4*b[5] + 2*b[8] + 2*b[9] - 2*b[12] - 2*b[13]
                + 2*b[24] - 2*b[25] + 2*b[28] - 2*b[29] + b[40] + b[41] + b[44] + b[45];
            a[52] = 2*b[16] - 2*b[20] + b[48] + b[52];
            a[53] = 2*b[32] - 2*b[36] + b[56] + b[60];
            a[54] = -6*b[16] + 6*b[17] + 6*b[20] - 6*b[21] - 4*b[32] - 2*b[33] + 4*b[36] + 2*b[37]
                - 3*b[48] + 3*b[49] - 3*b[52] + 3*b[53] - 2*b[56] - b[57] - 2*b[60] - b[61];
            a[55] = 4*b[16] - 4*b[17] - 4*b[20] + 4*b[21] + 2*b[32] + 2*b[33] - 2*b[36] - 2*b[37]
                + 2*b[48] - 2*b[49] + 2*b[52] - 2*b[53] + b[56] + b[57] + b[60] + b[61];
            a[56] = -6*b[0] + 6*b[2] + 6*b[4] - 6*b[6] - 4*b[16] - 2*b[18] + 4*b[20] + 2*b[22]
                - 3*b[24] + 3*b[26] - 3*b[28] + 3*b[30] - 2*b[48] - b[50] - 2*b[52] - b[54];
            a[57] = -6*b[8] + 6*b[10] + 6*b[12] - 6*b[14] - 4*b[32] - 2*b[34] + 4*b[36] + 2*b[38]
                - 3*b[40] + 3*b[42] - 3*b[44] + 3*b[46] - 2*b[56] - b[58] - 2*b[60] - b[62];
           a[58] = 18*b[0] - 18*b[1] - 18*b[2] + 18*b[3] - 18*b[4] + 18*b[5] + 18*b[6] - 18*b[7]
                + 12*b[8] + 6*b[9] - 12*b[10] - 6*b[11] - 12*b[12] - 6*b[13] + 12*b[14] + 6*b[15]
                + 12*b[16] - 12*b[17] + 6*b[18] - 6*b[19] - 12*b[20] + 12*b[21] - 6*b[22] + 6*b[23]
                + 9*b[24] - 9*b[25] - 9*b[26] + 9*b[27] + 9*b[28] - 9*b[29] - 9*b[30] + 9*b[31]
                + 8*b[32] + 4*b[33] + 4*b[34] + 2*b[35] - 8*b[36] - 4*b[37] - 4*b[38] - 2*b[39]
                + 6*b[40] + 3*b[41] - 6*b[42] - 3*b[43] + 6*b[44] + 3*b[45] - 6*b[46] - 3*b[47]
                + 6*b[48] - 6*b[49] + 3*b[50] - 3*b[51] + 6*b[52] - 6*b[53] + 3*b[54] - 3*b[55]
                + 4*b[56] + 2*b[57] + 2*b[58] + b[59] + 4*b[60] + 2*b[61] + 2*b[62] + b[63];
            a[59] = -12*b[0] + 12*b[1] + 12*b[2] - 12*b[3] + 12*b[4] - 12*b[5] - 12*b[6] + 12*b[7]
                - 6*b[8] - 6*b[9] + 6*b[10] + 6*b[11] + 6*b[12] + 6*b[13] - 6*b[14] - 6*b[15]
                - 8*b[16] + 8*b[17] - 4*b[18] + 4*b[19] + 8*b[20] - 8*b[21] + 4*b[22] - 4*b[23]
                - 6*b[24] + 6*b[25] + 6*b[26] - 6*b[27] - 6*b[28] + 6*b[29] + 6*b[30] - 6*b[31]
                - 4*b[32] - 4*b[33] - 2*b[34] - 2*b[35] + 4*b[36] + 4*b[37] + 2*b[38] + 2*b[39]
                - 3*b[40] - 3*b[41] + 3*b[42] + 3*b[43] - 3*b[44] - 3*b[45] + 3*b[46] + 3*b[47]
                - 4*b[48] + 4*b[49] - 2*b[50] + 2*b[51] - 4*b[52] + 4*b[53] - 2*b[54] + 2*b[55]
                - 2*b[56] - 2*b[57] - b[58] - b[59] - 2*b[60] - 2*b[61] - b[62] - b[63];
            a[60] = 4*b[0] - 4*b[2] - 4*b[4] + 4*b[6] + 2*b[16] + 2*b[18] - 2*b[20] - 2*b[22]
                + 2*b[24] - 2*b[26] + 2*b[28] - 2*b[30] + b[48] + b[50] + b[52] + b[54];
            a[61] = 4*b[8] - 4*b[10] - 4*b[12] + 4*b[14] + 2*b[32] + 2*b[34] - 2*b[36] - 2*b[38]
                + 2*b[40] - 2*b[42] + 2*b[44] - 2*b[46] + b[56] + b[58] + b[60] + b[62];
            a[62] = -12*b[0] + 12*b[1] + 12*b[2] - 12*b[3] + 12*b[4] - 12*b[5] - 12*b[6] + 12*b[7]
                - 8*b[8] - 4*b[9] + 8*b[10] + 4*b[11] + 8*b[12] + 4*b[13] - 8*b[14] - 4*b[15]
                - 6*b[16] + 6*b[17] - 6*b[18] + 6*b[19] + 6*b[20] - 6*b[21] + 6*b[22] - 6*b[23]
                - 6*b[24] + 6*b[25] + 6*b[26] - 6*b[27] - 6*b[28] + 6*b[29] + 6*b[30] - 6*b[31]
                - 4*b[32] - 2*b[33] - 4*b[34] - 2*b[35] + 4*b[36] + 2*b[37] + 4*b[38] + 2*b[39]
                - 4*b[40] - 2*b[41] + 4*b[42] + 2*b[43] - 4*b[44] - 2*b[45] + 4*b[46] + 2*b[47]
                - 3*b[48] + 3*b[49] - 3*b[50] + 3*b[51] - 3*b[52] + 3*b[53] - 3*b[54] + 3*b[55]
                - 2*b[56] - b[57] - 2*b[58] - b[59] - 2*b[60] - b[61] - 2*b[62] - b[63];
            a[63] = 8*b[0] - 8*b[1] - 8*b[2] + 8*b[3] - 8*b[4] + 8*b[5] + 8*b[6] - 8*b[7]
                + 4*b[8] + 4*b[9] - 4*b[10] - 4*b[11] - 4*b[12] - 4*b[13] + 4*b[14] + 4*b[15]
                + 4*b[16] - 4*b[17] + 4*b[18] - 4*b[19] - 4*b[20] + 4*b[21] - 4*b[22] + 4*b[23]
                + 4*b[24] - 4*b[25] - 4*b[26] + 4*b[27] + 4*b[28] - 4*b[29] - 4*b[30] + 4*b[31]
                + 2*b[32] + 2*b[33] + 2*b[34] + 2*b[35] - 2*b[36] - 2*b[37] - 2*b[38] - 2*b[39]
                + 2*b[40] + 2*b[41] - 2*b[42] - 2*b[43] + 2*b[44] + 2*b[45] - 2*b[46] - 2*b[47]
                + 2*b[48] - 2*b[49] + 2*b[50] - 2*b[51] + 2*b[52] - 2*b[53] + 2*b[54] - 2*b[55]
                + b[56] + b[57] + b[58] + b[59] + b[60] + b[61] + b[62] + b[63];
        }
DEVICE void RigidBodyGrid::compute_b(float * __restrict__ b, int * __restrict__ inds) const
        {
            int k[3];
            k[0] = nx;
            k[1] = ny;
            k[2] = nz;

            int inds2[3] = {0,0,0};

            for (int i0 = 0; i0 < 8; i0++) {
                inds2[0] = 0;
                inds2[1] = 0;
                inds2[2] = 0;

                /* printf("%d\n", inds2[0]); */
                /* printf("%d\n", inds2[1]); */
                /* printf("%d\n", inds2[2]); */

                bool zero_derivs = false;

                int bit = 1;    // bit = 2^i1 in the below loop
                for (int i1 = 0; i1 < 3; i1++) {
                    inds2[i1] = (inds[i1] + ((i0 & bit) ? 1 : 0)) % k[i1];
                    bit <<= 1;  // i.e. multiply by 2
                }
                //int d_hi[3] = {1, 1, 1};
                int d_lo[3] = {1, 1, 1};
                float voffs[3] = {0.0f, 0.0f, 0.0f};
                float dscales[3] = {0.5, 0.5, 0.5};

                for (int i1 = 0; i1 < 3; i1++) {
                    if (inds2[i1] == 0) {
                        zero_derivs = true;
                    }
                    else if (inds2[i1] == k[i1]-1) {
                        zero_derivs = true;
                    }
                    else {
                        voffs[i1] = 0.0;
                    }
                }

                // V
                b[i0] = getValue(inds2[0],inds2[1],inds2[2]);

                if (zero_derivs) {
                    b[8+i0] = 0.0;
                    b[16+i0] = 0.0;
                    b[24+i0] = 0.0;
                    b[32+i0] = 0.0;
                    b[40+i0] = 0.0;
                    b[48+i0] = 0.0;
                    b[56+i0] = 0.0;
                } else {
                    b[8+i0]  = dscales[0] * (getValue(inds2[0]+1,inds2[1],inds2[2]) - getValue(inds2[0]-d_lo[0],inds2[1],inds2[2]) + voffs[0]); //  dV/dx
                    b[16+i0] = dscales[1] * (getValue(inds2[0],inds2[1]+1,inds2[2]) - getValue(inds2[0],inds2[1]-d_lo[1],inds2[2]) + voffs[1]); //  dV/dy
                    b[24+i0] = dscales[2] * (getValue(inds2[0],inds2[1],inds2[2]+1) - getValue(inds2[0],inds2[1],inds2[2]-d_lo[2]) + voffs[2]); //  dV/dz
                    b[32+i0] = dscales[0] * dscales[1] *
                        (getValue(inds2[0]+1,inds2[1]+1,inds2[2]) - getValue(inds2[0]-d_lo[0],inds2[1]+1,inds2[2])
                       - getValue(inds2[0]+1,inds2[1]-d_lo[1],inds2[2]) + getValue(inds2[0]-d_lo[0],inds2[1]-d_lo[1],inds2[2]));      //  d2V/dxdy

                    b[40+i0] = dscales[0] * dscales[2] *
                              (getValue(inds2[0]+1,inds2[1],inds2[2]+1) - getValue(inds2[0]-d_lo[0],inds2[1],inds2[2]+1)
                             - getValue(inds2[0]+1,inds2[1],inds2[2]-d_lo[2]) + getValue(inds2[0]-d_lo[0],inds2[1],inds2[2]-d_lo[2]));      //  d2V/dxdz

                    b[48+i0] = dscales[1] * dscales[2] *
                               (getValue(inds2[0],inds2[1]+1,inds2[2]+1) - getValue(inds2[0],inds2[1]-d_lo[1],inds2[2]+1)
                              - getValue(inds2[0],inds2[1]+1,inds2[2]-d_lo[2]) + getValue(inds2[0],inds2[1]-d_lo[1],inds2[2]-d_lo[2]));      //  d2V/dydz

                    b[56+i0] = dscales[0] * dscales[1] * dscales[2] *                                    // d3V/dxdydz
                       (getValue(inds2[0]+1,inds2[1]+1,inds2[2]+1) - getValue(inds2[0]+1,inds2[1]+1,inds2[2]-d_lo[2]) -
                        getValue(inds2[0]+1,inds2[1]-d_lo[1],inds2[2]+1) - getValue(inds2[0]-d_lo[0],inds2[1]+1,inds2[2]+1) +
                        getValue(inds2[0]+1,inds2[1]-d_lo[1],inds2[2]-d_lo[2]) + getValue(inds2[0]-d_lo[0],inds2[1]+1,inds2[2]-d_lo[2]) +
                        getValue(inds2[0]-d_lo[0],inds2[1]-d_lo[1],inds2[2]+1) - getValue(inds2[0]-d_lo[0],inds2[1]-d_lo[1],inds2[2]-d_lo[2]));

                        }
                    }
                }

